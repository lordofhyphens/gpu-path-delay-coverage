#include "hip/hip_runtime.h"
#include "gpuiscas.h"

void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
		if (err == hipErrorInvalidValue)
			DPRINT("hipErrorInvalidValue: ");
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            DPRINT("Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

GPUNODE* gpuLoadCircuit(const GPUNODE* graph, int maxid) {
	GPUNODE *devAr, *testAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(GPUNODE)*(maxid)));
	HANDLE_ERROR(hipMemcpy(devAr, graph, (maxid) * sizeof(GPUNODE),hipMemcpyHostToDevice));
//	DPRINT("Verifying GPUNODE graph copy\n");
	testAr = (GPUNODE*)malloc(sizeof(GPUNODE)*(maxid));	
	HANDLE_ERROR(hipMemcpy(testAr, devAr, (maxid) * sizeof(GPUNODE),hipMemcpyDeviceToHost));

	for (int i = 0; i < maxid; i++) {
		assert(testAr[i].type == graph[i].type && testAr[i].nfi == graph[i].nfi &&testAr[i].nfo == graph[i].nfo && testAr[i].po == graph[i].po && testAr[i].offset == graph[i].offset);
	}
	free(testAr);
	return devAr;
}
LINE* gpuLoadLines(LINE* graph, int maxid) {
	LINE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(LINE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, sizeof(LINE)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
int* gpuLoadFans(int* offset, int maxid) {
	int* devAr = NULL;
	hipError_t returncode;
	returncode = hipMalloc(&devAr, sizeof(int)*maxid);
	assert(returncode == hipSuccess);
	assert(devAr != NULL);
	returncode = hipMemcpy(devAr, offset, sizeof(int)*maxid,hipMemcpyHostToDevice);
	assert(returncode == hipSuccess);
#ifndef NDEBUG
		int *tmp = (int*)malloc(sizeof(int)*maxid);
		assert(tmp != NULL);
		for (int r =0; r < maxid;r++)
			tmp[r] = -1;
		hipMemcpy(tmp, devAr, sizeof(int)*(maxid),hipMemcpyDeviceToHost);
		for (int i = 0; i < maxid; i++) {
			assert(offset[i]==tmp[i]);
		}
		free(tmp);
#endif // debugging memory check and assertion
	return devAr;
}
void gpuShiftVectors(int* input, size_t width, size_t height) {
	int* tgt = NULL;
	// create a temporary buffer area on the device
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width)));
	HANDLE_ERROR(hipMemcpy(tgt, input,sizeof(int)*(width),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input, input+width,sizeof(int)*(width)*(height-1),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input+(height-1)*(width),tgt, sizeof(int)*(width), hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipFree(tgt));
}
ARRAY2D<int> gpuAllocateResults(size_t width, size_t height) {
	int *tgt = NULL;
	size_t pitch = 0;
	DPRINT("Attempting to allocate %u * %u = %lu bytes... %G megabytes ",(int)sizeof(unsigned)*(unsigned)width,(unsigned)height, sizeof(int)*width*height, sizeof(int)*width*height / pow(2,20));
	HANDLE_ERROR(hipMallocPitch(&tgt, &pitch, sizeof(int)*(width),height));
	DPRINT("...complete.\n");
	DPRINT("Allocated %u*%u = %lu bytes, %G megabytes\n", (unsigned)pitch,(unsigned)height, pitch*height, (pitch*width)/pow(2,20));
	HANDLE_ERROR(hipMemset2D(tgt, pitch,0, sizeof(int)*width,height));
	return ARRAY2D<int>(tgt, height, width, pitch);
}
int* gpuLoad1DVector(int* input, size_t width, size_t height) {
	int *tgt;
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width)*(height)));
	HANDLE_ERROR(hipMemcpy(tgt, input,sizeof(int)*(width)*(height),hipMemcpyHostToDevice));
	return tgt;
}
int* loadPinned(int* input, size_t vcnt) {
	int* tgt;
	hipHostMalloc(&tgt, vcnt*sizeof(int));
	hipMemcpy(tgt, input, sizeof(int)*vcnt, hipMemcpyHostToHost);
	return tgt;
}
void freeMemory(int* data) {
	hipFree(data);
}
void freeMemory(GPUNODE* data) {
	hipFree(data);
}

