#include <hip/hip_runtime.h>
#include "iscas.h"
#include "gpuiscas.h"
#define NDEBUG
#include "defines.h"
#include <cassert>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            DPRINT( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

GPUNODE* gpuLoadCircuit(const GPUNODE* graph, int maxid) {
	GPUNODE *devAr, *testAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(GPUNODE)*(1+maxid)));
	HANDLE_ERROR(hipMemcpy(devAr, graph, (maxid+1) * sizeof(GPUNODE),hipMemcpyHostToDevice));
	DPRINT("Verifying GPUNODE graph copy\n");
	DPRINT("ID\tTYPE\tFANIN\tFANOUT\tPO\tOFFSET\n");
	testAr = (GPUNODE*)malloc(sizeof(GPUNODE)*(maxid+1));	
	HANDLE_ERROR(hipMemcpy(testAr, devAr, (1+maxid) * sizeof(GPUNODE),hipMemcpyDeviceToHost));

	for (int i = 0; i <= maxid; i++) {
		DPRINT("%d:\t%d\t%d\t%d\t%d\t%d\n", i, testAr[i].type,testAr[i].nfi,testAr[i].nfo,testAr[i].po,testAr[i].offset);
		assert(testAr[i].type == graph[i].type && testAr[i].nfi == graph[i].nfi &&testAr[i].nfo == graph[i].nfo && testAr[i].po == graph[i].po && testAr[i].offset == graph[i].offset);
	}
	free(testAr);
	return devAr;
}
LINE* gpuLoadLines(LINE* graph, int maxid) {
	LINE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(LINE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, sizeof(LINE)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
int* gpuLoadFans(int* offset, int maxid) {
	int* devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(int)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, offset, sizeof(int)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
void gpuShiftVectors(int* loca, size_t width, size_t height) {
	int* tgt;
	// create a temporary buffer area on the device
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width+1)));
	hipMemcpy(tgt, input+sizeof(int)*(width+1)*(height+1)-sizeof(int)*(width+1),sizeof(int)*(width+1),hipMemcpyDeviceToDevice);
	hipMemcpy(input, input+(sizeof(int)*width+1),sizeof(int)*(width+1)*(height+1) - sizeof(int)*(width+1),hipMemcpyDeviceToDevice);
	hipMemcpy(input+sizeof(int)*(width+1)*(height+1)-sizeof(int)*(width+1),tgt, sizeof(int)*(width+1), hipMemcpyDeviceToDevice);
	hipFree(tgt);
}
int* gpuLoadVectors(int** input, size_t width, size_t height) {
	int *tgt;
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width+1)*(height+1)));
	int *row;
	for (int i = 0; i < height; i++) {
		row = (int*)((char*)tgt + i*(width)*sizeof(int));
		hipMemcpy(row, input[i],sizeof(int)*(width+1),hipMemcpyHostToDevice);
#ifndef NDEBUG
		int *tmp = (int*)malloc(sizeof(int)*width);
		for (int i =0; i <= width; i++)
			tmp[i] = -1;
		hipMemcpy(tmp, row, sizeof(int)*(width+1),hipMemcpyDeviceToHost);
		for (int j = 0; j <= width; j++) {
			assert(input[i][j]==tmp[j]);
		}
		free(tmp);
#endif // debugging memory check and assertion
	}
	return tgt;
}
int* gpuLoad1DVector(int* input, size_t width, size_t height) {
	int *tgt, *row;
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width+1)*(height+1)));
	hipMemcpy(tgt, input,sizeof(int)*(width+1)*(height+1),hipMemcpyHostToDevice);
	return tgt;
}
