#include <hip/hip_runtime.h>
#include "iscas.h"
#include "gpuiscas.h"
#define NDEBUG
#include "defines.h"
#include <cassert>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf("Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

GPUNODE* gpuLoadCircuit(const GPUNODE* graph, int maxid) {
	GPUNODE *devAr, *testAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(GPUNODE)*(1+maxid)));
	HANDLE_ERROR(hipMemcpy(devAr, graph, (maxid+1) * sizeof(GPUNODE),hipMemcpyHostToDevice));
	DPRINT("Verifying GPUNODE graph copy\n");
	testAr = (GPUNODE*)malloc(sizeof(GPUNODE)*(maxid+1));	
	HANDLE_ERROR(hipMemcpy(testAr, devAr, (1+maxid) * sizeof(GPUNODE),hipMemcpyDeviceToHost));

	for (int i = 0; i <= maxid; i++) {
		assert(testAr[i].type == graph[i].type && testAr[i].nfi == graph[i].nfi &&testAr[i].nfo == graph[i].nfo && testAr[i].po == graph[i].po && testAr[i].offset == graph[i].offset);
	}
	free(testAr);
	return devAr;
}
LINE* gpuLoadLines(LINE* graph, int maxid) {
	LINE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(LINE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, sizeof(LINE)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
int* gpuLoadFans(int* offset, int maxid) {
	int* devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(int)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, offset, sizeof(int)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
void gpuShiftVectors(int* input, size_t width, size_t height) {
	int* tgt;
	// create a temporary buffer area on the device
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width)));
	HANDLE_ERROR(hipMemcpy(tgt, input,sizeof(int)*(width),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input, input+width,sizeof(int)*(width)*(height-1),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input+(height-1)*(width),tgt, sizeof(int)*(width), hipMemcpyDeviceToDevice));
	hipFree(tgt);
}
int* gpuLoadVectors(int** input, size_t width, size_t height) {
	int *tgt;
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width+1)*(height+1)));
	int *row;
	for (int i = 0; i < height; i++) {
		row = (int*)((char*)tgt + i*(width)*sizeof(int));
		hipMemcpy(row, input[i],sizeof(int)*(width+1),hipMemcpyHostToDevice);
#ifndef NDEBUG
		int *tmp = (int*)malloc(sizeof(int)*width);
		for (int i =0; i <= width; i++)
			tmp[i] = -1;
		hipMemcpy(tmp, row, sizeof(int)*(width+1),hipMemcpyDeviceToHost);
		for (int j = 0; j <= width; j++) {
			assert(input[i][j]==tmp[j]);
		}
		free(tmp);
#endif // debugging memory check and assertion
	}
	return tgt;
}
int* gpuLoad1DVector(int* input, size_t width, size_t height) {
	int *tgt;
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width+1)*(height+1)));
	hipMemcpy(tgt, input,sizeof(int)*(width+1)*(height+1),hipMemcpyHostToDevice);
	return tgt;
}
