#include <hip/hip_runtime.h>
#include "iscas.h"
#include "gpuiscas.h"
#include "defines.h"
#include <cassert>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf("Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

GPUNODE* gpuLoadCircuit(const GPUNODE* graph, int maxid) {
	GPUNODE *devAr, *testAr;
	hipError_t returncode;
	returncode = hipMalloc(&devAr, sizeof(GPUNODE)*(maxid));
	assert(returncode == hipSuccess);
	HANDLE_ERROR(hipMemcpy(devAr, graph, (maxid) * sizeof(GPUNODE),hipMemcpyHostToDevice));
//	DPRINT("Verifying GPUNODE graph copy\n");
	testAr = (GPUNODE*)malloc(sizeof(GPUNODE)*(maxid));	
	HANDLE_ERROR(hipMemcpy(testAr, devAr, (maxid) * sizeof(GPUNODE),hipMemcpyDeviceToHost));

	for (int i = 0; i < maxid; i++) {
		assert(testAr[i].type == graph[i].type && testAr[i].nfi == graph[i].nfi &&testAr[i].nfo == graph[i].nfo && testAr[i].po == graph[i].po && testAr[i].offset == graph[i].offset);
	}
	free(testAr);
	return devAr;
}
LINE* gpuLoadLines(LINE* graph, int maxid) {
	LINE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(LINE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, sizeof(LINE)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
int* gpuLoadFans(int* offset, int maxid) {
	int* devAr = NULL;
	hipError_t returncode;
	returncode = hipMalloc(&devAr, sizeof(int)*maxid);
	assert(returncode == hipSuccess);
	assert(devAr != NULL);
	returncode = hipMemcpy(devAr, offset, sizeof(int)*maxid,hipMemcpyHostToDevice);
	assert(returncode == hipSuccess);
#ifndef NDEBUG
		int *tmp = (int*)malloc(sizeof(int)*maxid);
		assert(tmp != NULL);
		for (int r =0; r < maxid;r++)
			tmp[r] = -1;
		hipMemcpy(tmp, devAr, sizeof(int)*(maxid),hipMemcpyDeviceToHost);
		for (int i = 0; i < maxid; i++) {
			assert(offset[i]==tmp[i]);
		}
		free(tmp);
#endif // debugging memory check and assertion
	return devAr;
}
void gpuShiftVectors(int* input, size_t width, size_t height) {
	int* tgt = NULL;
	// create a temporary buffer area on the device
	hipError_t returncode;
	returncode = hipMalloc(&tgt, sizeof(int)*(width));
	assert(returncode == hipSuccess);
	assert(tgt != NULL);
	HANDLE_ERROR(hipMemcpy(tgt, input,sizeof(int)*(width),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input, input+width,sizeof(int)*(width)*(height-1),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input+(height-1)*(width),tgt, sizeof(int)*(width), hipMemcpyDeviceToDevice));
	hipFree(tgt);
}
int* gpuLoadVectors(int** input, size_t width, size_t height) {
	int *tgt = NULL;
	hipError_t returncode;
	returncode = hipMalloc(&tgt, sizeof(int)*(width)*(height));
	assert(returncode == hipSuccess);
	returncode = hipMemset(tgt, 0, sizeof(int)*width*height);
	assert(returncode == hipSuccess);
	/*
	int *row;
	for (int i = 0; i < height; i++) {
		row = (int*)((char*)tgt + i*(width)*sizeof(int));
		hipMemcpy(row, input[i],sizeof(int)*(width+1),hipMemcpyHostToDevice);
#ifndef NDEBUG
		int *tmp = (int*)malloc(sizeof(int)*width);
		for (int r =0; r <= width;r++)
			tmp[r] = -1;
		hipMemcpy(tmp, row, sizeof(int)*(width+1),hipMemcpyDeviceToHost);
		for (int j = 0; j <= width; j++) {
			assert(input[i][j]==tmp[j]);
		}
		free(tmp);
#endif // debugging memory check and assertion
	}*/
	return tgt;
}
int* gpuLoad1DVector(int* input, size_t width, size_t height) {
	int *tgt;
	hipError_t returncode; 
	returncode = hipMalloc(&tgt, sizeof(int)*(width)*(height));
	assert(returncode == hipSuccess);
	returncode = hipMemcpy(tgt, input,sizeof(int)*(width)*(height),hipMemcpyHostToDevice);
	assert(returncode == hipSuccess);
	return tgt;
}
int* loadPinned(int* input, size_t vcnt) {
	int* tgt;
	hipHostMalloc(&tgt, vcnt*sizeof(int));
	hipMemcpy(tgt, input, sizeof(int)*vcnt, hipMemcpyHostToHost);
	return tgt;
}
void freeMemory(int* data) {
	hipFree(data);
}
void freeMemory(GPUNODE* data) {
	hipFree(data);
}

