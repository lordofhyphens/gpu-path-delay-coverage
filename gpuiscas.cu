#include <hip/hip_runtime.h>
#include <cassert>
#include "iscas.h"
#include "gpuiscas.h"
#include "defines.h"
#define N 32
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            DPRINT( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

GPUNODE* gpuLoadCircuit(const GPUNODE* graph, int maxid) {
	GPUNODE *devAr, *testAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(GPUNODE)*(1+maxid)));
	HANDLE_ERROR(hipMemcpy(devAr, graph, (maxid+1) * sizeof(GPUNODE),hipMemcpyHostToDevice));
	DPRINT("Verifying GPUNODE graph copy\n");
	DPRINT("ID\tTYPE\tFANIN\tFANOUT\tPO\tOFFSET\n");
	testAr = (GPUNODE*)malloc(sizeof(GPUNODE)*(maxid+1));	
	HANDLE_ERROR(hipMemcpy(testAr, devAr, (1+maxid) * sizeof(GPUNODE),hipMemcpyDeviceToHost));

	for (int i = 0; i <= maxid; i++) {
//		DPRINT("%d:\t%d\t%d\t%d\t%d\t%d\n", i, testAr[i].type,testAr[i].nfi,testAr[i].nfo,testAr[i].po,testAr[i].offset);
		assert(testAr[i].type == graph[i].type && testAr[i].nfi == graph[i].nfi &&testAr[i].nfo == graph[i].nfo && testAr[i].po == graph[i].po && testAr[i].offset == graph[i].offset);
	}
	free(testAr);
	return devAr;
}
LINE* gpuLoadLines(LINE* graph, int maxid) {
	LINE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(LINE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, sizeof(LINE)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
int* gpuLoadFans(int* offset, int maxid) {
	int* devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(int)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, offset, sizeof(int)*maxid,hipMemcpyHostToDevice));
	return devAr;
}

int* gpuLoadVectors(int** input, size_t width, size_t height) {
	int *tgt;
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width+1)*(height+1)));
	int *row;
	int *tmp = (int*)malloc(sizeof(int)*width);
	for (int i =0; i <= width; i++)
		tmp[i] = -1;
	for (int i = 0; i < height; i++) {
		row = (int*)((char*)tgt + i*(width)*sizeof(int));
		hipMemcpy(row, input[i],sizeof(int)*(width+1),hipMemcpyHostToDevice);
#ifndef NDEBUG
		hipMemcpy(tmp, row, sizeof(int)*(width+1),hipMemcpyDeviceToHost);
		for (int j = 0; j <= width; j++) {
			assert(input[i][j]==tmp[j]);
		}
#endif
	}
	return tgt;
}
