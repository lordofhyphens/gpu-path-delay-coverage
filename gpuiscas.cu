#include <hip/hip_runtime.h>
#include "iscas.h"
#include "gpuiscas.h"
#define N 32
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

NODE* gpuLoadCircuit(const NODE* graph, int maxid) {
	NODE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(NODE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, maxid * sizeof(NODE),hipMemcpyHostToDevice));
	return devAr;
}
LINE* gpuLoadLines(LINE* graph, int maxid) {
	LINE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(LINE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, sizeof(LINE)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
int* gpuLoadFans(int* offset, int maxid) {
	int* devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(int)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, offset, sizeof(int)*maxid,hipMemcpyHostToDevice));
	return devAr;
}

int* gpuLoadVectors(int** input, size_t width, size_t height) {
	int *tgt;
	size_t pitch;
	HANDLE_ERROR(hipMallocPitch(&tgt, &pitch, width, height));
	HANDLE_ERROR(hipMemcpy2D(tgt,pitch,input,pitch,width,height,hipMemcpyHostToDevice));
	return tgt;
}
