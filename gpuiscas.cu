#include <hip/hip_runtime.h>
#include "iscas.h"
#include "gpuiscas.h"
#include "defines.h"
#include <cassert>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf("Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

GPUNODE* gpuLoadCircuit(const GPUNODE* graph, int maxid) {
	GPUNODE *devAr, *testAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(GPUNODE)*(1+maxid)));
	HANDLE_ERROR(hipMemcpy(devAr, graph, (maxid+1) * sizeof(GPUNODE),hipMemcpyHostToDevice));
//	DPRINT("Verifying GPUNODE graph copy\n");
	testAr = (GPUNODE*)malloc(sizeof(GPUNODE)*(maxid+1));	
	HANDLE_ERROR(hipMemcpy(testAr, devAr, (1+maxid) * sizeof(GPUNODE),hipMemcpyDeviceToHost));

	for (int i = 0; i <= maxid; i++) {
		assert(testAr[i].type == graph[i].type && testAr[i].nfi == graph[i].nfi &&testAr[i].nfo == graph[i].nfo && testAr[i].po == graph[i].po && testAr[i].offset == graph[i].offset);
	}
	free(testAr);
	return devAr;
}
LINE* gpuLoadLines(LINE* graph, int maxid) {
	LINE *devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(LINE)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, graph, sizeof(LINE)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
int* gpuLoadFans(int* offset, int maxid) {
	int* devAr;
	HANDLE_ERROR(hipMalloc(&devAr, sizeof(int)*maxid));
	HANDLE_ERROR(hipMemcpy(devAr, offset, sizeof(int)*maxid,hipMemcpyHostToDevice));
	return devAr;
}
void gpuShiftVectors(int* input, size_t width, size_t height) {
	int* tgt;
	// create a temporary buffer area on the device
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width)));
	HANDLE_ERROR(hipMemcpy(tgt, input,sizeof(int)*(width),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input, input+width,sizeof(int)*(width)*(height-1),hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(input+(height-1)*(width),tgt, sizeof(int)*(width), hipMemcpyDeviceToDevice));
	hipFree(tgt);
}
int* gpuLoadVectors(int** input, size_t width, size_t height) {
	int *tgt;
	HANDLE_ERROR(hipMalloc(&tgt, sizeof(int)*(width)*(height)));
	hipMemset(tgt, 0, sizeof(int)*width*height);
	/*
	int *row;
	for (int i = 0; i < height; i++) {
		row = (int*)((char*)tgt + i*(width)*sizeof(int));
		hipMemcpy(row, input[i],sizeof(int)*(width+1),hipMemcpyHostToDevice);
#ifndef NDEBUG
		int *tmp = (int*)malloc(sizeof(int)*width);
		for (int r =0; r <= width;r++)
			tmp[r] = -1;
		hipMemcpy(tmp, row, sizeof(int)*(width+1),hipMemcpyDeviceToHost);
		for (int j = 0; j <= width; j++) {
			assert(input[i][j]==tmp[j]);
		}
		free(tmp);
#endif // debugging memory check and assertion
	}*/
	return tgt;
}
int* gpuLoad1DVector(int* input, size_t width, size_t height) {
	int *tgt;
	hipError_t returncode; 
	returncode = hipMalloc(&tgt, sizeof(int)*(width)*(height));
	assert(returncode == hipSuccess);
//	DPRINT("Malloc is fine...");
	returncode = hipMemcpy(tgt, input,sizeof(int)*(width)*(height),hipMemcpyHostToDevice);
//	DPRINT("Memcpy is fine...");
	assert(returncode == hipSuccess);
	return tgt;
}
int* loadPinned(int* input, size_t vcnt) {
	int* tgt;
	hipHostMalloc(&tgt, vcnt*sizeof(int));
	hipMemcpy(tgt, input, sizeof(int)*vcnt, hipMemcpyHostToHost);
	return tgt;
}
void freeMemory(int* data) {
	hipFree(data);
}
void freeMemory(GPUNODE* data) {
	hipFree(data);
}

