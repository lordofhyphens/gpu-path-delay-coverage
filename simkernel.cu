#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "defines.h"
#include "iscas.h"
#include "simkernel.h"

#define THREAD_PER_BLOCK 256
texture<int, 2> and2LUT;
texture<int, 2> nand2LUT;
texture<int, 2> or2LUT;
texture<int, 2> nor2LUT;
texture<int, 2> xor2LUT;
texture<int, 2> xnor2LUT;
texture<int, 2> stableLUT;
texture<int, 1> notLUT;

__global__ void kernSimulate(GPUNODE* graph, int* res, int* input, int* fans, size_t iwidth, size_t width, size_t height, int pass) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	__shared__ int rowids[100]; // handle up to fanins of 1000 / 
	int pi = 0;
	int *row;
	int goffset, nfi, val, j,type, r;
	if (tid < height) {
		row = (int*)((char*)res + tid*width*sizeof(int)); // get the current row?
		for (int i = 0; i < width; i++) {
			nfi = graph[i].nfi;
			if (threadIdx.x == 0) { // first thread in every block does the preload.
				goffset = graph[i].offset;
//				printf("Offset (gate %d): %d\n", i, goffset);
				// preload all of the fanin line #s for this gate to shared memory.
				for (int j = 0; j < nfi;j++) {
					rowids[j] = fans[goffset+j];
//					printf("Gate %d, fanin %d = %d (wrote %d)\n",i, j, fans[goffset+j],rowids[j]);
				}
					
			}
			__syncthreads();
			type = graph[i].type;
			switch (type) {
				case 0: break;
				case INPT:
						val = input[(pi+iwidth*tid)];
						if (pass > 1) {
							row[i] = tex2D(stableLUT, row[i], val);  
						} else {
							row[i] = val;
						}
						pi++;
						break;
				default: 
						// we're guaranteed at least one fanin per 
						// gate if not on an input.
						__syncthreads();
						if (rowids[0] < 0) {
							printf("T: %d Node %d, Type %d, Rowid0 %d\n", tid, i, graph[i].type, rowids[0]);
						}
						if (graph[i].type != NOT) {
							val = row[rowids[0]];
						} else {
							val = tex1D(notLUT, row[rowids[0]]);
						}
						
						j = 1;
						while (j < nfi) {
							r = row[rowids[j]]; 
							switch(type) {
								case XOR:
									val = tex2D(xor2LUT, val, r );break;
								case XNOR:
									val = tex2D(xnor2LUT, val, r);break;
								case OR:
									val = tex2D(or2LUT, val, r);break;
								case NOR:
									val = tex2D(nor2LUT, val, r);break;
								case AND:
									val = tex2D(and2LUT, val, r);break;
								case NAND:
									if (tid == 664) { 
//										printf("\n");
									}
									val = tex2D(nand2LUT, val, r);break;
							}
							j++;
						}
						if (pass > 1 && type != FROM && type != BUFF) {
							row[i] = tex2D(stableLUT, row[i], val);  
						} else {
							row[i] = val;
						}
			}

		}
	}
}

void loadSimLUTs() {
	int nand2[16] = {1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0};
	int and2[16]  = {0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1};
	int nor2[16]  = {1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0};
	int or2[16]   = {0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1};
	int xnor2[16] = {1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1};
	int xor2[16]  = {0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0};
	int stable[4] = {S0, T0, T1, S1};
	int not_gate[4] = {1, 1, 0, 0};

	// device memory arrays, required. 
	hipArray *cuNandArray, *cuAndArray,*cuNorArray, *cuOrArray,*cuXnorArray,*cuXorArray, *cuNotArray,*cuStableArray;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();

	// Allocate space in device memory for the LUTs. 
	hipMallocArray(&cuNandArray, &channelDesc, 4,4);
	hipMallocArray(&cuAndArray, &channelDesc, 4,4);
	hipMallocArray(&cuNorArray, &channelDesc, 4,4);
	hipMallocArray(&cuOrArray, &channelDesc, 4,4);
	hipMallocArray(&cuXnorArray, &channelDesc, 4,4);
	hipMallocArray(&cuXorArray, &channelDesc, 4,4);
	hipMallocArray(&cuStableArray, &channelDesc, 2,2);
	hipMallocArray(&cuNotArray, &channelDesc, 4,1);

	// Copying the static arrays given to device memory.
	hipMemcpyToArray(cuNandArray, 0,0, nand2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuAndArray, 0,0, and2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuNorArray, 0,0, nor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuOrArray, 0,0, or2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXnorArray, 0,0, xnor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXorArray, 0,0, xor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuStableArray, 0,0, stable, sizeof(int)*4,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuNotArray, 0,0, not_gate, sizeof(int)*4,hipMemcpyHostToDevice);

	hipBindTextureToArray(and2LUT,cuAndArray,channelDesc);
	hipBindTextureToArray(nand2LUT,cuNandArray,channelDesc);
	hipBindTextureToArray(or2LUT,cuOrArray,channelDesc);
	hipBindTextureToArray(nor2LUT,cuNorArray,channelDesc);
	hipBindTextureToArray(xor2LUT,cuXorArray,channelDesc);
	hipBindTextureToArray(xnor2LUT,cuXnorArray,channelDesc);
	hipBindTextureToArray(stableLUT,cuStableArray,channelDesc);
	hipBindTextureToArray(notLUT,cuNotArray,channelDesc);
}

float gpuRunSimulation(ARRAY2D<int> results, ARRAY2D<int> inputs, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, int* fan, int pass = 1) {
	loadSimLUTs(); // set up our lookup tables for simulation.
#ifndef NTIMING
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
#endif // NTIMING
	int blockcount = (int)(results.height/THREAD_PER_BLOCK) + (results.height%THREAD_PER_BLOCK > 0);
//	DPRINT("Block count: %d, threads: %d\n", blockcount, THREAD_PER_BLOCK);
	kernSimulate<<<blockcount,THREAD_PER_BLOCK>>>(dgraph.data,results.data, inputs.data,fan,inputs.width, results.width, results.height, pass);
	hipDeviceSynchronize();

	// We're done simulating at this point.
#ifndef NTIMING
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}

void debugSimulationOutput(ARRAY2D<int> results, int pass = 1) {
#ifndef NDEBUG
	int *lvalues, *row;
	DPRINT("Post-simulation device results, pass %d:\n\n", pass);
	DPRINT("Line:   \t");
	for (int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (int r = 0;r < results.height; r++) {
		lvalues = (int*)malloc(results.bwidth());
		row = (int*)((char*)results.data + r*results.bwidth()); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		DPRINT("%s %d:\t", pass > 1 ? "Vector" : "Pattern",r);
		for (int i = 0; i < results.width; i++) {
			switch(lvalues[i]) {
				case S0:
					DPRINT("S0 "); break;
				case S1:
					DPRINT("S1 "); break;
				case T0:
					DPRINT("T0 "); break;
				case T1:
					DPRINT("T1 "); break;
				default:
					DPRINT("%2d ", lvalues[i]); break;
			}
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif
}
