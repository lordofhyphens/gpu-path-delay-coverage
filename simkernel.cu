#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "defines.h"
#include "iscas.h"
#include "simkernel.h"

void HandleSimError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleSimError( err, __FILE__, __LINE__ ))
#define THREAD_PER_BLOCK 512
texture<int, 2> and2LUT;
texture<int, 2> nand2LUT;
texture<int, 2> or2LUT;
texture<int, 2> nor2LUT;
texture<int, 2> xor2LUT;
texture<int, 2> xnor2LUT;
texture<int, 2> stableLUT;
texture<int, 1> notLUT;

__global__ void kernSimulate(GPUNODE* graph, char* res, int* input, int* fans, size_t iwidth, size_t width, size_t height, size_t pitch, int start, int level, int pass) {
	int tid = (blockIdx.y * blockDim.y) + threadIdx.x;
	int gid = blockIdx.x;
	__shared__ int rowids[100]; // handle up to fanins of 1000 / 
	char *row;
	int goffset, nfi, val, j,type, r;
	if (tid < height) {
		row = ((char*)res + tid*pitch); // get the current row?
		int i = gid + start;
		nfi = graph[i].nfi;
		if (threadIdx.x == 0) { // first thread in every block does the preload.
			goffset = graph[i].offset;
			// preload all of the fanin line #s for this gate to shared memory.
			for (int j = 0; j < nfi;j++) {
				rowids[j] = fans[goffset+j];
			}

		}
		__syncthreads();
		type = graph[i].type;
		switch (type) {
			case 0: break;
			case INPT:
					val = input[(gid+iwidth*tid)];
					if (pass > 1) {
						row[i] = tex2D(stableLUT, row[i], val);  
					} else {
						row[i] = val;
					}
					break;
			default: 
					// we're guaranteed at least one fanin per 
					// gate if not on an input.
					__syncthreads();
					if (rowids[0] < 0) {
						printf("T: %d Node %d, Type %d, Rowid0 %d\n", tid, i, graph[i].type, rowids[0]);
					}
					if (graph[i].type != NOT) {
						val = row[rowids[0]];
					} else {
						val = tex1D(notLUT, row[rowids[0]]);
					}

					j = 1;
					while (j < nfi) {
						r = row[rowids[j]]; 
						switch(type) {
							case XOR:
								val = tex2D(xor2LUT, val, r );break;
							case XNOR:
								val = tex2D(xnor2LUT, val, r);break;
							case OR:
								val = tex2D(or2LUT, val, r);break;
							case NOR:
								val = tex2D(nor2LUT, val, r);break;
							case AND:
								val = tex2D(and2LUT, val, r);break;
							case NAND:
								val = tex2D(nand2LUT, val, r);break;
						}
						j++;
					}
					if (pass > 1 && type != FROM && type != BUFF) {
						row[i] = tex2D(stableLUT, row[i], val);  
					} else {
						row[i] = val;
					}
		}

	}
}

void loadSimLUTs() {
	int nand2[16] = {1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0};
	int and2[16]  = {0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1};
	int nor2[16]  = {1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0};
	int or2[16]   = {0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1};
	int xnor2[16] = {1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1};
	int xor2[16]  = {0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0};
	int stable[4] = {S0, T0, T1, S1};
	int not_gate[4] = {1, 1, 0, 0};

	// device memory arrays, required. 
	hipArray *cuNandArray, *cuAndArray,*cuNorArray, *cuOrArray,*cuXnorArray,*cuXorArray, *cuNotArray,*cuStableArray;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();

	// Allocate space in device memory for the LUTs. 
	HANDLE_ERROR(hipMallocArray(&cuNandArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuAndArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuNorArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXnorArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuStableArray, &channelDesc, 2,2));
	HANDLE_ERROR(hipMallocArray(&cuNotArray, &channelDesc, 4,1));

	// Copying the static arrays given to device memory.
	HANDLE_ERROR(hipMemcpyToArray(cuNandArray, 0,0, nand2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndArray, 0,0, and2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuNorArray, 0,0, nor2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrArray, 0,0, or2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXnorArray, 0,0, xnor2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorArray, 0,0, xor2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuStableArray, 0,0, stable, sizeof(int)*4,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuNotArray, 0,0, not_gate, sizeof(int)*4,hipMemcpyHostToDevice));

	HANDLE_ERROR(hipBindTextureToArray(and2LUT,cuAndArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(nand2LUT,cuNandArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2LUT,cuOrArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(nor2LUT,cuNorArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2LUT,cuXorArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xnor2LUT,cuXnorArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(stableLUT,cuStableArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(notLUT,cuNotArray,channelDesc));
}

float gpuRunSimulation(ARRAY2D<char> results, ARRAY2D<int> inputs, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, int* fan, int maxlevels, int pass = 1) {
	loadSimLUTs(); // set up our lookup tables for simulation.
	int startGate = 0;
	int *gatesinLevel;
	gatesinLevel = new int[maxlevels];
	for (int i = 0; i < maxlevels; i++) {
		gatesinLevel[i] = 0;
		for (unsigned int j = 0; j < results.width; j++) {
			if (graph[j].level == i) {
				gatesinLevel[i]++;
			}
		}
	}
	int blockcount_y = (int)(results.height/THREAD_PER_BLOCK) + (results.height%THREAD_PER_BLOCK > 0);
#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	for (int i = 0; i < maxlevels; i++) {
		dim3 numBlocks(gatesinLevel[i],blockcount_y);
//		DPRINT("Level %d: Simulating %d gates in parallel.\n",i,gatesinLevel[i]);
		kernSimulate<<<numBlocks,THREAD_PER_BLOCK>>>(dgraph.data,results.data, inputs.data,fan,inputs.pitch, results.width, results.height, results.pitch, startGate, i, pass);
		startGate += gatesinLevel[i];
	}
	hipDeviceSynchronize();
	// We're done simulating at this point.
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}

void debugSimulationOutput(ARRAY2D<char> results, int pass = 1) {
#ifndef NDEBUG
	char *lvalues, *row;
	DPRINT("Post-simulation device results, pass %d:\n\n", pass);
	DPRINT("Line:   \t");
	for (unsigned int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (unsigned int r = 0;r < results.height; r++) {
		lvalues = (char*)malloc(results.bwidth());
		row = ((char*)results.data + r*results.pitch); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		DPRINT("%s %d:\t", pass > 1 ? "Vector" : "Pattern",r);
		for (unsigned int i = 0; i < results.width; i++) {
			switch(lvalues[i]) {
				case S0:
					DPRINT("S0 "); break;
				case S1:
					DPRINT("S1 "); break;
				case T0:
					DPRINT("T0 "); break;
				case T1:
					DPRINT("T1 "); break;
				default:
					DPRINT("%2d ", lvalues[i]); break;
			}
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif
}
