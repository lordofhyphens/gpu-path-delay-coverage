#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "defines.h"
#include "iscas.h"
#include "simkernel.h"

void HandleSimError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleSimError( err, __FILE__, __LINE__ ))
texture<int, 2> and2LUT;
texture<int, 2> nand2LUT;
texture<int, 2> or2LUT;
texture<int, 2> nor2LUT;
texture<int, 2> xor2LUT;
texture<int, 2> xnor2LUT;
texture<int, 2> stableLUT;
texture<int, 1> notLUT;
__global__ void kernSimulateP1(GPUNODE* graph, int* pi_data, size_t pi_pitch, char* output_data, size_t pitch,size_t pattern_count, int* fanout_index, int start_offset) {
	int tid = (blockIdx.y * blockDim.y) + threadIdx.x;
	int gid = blockIdx.x+start_offset;
	__shared__ char rowcache[SIM_BLOCK];
	char *row, r, val;
	int goffset, nfi, j,type;
	if (tid < pattern_count) {
		row = ((char*)output_data + gid*pitch); // get the line row for the current gate
		goffset = graph[gid].offset;
		nfi = graph[gid].nfi;
		type = graph[gid].type;

		__syncthreads();
		rowcache[threadIdx.x] = ((char*)output_data+(fanout_index[goffset]*pitch))[tid];
		switch (type) {
			case INPT:
				val = pi_data[gid+pi_pitch*(tid)];
				break;
			default: 
					// we're guaranteed at least one fanin per 
					// gate if not on an input.
					__syncthreads();
					if (type != NOT) {
						val = rowcache[threadIdx.x];
					} else {
						val = tex1D(notLUT, rowcache[threadIdx.x]);
					}

					j = 1;
					while (j < nfi) {
						__syncthreads();
						r = REF2D(char,output_data,pitch,FIN(fanout_index,goffset,j),tid); //((char*)output_data+(fanout_index[goffset+j]*pitch))[tid]; 
						switch(type) {
							case XOR:
								val = tex2D(xor2LUT, val, r);break;
							case XNOR:
								val = tex2D(xnor2LUT, val, r);break;
							case OR:
								val = tex2D(or2LUT, val, r);break;
							case NOR:
								val = tex2D(nor2LUT, val, r);break;
							case AND:
								val = tex2D(and2LUT, val, r);break;
							case NAND:
								val = tex2D(nand2LUT, val, r);break;
						}
						j++;
					}
		}
		row[tid] = val;
	}
}
__global__ void kernSimulateP2(GPUNODE* graph, int* pi_data, size_t pi_pitch, char* output_data, size_t pitch,size_t pattern_count,  int* fanout_index, int start_offset) {
	int tid = (blockIdx.y * blockDim.y) + threadIdx.x, prev=0;
	int gid = blockIdx.x+start_offset;
	__shared__ char rowcache[SIM_BLOCK];
	char *row, r;
	int goffset, nfi, val, j,type;

	if (tid < pattern_count) {
		row = ((char*)output_data + gid*pitch)+tid; // get the line row for the current gate
		goffset = graph[gid].offset;
		nfi = graph[gid].nfi;
		type = graph[gid].type;
		prev = *row;

		rowcache[threadIdx.x] = ((char*)output_data+(fanout_index[goffset]*pitch))[tid];
		switch (type) {
			case INPT:
				val = pi_data[gid+pi_pitch*(tid)];
				break;
			default: 
					// we're guaranteed at least one fanin per 
					// gate if not on an input.
					__syncthreads();
					if (type != NOT) {
						val = rowcache[threadIdx.x];
					} else {
						val = tex1D(notLUT, rowcache[threadIdx.x]);
					}

					j = 1;
					while (j < nfi) {
						__syncthreads();
						r = ((char*)output_data+(fanout_index[goffset+j]*pitch))[tid]; 
						switch(type) {
							case XOR:
								val = tex2D(xor2LUT, val, r );break;
							case XNOR:
								val = tex2D(xnor2LUT, val, r);break;
							case OR:
								val = tex2D(or2LUT, val, r);break;
							case NOR:
								val = tex2D(nor2LUT, val, r);break;
							case AND:
								val = tex2D(and2LUT, val, r);break;
							case NAND:
								val = tex2D(nand2LUT, val, r);break;
						}
						j++;
					}
		}
		if (type == FROM || type == BUFF)
			*row = val;
		else {
			*row = tex2D(stableLUT,prev,val);
		}
	}
}

void loadSimLUTs() {
	int nand2[16] = {1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0};
	int and2[16]  = {0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1};
	int nor2[16]  = {1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0};
	int or2[16]   = {0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1};
	int xnor2[16] = {1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1};
	int xor2[16]  = {0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0};
	int stable[4] = {S0, T0, T1, S1};
	int not_gate[4] = {1, 1, 0, 0};

	// device memory arrays, required. 
	hipArray *cuNandArray, *cuAndArray,*cuNorArray, *cuOrArray,*cuXnorArray,*cuXorArray, *cuNotArray,*cuStableArray;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();

	// Allocate space in device memory for the LUTs. 
	HANDLE_ERROR(hipMallocArray(&cuNandArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuAndArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuNorArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXnorArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorArray, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuStableArray, &channelDesc, 2,2));
	HANDLE_ERROR(hipMallocArray(&cuNotArray, &channelDesc, 4,1));

	// Copying the static arrays given to device memory.
	HANDLE_ERROR(hipMemcpyToArray(cuNandArray, 0,0, nand2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndArray, 0,0, and2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuNorArray, 0,0, nor2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrArray, 0,0, or2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXnorArray, 0,0, xnor2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorArray, 0,0, xor2, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuStableArray, 0,0, stable, sizeof(int)*4,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuNotArray, 0,0, not_gate, sizeof(int)*4,hipMemcpyHostToDevice));

	HANDLE_ERROR(hipBindTextureToArray(and2LUT,cuAndArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(nand2LUT,cuNandArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2LUT,cuOrArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(nor2LUT,cuNorArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2LUT,cuXorArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xnor2LUT,cuXnorArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(stableLUT,cuStableArray,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(notLUT,cuNotArray,channelDesc));
}

float gpuRunSimulation(ARRAY2D<char> results, ARRAY2D<int> inputs, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, int* fan, int maxlevels, int pass = 1) {
	loadSimLUTs(); // set up our lookup tables for simulation.
	int startGate = 0;
	int *gatesinLevel;
	gatesinLevel = new int[maxlevels];
	for (int i = 0; i < maxlevels; i++) {
		gatesinLevel[i] = 0;
		for (unsigned int j = 0; j < results.width; j++) {
			if (graph[j].level == i) {
				gatesinLevel[i]++;
			}
		}
	}
	int blockcount_y = (int)(results.height/SIM_BLOCK) + (results.height%SIM_BLOCK > 0);
#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	for (int i = 0; i < maxlevels; i++) {
		dim3 numBlocks(gatesinLevel[i],blockcount_y);
		if (pass > 1) {
			kernSimulateP2<<<numBlocks,SIM_BLOCK>>>(dgraph.data,inputs.data, inputs.width, results.data, results.pitch, inputs.height, fan, startGate);
		} else {
			kernSimulateP1<<<numBlocks,SIM_BLOCK>>>(dgraph.data,inputs.data, inputs.width, results.data, results.pitch, inputs.height, fan, startGate);
		}
		startGate += gatesinLevel[i];
		hipDeviceSynchronize();
	}
	free(gatesinLevel);
	// We're done simulating at this point.
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}

void debugSimulationOutput(ARRAY2D<char> results, int pass = 1) {
#ifndef NDEBUG
	char *lvalues, *row;
	DPRINT("Post-simulation device results, pass %d:\n\n", pass);
	DPRINT("Vector:   \t");
	for (unsigned int i = 0; i < results.height; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (unsigned int r = 0;r < results.width; r++) {
		lvalues = (char*)malloc(results.height*sizeof(char));
		row = ((char*)results.data + r*results.pitch); // get the current row?
		hipMemcpy(lvalues,row,results.height*sizeof(char),hipMemcpyDeviceToHost);
		DPRINT("%s %d:\t", pass > 1 ? "Line " : "Line ",r);
		for (unsigned int i = 0; i < results.height; i++) {
			switch(lvalues[i]) {
				case S0:
					DPRINT("S0 "); break;
				case S1:
					DPRINT("S1 "); break;
				case T0:
					DPRINT("T0 "); break;
				case T1:
					DPRINT("T1 "); break;
				default:
					DPRINT("%2d ", lvalues[i]); break;
			}
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif
}
