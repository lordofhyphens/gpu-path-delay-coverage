#include "hip/hip_runtime.h"
#include "mergekernel.h"
#include <hip/hip_runtime.h>
#include "util/gpuckt.h"
#include "util/gpudata.h"
#include "util/segment.cuh"
#include "util/utility.cuh"
#include "markkernel.h"

#include <mgpuhost.cuh>
#include <mgpudevice.cuh>
#include <device/ctascan.cuh>
#include <device/intrinsics.cuh>
#undef N
#undef MERGE_SIZE
void HandleMergeError( hipError_t err, const char *file, uint32_t line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
 }
#define HANDLE_ERROR( err ) (HandleMergeError( err, __FILE__, __LINE__ ))

template<int N>
void debugSegmentList(segment<N,int2>* seglist, const unsigned int& size, std::string outfile) {
	#ifndef NDEBUG
	segment<N,int2> *lvalues;
	std::ofstream ofile(outfile.c_str());
	lvalues = new segment<N,int2>[size];
	hipMemcpy(lvalues,seglist,size*sizeof(segment<N,int2>),hipMemcpyDeviceToHost);
	for (size_t r = 0;r < size; r++) {
		ofile << "Segment " << r << "(" ;
		segment<N,int2> z = lvalues[r];//REF2D(uint8_t, lvalues, results.pitch, r, i);
		#pragma unroll
		for (int j = 0; j < N; j++) {
			ofile << z.key.num[j];
			if (j != N-1) 
				ofile << ",";
		}
		ofile << "):\t";
		ofile << std::setw(OUTJUST) << z.pattern.x << "," << z.pattern.y << " ";
		ofile << std::endl;
		}
	delete lvalues;
	ofile.close();
#endif
}
namespace mgpu {
	struct ScanOpMinPos {
		enum { Communative = true};
		typedef int2 input_type;
		typedef uint2 value_type;
		typedef int2 result_type;

		MGPU_HOST_DEVICE value_type Extract (value_type t, int index) { return t;}
		MGPU_HOST_DEVICE value_type Plus(value_type t1, value_type t2) { return make_uint2(min(t1.x,t2.x),min(t1.y,t2.x));};
		MGPU_HOST_DEVICE value_type Combine(value_type t1, value_type t2) { return t2; }
		MGPU_HOST_DEVICE value_type Identity() { return make_uint2((unsigned)_ident.x,(unsigned)_ident.y); }
		MGPU_HOST_DEVICE ScanOpMinPos(input_type ident) : _ident(ident) { }
		MGPU_HOST_DEVICE ScanOpMinPos() {
			_ident = make_int2(numeric_limits<int>::max(), numeric_limits<int>::max());
		}
		input_type _ident;
	};
}

const unsigned int BLOCK_STEP = 65535; // # of SIDs to process at once.
const unsigned int PARALLEL_SEGS = 16;
const unsigned int WARP_SIZE = 32;
const unsigned int MERGE_SIZE = 1024;

inline __host__ __device__ int2 min(int2 a, int2 b) { 
	return make_int2(min((unsigned)a.x, (unsigned)b.x), min((unsigned)a.y, (unsigned)b.y));
}
inline __host__ __device__ bool operator==(const int2& a, const int2&b) {
	return (a.x == b.x) && (a.y == b.y);
}

template <unsigned int blockSize>
__device__ void warpReduceMin(volatile int2 sdata[], unsigned int tid) {
	if (blockSize >= 64) sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 32].x);
	if (blockSize >= 64) sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 32].y); 
	if (blockSize >= 32) sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 16].x);
	if (blockSize >= 32) sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 16].y);
	if (blockSize >= 16) sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 8].x);
	if (blockSize >= 16) sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 8].y);
	if (blockSize >= 8)  sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 4].x);
	if (blockSize >= 8)  sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 4].y);
	if (blockSize >= 4)  sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 2].x);
	if (blockSize >= 4)  sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 2].y);
	if (blockSize >= 2)  sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 1].x);
	if (blockSize >= 2)  sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 1].y);
}
template <unsigned int blockSize>
__device__ void warpReduceMin(volatile int2 sdata[blockSize][WARP_SIZE], const unsigned int& x, const unsigned int& y) {
	if (blockSize >= 64) sdata[y][x].x = min((unsigned)sdata[y][x].x, (unsigned)sdata[y][x + 32].x);
	if (blockSize >= 64) sdata[y][x].y = min((unsigned)sdata[y][x].y, (unsigned)sdata[y][x + 32].y); 
	if (blockSize >= 32) sdata[y][x].x = min((unsigned)sdata[y][x].x, (unsigned)sdata[y][x + 16].x);
	if (blockSize >= 32) sdata[y][x].y = min((unsigned)sdata[y][x].y, (unsigned)sdata[y][x + 16].y);
	if (blockSize >= 16) sdata[y][x].x = min((unsigned)sdata[y][x].x, (unsigned)sdata[y][x + 8].x);
	if (blockSize >= 16) sdata[y][x].y = min((unsigned)sdata[y][x].y, (unsigned)sdata[y][x + 8].y);
	if (blockSize >= 8)  sdata[y][x].x = min((unsigned)sdata[y][x].x, (unsigned)sdata[y][x + 4].x);
	if (blockSize >= 8)  sdata[y][x].y = min((unsigned)sdata[y][x].y, (unsigned)sdata[y][x + 4].y);
	if (blockSize >= 4)  sdata[y][x].x = min((unsigned)sdata[y][x].x, (unsigned)sdata[y][x + 2].x);
	if (blockSize >= 4)  sdata[y][x].y = min((unsigned)sdata[y][x].y, (unsigned)sdata[y][x + 2].y);
	if (blockSize >= 2)  sdata[y][x].x = min((unsigned)sdata[y][x].x, (unsigned)sdata[y][x + 1].x);
	if (blockSize >= 2)  sdata[y][x].y = min((unsigned)sdata[y][x].y, (unsigned)sdata[y][x + 1].y);
}
// Read the segment from the entry, determine the earliest pattern that marks it, and then write that (atomically).
// (likely) RESTRICTION: blockDim.x MUST be a power of 2
template <int N, unsigned int blockSize>
__global__ void kernSegmentReduce(segment<N, int2>* seglist, const GPU_DATA_type<coalesce_t> mark, const GPU_DATA_type<coalesce_t> sim, uint32_t startSegment, uint32_t startPattern) {
	__shared__ int2 midWarp[blockSize];
	midWarp[threadIdx.x] = make_int2(-1,-1);
	__syncthreads();

	uint32_t pid = threadIdx.x + blockIdx.x*blockDim.x;
	uint32_t real_pid = pid * 4 + startPattern; // unroll constant for coalesce_t
	pid += startPattern;
	uint32_t sid = blockIdx.y+startSegment;
	int2 simple = make_int2(-1,-1);
	if (real_pid < mark.width) {
		// In each thread, get 4 results-worth. 
		// This should unroll, as the trip count is known at compile time.
		// Get a batch of mark and sim results
		unsigned int mark_set = 0xffffffff;
		#pragma unroll
		for (uint8_t i = 0; i < N; i++) {
			// AND each mark result together.
			mark_set &= REF2D(mark, pid, seglist[sid].key.num[i]);
		}

		// check to see which position got marked. This will be one of 4 possible positions:
		// 1, 9, 17, 25 (as returned by ffs).
		// Post brev:
		// 32 = offset 0
		// 24 = offset 1
		// 16 = offset 2 // 8 = offset 3
		// We OR the original set with itself, shifted and then mask off the garbage.
		mark_set =  (mark_set | (mark_set >> 7) | (mark_set >> 14) | (mark_set >> 20)) & 0x0000000F;
		// reversing the bits puts the lowest bit first, 
		mark_set = __brev(mark_set);
		//which lets us get its position with ffs.
		unsigned int offset = (32 - __ffs(mark_set));
		offset *= (offset < 5);

//		int sim_row = (uint32_t*)((char*)sim.data + sim.pitch*3)[2].r;

		// Figure out the relevant entry on the simualtion table.
		unsigned int sim_type = REF2D(sim, pid, seglist[sid].key.num[0]).rows[offset];
	//	printf("%d,%d: %8X\n",seglist[sid].key.num[0],  offset, sim_type);

		// If the simulation results is T0 (or 2), then the real PID needs to be put into shared mem, x location
		midWarp[threadIdx.x].x = (offset+real_pid+1)*((mark_set > 0)&&(sim_type == T0)) - 1;
		
		// If the simulation results is T1 (or 3), then the real PID needs to be put into shared mem, x location
		midWarp[threadIdx.x].y = (offset+real_pid+1)*((mark_set > 0)&&(sim_type == T1)) - 1;
		
		// actual PID + 1 we are comparing against or 0 if not found.
		// Place in shared memory, decrementing to correct real PID.


		__syncthreads();
		// Now do the reduction inside the same warp, looking for min-positive.
		if (blockSize >= 512) { if (threadIdx.x < 256) { midWarp[threadIdx.x] = min(midWarp[threadIdx.x], midWarp[threadIdx.x+256]); } __syncthreads();} 
		if (blockSize >= 256) { if (threadIdx.x < 128) { midWarp[threadIdx.x] =min(midWarp[threadIdx.x], midWarp[threadIdx.x+128]); } __syncthreads();} 
		if (blockSize >= 128) { if (threadIdx.x < 64) { midWarp[threadIdx.x] =min(midWarp[threadIdx.x], midWarp[threadIdx.x+64]); } __syncthreads();} 
		if (threadIdx.x < 32) warpReduceMin<blockSize>(midWarp,threadIdx.x);
			__syncthreads();
		if (threadIdx.x == 0) {
			// brief loop until we know that the item we wrote with AtomicMin is the correct positive minimum
			int2 candidate = midWarp[0];
			int2 evict = make_int2(-1,-1);

			do { evict.x = atomicMin(&(seglist[sid].pattern.x), (unsigned)(candidate.x));
			} while (min((unsigned)evict.x,(unsigned)candidate.x) == evict.x && evict.x != candidate.x);
			do { evict.y = atomicMin(&(seglist[sid].pattern.y), (unsigned)(candidate.y));
			} while (min((unsigned)evict.y,(unsigned)candidate.y) == evict.y && evict.y != candidate.y);

//			printf("Lowest for block %d: (%d, %d)\n", sid, midWarp[0].x, midWarp[0].y);
		}
	}
	
}
/* Reduction strategy - X/1024 pattern blocks, Y blocks of lines/gates. Each
 * block gets the minimum ID within the block and places it into a temporary
 * location [BLOCK_X,BLOCK_Y] 
*/

template <int N, unsigned int blockSize>
__global__ void kernSegmentReduce2(segment<N, int2>* seglist, int maxSegment, const GPU_DATA_type<coalesce_t> mark, const GPU_DATA_type<coalesce_t> sim, uint32_t startSegment, uint32_t startPattern) {
	__shared__ int2 midWarp[blockSize][32];
	int2 final_value = make_int2(-1,-1);
	// Current segment for this thread group.
	midWarp[threadIdx.y][threadIdx.x] = make_int2(-1,-1);
	__syncthreads();
	unsigned int sid = startSegment + threadIdx.y + blockIdx.y*blockDim.y;
#pragma unroll 2
	for (int ref_pid = 0; ref_pid <= MERGE_SIZE; ref_pid += 32) {
		uint32_t pid = threadIdx.x + ref_pid;
		uint32_t real_pid = pid * 4 + startPattern; // unroll constant for coalesce_t
		
		if (real_pid < mark.width && sid < maxSegment) {
			unsigned int mark_set = 0xffffffff;
#pragma unroll
			for (uint8_t i = 0; i < N; i++) {
				// AND each mark result together.
				mark_set &= REF2D(mark, pid, seglist[sid].key.num[i]);
			}

			// check to see which position got marked. This will be one of 4 possible positions:
			// 1, 9, 17, 25 (as returned by ffs).
			// Post brev:
			// 32 = offset 0
			// 24 = offset 1
			// 16 = offset 2 // 8 = offset 3
			// We OR the original set with itself, shifted and then mask off the garbage.
			mark_set =  (mark_set | (mark_set >> 7) | (mark_set >> 14) | (mark_set >> 20)) & 0x0000000F;
			// reversing the bits puts the lowest bit first, 
			mark_set = __brev(mark_set);
			//which lets us get its position with ffs.
			unsigned int offset = (32 - __ffs(mark_set));
			offset *= (offset < 5);

			// Figure out the relevant entry on the simualtion table.
			unsigned int sim_type = REF2D(sim, pid, seglist[sid].key.num[0]).rows[offset];

			// If the simulation results is T0 (or 2), then the real PID needs to be put into shared mem, x location
			midWarp[threadIdx.y][threadIdx.x].x = (offset+real_pid+1)*((mark_set > 0)&&(sim_type == T0)) - 1;

			// If the simulation results is T1 (or 3), then the real PID needs to be put into shared mem, x location
			midWarp[threadIdx.y][threadIdx.x].y = (offset+real_pid+1)*((mark_set > 0)&&(sim_type == T1)) - 1;

			// actual PID + 1 we are comparing against or 0 if not found.
			// Place in shared memory, decrementing to correct real PID.
			if (threadIdx.x < blockSize / 2) 
				warpReduceMin<blockSize>(midWarp, threadIdx.x, threadIdx.y);

			__syncthreads();
			if (threadIdx.x == 0) { final_value = min(midWarp[threadIdx.y][0],final_value); }
		}
	}
	__syncthreads();
	if (threadIdx.x == 0 && sid < maxSegment) {
		int2 candidate = make_int2(-1,-1);
		if (final_value.x >= 0) {
			while ((unsigned)final_value.x < (unsigned)candidate.x) {
				candidate.x = atomicMin((unsigned*)&(seglist[sid].pattern.x), (unsigned)(final_value.x));
			}
		}
		if (final_value.y >= 0) { 
			while ((unsigned)final_value.y < (unsigned)candidate.y) {
				candidate.y = atomicMin((unsigned*)&(seglist[sid].pattern.y), (unsigned)(final_value.y));
			}
		}
	}
}

float gpuMergeSegments(GPU_Data& mark, GPU_Data& sim, GPU_Circuit& ckt, size_t chunk, uint32_t ext_startPattern, void** seglist, int& numseg) {

	// if g_hashlist == NULL, copy the hash list to the GPU
//	ckt.print();
	segment<2, int2>* dc_seglist = (segment<2, int2>*)*seglist;
	uint32_t startPattern = ext_startPattern;
	uint32_t segcount = 0;
	if (dc_seglist == NULL) { 
		segment<2, int2>* h_seglist = NULL;
		generateSegmentList(&h_seglist,ckt);
		
		//displaySegmentList(h_seglist, ckt);
		while (h_seglist[segcount].key.num[0] < ckt.size()) { h_seglist[segcount].pattern.x = -1; h_seglist[segcount].pattern.y = -1; segcount++;}
		hipMalloc(&dc_seglist, sizeof(segment<2,int2>)*segcount);
		hipMemcpy(dc_seglist, h_seglist, sizeof(segment<2,int2>)*segcount, hipMemcpyHostToDevice);
		std::cerr << "Allocating hashmap space of " << segcount << ".\n";
		HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
		numseg = segcount;
	} else { segcount = numseg; }
	#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
	// assume that the 0th entry is the widest, which is true given the chunking method.
#endif // NTIMING
	uint32_t count = 0;
	size_t remaining_blocks = (segcount / PARALLEL_SEGS) + ((segcount % PARALLEL_SEGS) > 0);
	const size_t block_x = (mark.gpu(chunk).width / MERGE_SIZE) + ((mark.gpu(chunk).width % MERGE_SIZE) > 0);
	size_t block_y = (remaining_blocks > BLOCK_STEP ? BLOCK_STEP : remaining_blocks);
	std::cerr << "Working with " << block_y * PARALLEL_SEGS<< " / " << remaining_blocks * PARALLEL_SEGS << " sids.\n"; 

	GPU_DATA_type<coalesce_t> marks = toPod<coalesce_t>(mark,chunk);
	GPU_DATA_type<coalesce_t> sims = toPod<coalesce_t>(sim,chunk);
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting on memory allocation
	do {
		dim3 threads(WARP_SIZE, PARALLEL_SEGS);
		dim3 blocks(block_x, block_y);
		kernSegmentReduce2<2,PARALLEL_SEGS><<<blocks, threads>>>(dc_seglist, segcount, marks, sims,count, startPattern);

		//kernSegmentReduce<2,MERGE_SIZE><<<blocks, MERGE_SIZE>>>(dc_seglist, marks, sims,count, startPattern);
		count+=BLOCK_STEP;
		HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
		dim3 blocksmin(1, block_y);
		if (remaining_blocks < BLOCK_STEP) { remaining_blocks = 0;}
		if (remaining_blocks >= BLOCK_STEP) { remaining_blocks -= BLOCK_STEP; }
		block_y = (remaining_blocks > BLOCK_STEP ? BLOCK_STEP : remaining_blocks);
		hipDeviceSynchronize();
	} while (remaining_blocks > 0);
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
#ifdef LOGEXEC
	std::cerr << "Printing merge log." << std::endl;
	debugSegmentList(dc_seglist, numseg, "gpumerge.log");
#endif //LOGEXEC
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING

}
// scan through input until the first 1 is found, save the identifier and set all indicies above that.
void debugMergeOutput(size_t size, const void* res, std::string outfile) {
#ifndef NDEBUG
	int2 *lvalues, *results = (int2*)res;
	std::ofstream ofile(outfile.c_str());
	lvalues = new int2[size];
	hipMemcpy(lvalues,results,size*sizeof(int2),hipMemcpyDeviceToHost);
	for (size_t r = 0;r < size; r++) {
		ofile << "Gate " << r << ":\t";
		int2 z = lvalues[r];//REF2D(uint8_t, lvalues, results.pitch, r, i);
		ofile << std::setw(OUTJUST) << z.x << "," << z.y << " ";
		ofile << std::endl;
		}
	delete lvalues;
	ofile.close();
#endif
}
