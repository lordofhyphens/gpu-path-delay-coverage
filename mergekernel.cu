#include "hip/hip_runtime.h"
#include "mergekernel.h"
#include <hip/hip_runtime.h>
#include "util/gpuckt.h"
#include "util/gpudata.h"
#include "util/segment.cuh"
#include "util/utility.cuh"
#include "markkernel.h"

#include <mgpuhost.cuh>
#include <mgpudevice.cuh>
#include <device/ctascan.cuh>
#undef N
#undef MERGE_SIZE
#define MERGE_SIZE 512
void HandleMergeError( hipError_t err, const char *file, uint32_t line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
 }
#define HANDLE_ERROR( err ) (HandleMergeError( err, __FILE__, __LINE__ ))

template<int N>
void debugSegmentList(segment<N,int2>* seglist, const unsigned int& size, std::string outfile) {
	#ifndef NDEBUG
	segment<N,int2> *lvalues;
	std::ofstream ofile(outfile.c_str());
	lvalues = new segment<N,int2>[size];
	hipMemcpy(lvalues,seglist,size*sizeof(segment<N,int2>),hipMemcpyDeviceToHost);
	for (size_t r = 0;r < size; r++) {
		ofile << "Segment " << r << "(" ;
		segment<N,int2> z = lvalues[r];//REF2D(uint8_t, lvalues, results.pitch, r, i);
		#pragma unroll
		for (int j = 0; j < N; j++) {
			ofile << z.key.num[j];
			if (j != N-1) 
				ofile << ",";
		}
		ofile << "):\t";
		ofile << std::setw(OUTJUST) << z.pattern.x << "," << z.pattern.y << " ";
		ofile << std::endl;
		}
	delete lvalues;
	ofile.close();
#endif
}
namespace mgpu {
	struct ScanOpMinPos {
		enum { Communative = true};
		typedef int2 input_type;
		typedef uint2 value_type;
		typedef int2 result_type;

		MGPU_HOST_DEVICE value_type Extract (value_type t, int index) { return t;}
		MGPU_HOST_DEVICE value_type Plus(value_type t1, value_type t2) { return make_uint2(min(t1.x,t2.x),min(t1.y,t2.x));};
		MGPU_HOST_DEVICE value_type Combine(value_type t1, value_type t2) { return t2; }
		MGPU_HOST_DEVICE value_type Identity() { return make_uint2((unsigned)_ident.x,(unsigned)_ident.y); }
		MGPU_HOST_DEVICE ScanOpMinPos(input_type ident) : _ident(ident) { }
		MGPU_HOST_DEVICE ScanOpMinPos() {
			_ident = make_int2(numeric_limits<int>::max(), numeric_limits<int>::max());
		}
		input_type _ident;
	};
}

const unsigned int BLOCK_STEP = 1; // # of SIDs to process at once.

__host__ __device__ inline int32_t min_pos(int32_t a, int32_t b) { return min((unsigned)a, (unsigned)b);}
__host__ __device__ inline int2 min_pos(int2 a, int2 b) { return make_int2(min((unsigned)a.x, (unsigned)b.x),min((unsigned)a.y,(unsigned)b.y));}

template <unsigned int blockSize>
__device__ void warpReduceMin(volatile int2 * sdata , unsigned int tid) {
	if (blockSize >= 64) min_pos(sdata[tid], sdata[tid + 32]) ; 
	if (blockSize >= 32) min_pos(sdata[tid], sdata[tid + 16]) ;
	if (blockSize >= 16) min_pos(sdata[tid],sdata[tid + 8]);
	if (blockSize >= 8) min_pos(sdata[tid], sdata[tid + 4]);
	if (blockSize >= 4) min_pos(sdata[tid], sdata[tid + 2]);
	if (blockSize >= 2) min_pos(sdata[tid], sdata[tid + 1]);
}

__host__ __device__ inline uint32_t pred_gate(uint32_t a, bool b) { return 0xFFFFFFFFU >> (32-b)&a; }
// Read the segment from the entry, determine the earliest pattern that marks it, and then write that (atomically).
// (likely) RESTRICTION: blockDim.x MUST be a power of 2
template <int N, unsigned int blockSize>
__global__ void kernSegmentReduce(segment<N, int2>* seglist, const GPU_DATA_type<coalesce_t> mark, const GPU_DATA_type<coalesce_t> sim, uint32_t startSegment, uint32_t startPattern) {
	__shared__ int2 midWarp[blockSize];

	uint32_t pid = threadIdx.x + blockIdx.x*blockDim.x;
	uint32_t real_pid = pid * 4 + startPattern; // unroll constant for coalesce_t
	pid += startPattern;
	uint32_t sid = blockIdx.y+startSegment;
	int2 simple = make_int2(-1,-1);
	if (real_pid < mark.width) {
		// In each thread, get 4 results-worth. 
		// This should unroll, as the trip count is known at compile time.
		// Get a batch of mark and sim results
		unsigned int mark_set = 0xffffffff;
		#pragma unroll
		for (uint8_t i = 0; i < N; i++) {
			// AND each mark result together.
			mark_set &= REF2D(mark, pid, seglist[sid].key.num[i]);
		}

		// check to see which position got marked. This will be one of 4 possible positions:
		// 1, 9, 17, 25 (as returned by ffs).
		// Post brev:
		// 32 = offset 0
		// 24 = offset 1
		// 16 = offset 2 // 8 = offset 3
		mark_set =  (mark_set | (mark_set >> 7) | (mark_set >> 14) | (mark_set >> 20)) & 0x0000000F;
		mark_set = __brev(mark_set);
		unsigned int offset = (32 - __ffs(mark_set));
		unsigned int sim_type = (REF2D(sim, pred_gate(offset,offset>5)+real_pid, seglist[sid].key.num[0]) & (0x03 << offset*8)) >> (offset * 8);
		midWarp[threadIdx.x].x = pred_gate((offset+real_pid)+1, sim_type == T0) - 1;
		midWarp[threadIdx.x].y = pred_gate((offset+real_pid)+1, sim_type == T1) - 1;
		printf("Single-thread results: (%d, %d) %8x, = %d (%d,%d) %d\n",sid,pid,mark_set,offset,midWarp[threadIdx.x].x,midWarp[threadIdx.x].y,seglist[sid].key.num[0]);
		// actual PID + 1 we are comparing against or 0 if not found.
		// Place in shared memory, decrementing to correct real PID.

		// Now do the reduction inside the same warp, looking for min-positive.
		if (blockSize >= 512) { if (threadIdx.x < 256) { min_pos(midWarp[threadIdx.x], midWarp[threadIdx.x+256]); } __syncthreads();} 
		if (blockSize >= 256) { if (threadIdx.x < 128) { min_pos(midWarp[threadIdx.x], midWarp[threadIdx.x+128]); } __syncthreads();} 
		if (blockSize >= 128) { if (threadIdx.x < 64) { min_pos(midWarp[threadIdx.x], midWarp[threadIdx.x+64]); } __syncthreads();} 
		if (threadIdx.x < 32) warpReduceMin<blockSize>(midWarp,threadIdx.x);
		if (threadIdx.x == 0) {
			// brief loop until we know that the item we wrote with AtomicMin is the correct positive minimum
			int2 candidate = midWarp[0];
			int2 evict = make_int2(-1,-1);

			evict.x = atomicMin(&(seglist[sid].pattern.x), (unsigned)(candidate.x));
			evict.y = atomicMin(&(seglist[sid].pattern.y), (unsigned)(candidate.y));

			printf("Lowest for block %d: (%d, %d)\n", sid, midWarp[0].x, midWarp[0].y);
		}
	}
	
}
/* Reduction strategy - X/1024 pattern blocks, Y blocks of lines/gates. Each
 * block gets the minimum ID within the block and places it into a temporary
 * location [BLOCK_X,BLOCK_Y] 
*/

float gpuMergeSegments(GPU_Data& mark, GPU_Data& sim, GPU_Circuit& ckt, size_t chunk, uint32_t ext_startPattern, void** seglist, int& numseg) {
#ifndef NTIMING
	float elapsed;
	segment<2, int2>* dc_seglist = (segment<2, int2>*)*seglist;
	uint32_t startPattern = ext_startPattern;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
	// assume that the 0th entry is the widest, which is true given the chunking method.
#endif // NTIMING
	// if g_hashlist == NULL, copy the hash list to the GPU
//	ckt.print();
	uint32_t segcount = 0;
	if (dc_seglist == NULL) { 
		segment<2, int2>* h_seglist = NULL;
		generateSegmentList(&h_seglist,ckt);
		displaySegmentList(h_seglist, ckt);
		while (h_seglist[segcount].key.num[0] < ckt.size()) { segcount++;}
		hipMalloc(&dc_seglist, sizeof(segment<2,int2>)*segcount);
		hipMemcpy(dc_seglist, h_seglist, sizeof(segment<2,int2>)*segcount, hipMemcpyHostToDevice);
		std::cerr << "Allocating hashmap space of " << segcount << ".\n";
		HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
		numseg = segcount;
	} else { segcount = numseg; }
	uint32_t count = 0;
	size_t remaining_blocks = segcount;
	const size_t block_x = (mark.gpu(chunk).width / MERGE_SIZE) + ((mark.gpu(chunk).width % MERGE_SIZE) > 0);
	size_t block_y = (remaining_blocks > BLOCK_STEP ? BLOCK_STEP : remaining_blocks);
	std::cerr << "Working with " << block_y << " / " << remaining_blocks << " sids.\n"; 

	GPU_DATA_type<coalesce_t> marks = toPod<coalesce_t>(mark,chunk);
	std::cerr << "Podded pitch: " << marks.pitch << "\n";
	debugMarkOutput(&marks, ckt, chunk, ext_startPattern, "gpumark-test.log");
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting on memory allocation
	do {
		dim3 blocks(block_x, block_y);
		kernSegmentReduce<2,MERGE_SIZE><<<blocks, MERGE_SIZE>>>(dc_seglist, marks, toPod<coalesce_t>(sim,chunk),count, startPattern);
		HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
		dim3 blocksmin(1, block_y);
		count+=BLOCK_STEP;
		if (remaining_blocks < BLOCK_STEP) { remaining_blocks = 0;}
		if (remaining_blocks >= BLOCK_STEP) { remaining_blocks -= BLOCK_STEP; }
		block_y = (remaining_blocks > BLOCK_STEP ? BLOCK_STEP : remaining_blocks);
		hipDeviceSynchronize();
	} while (remaining_blocks > 0);
	hipDeviceSynchronize();
#ifdef LOGEXEC
	debugSegmentList(dc_seglist, numseg, "gpumerge.log");
#endif //LOGEXEC
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING

}
// scan through input until the first 1 is found, save the identifier and set all indicies above that.
void debugMergeOutput(size_t size, const void* res, std::string outfile) {
#ifndef NDEBUG
	int2 *lvalues, *results = (int2*)res;
	std::ofstream ofile(outfile.c_str());
	lvalues = new int2[size];
	hipMemcpy(lvalues,results,size*sizeof(int2),hipMemcpyDeviceToHost);
	for (size_t r = 0;r < size; r++) {
		ofile << "Gate " << r << ":\t";
		int2 z = lvalues[r];//REF2D(uint8_t, lvalues, results.pitch, r, i);
		ofile << std::setw(OUTJUST) << z.x << "," << z.y << " ";
		ofile << std::endl;
		}
	delete lvalues;
	ofile.close();
#endif
}
