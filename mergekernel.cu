#include "hip/hip_runtime.h"
#include "mergekernel.h"
#include <hip/hip_runtime.h>

void HandleMergeError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleMergeError( err, __FILE__, __LINE__ ))

#define MIN(A,B,AR) ( \
		(AR[A] > 0)*(AR[A] < AR[B])*AR[A] + \
		(AR[B] > 0)*(AR[B] < AR[A])*AR[B] + \
		AR[A]*(AR[B]==0) + \
		AR[B]*(AR[A]==0) )

__global__ void kernReduce(char* input, size_t height, size_t pitch, int goffset,int* meta, int mpitch) {
	int tid = threadIdx.x;
	__shared__ int sdata[MERGE_SIZE];
	char* row = input + pitch*(blockIdx.y+goffset);
	int* m = (int*)((char*)meta + mpitch*(blockIdx.y+goffset));
	unsigned int i = blockIdx.x*(MERGE_SIZE*2) + threadIdx.x;
	sdata[tid] = 0;
	// need to put the lower of i and i+MERGE_SIZE for which g_idata[i] == 1
	// Minimum ID given by this is 1.
	if (i < height) {
		if (i+MERGE_SIZE > height) { // correcting for blocks smaller than MERGE_SIZE
			sdata[tid] = (row[i] == 1)*(i+1);
		} else {
			sdata[tid] = (row[i] == 1)*(i+1) + (row[i+MERGE_SIZE] == 1)*(row[i] == 0)*(i+MERGE_SIZE+1);
		}
	}
	__syncthreads();

	// this is loop unrolling
    // do reduction in shared mem, comparisons against MERGE_SIZE are done at compile time.
    if (MERGE_SIZE >= 1024) { if (tid < 512 && tid+512 < height) { sdata[tid] = MIN(tid, tid+512,sdata); } __syncthreads(); }
    if (MERGE_SIZE >= 512) { if (tid < 256 && tid+256 < height) { sdata[tid] = MIN(tid, tid+256,sdata); } __syncthreads(); }
    if (MERGE_SIZE >= 256) { if (tid < 128 && tid+128 < height) { sdata[tid] = MIN(tid, tid+128,sdata); } __syncthreads(); }
    if (MERGE_SIZE >= 128) { if (tid <  64 && tid+64 < height) { sdata[tid] = MIN(tid, tid+64,sdata); } __syncthreads(); }
	if (tid < 32) {
		// Within a warp,  don't need __syncthreads();
		if (MERGE_SIZE >=  64) { if (tid+64 < height) { sdata[tid] = MIN(tid, tid + 32,sdata); } }
		if (MERGE_SIZE >=  32) { if (tid+32 < height) { sdata[tid] = MIN(tid, tid + 16,sdata); } }
		if (MERGE_SIZE >=  16) { if (tid+16 < height) { sdata[tid] = MIN(tid, tid +  8,sdata); } }
		if (MERGE_SIZE >=   8) { if (tid+8 < height) { sdata[tid] = MIN(tid, tid +  4,sdata); } }
		if (MERGE_SIZE >=   4) { if (tid+4 < height) { sdata[tid] = MIN(tid, tid +  2,sdata); } }
		if (MERGE_SIZE >=   2) { if (tid+2 < height) { sdata[tid] = MIN(tid, tid +  1,sdata); } }
	}
	
	// at this point, we have the position of the lowest. Correct by 1 to compensate for above.

//	if (tid ==0 ) { printf("Final Tid: %d, line %d, data+1 %d \n", tid, blockIdx.y, sdata[tid] - 1); }
	if (threadIdx.x == 0) { m[blockIdx.x] = sdata[0]-1; }
	sdata[tid] = (sdata[0]-1)*(sdata[0]>0) + (sdata[0] == 0)*MERGE_SIZE*2;
	__syncthreads();
}

__global__ void kernSetMin(int* g_odata, size_t pitch,int* intermediate, int length, int i_pitch, int goffset) {
	unsigned int tid = (blockIdx.x*blockDim.x)+threadIdx.x;
	unsigned int gid = blockIdx.y;
	int* blockset = (int*)((char*)intermediate + (i_pitch*gid));
	if (tid == 0) { //first thread 
		// scan sequentially until a thread ID is discovered;
		int i = 0;
		while (blockset[i] < 0 && i < length) {
			i++;
		}
		if (i == length) {
			g_odata[gid] = -1;
		} else {
			g_odata[gid] = blockset[i];
		}
	}
	__syncthreads();
}
// scan through input until the first 1 is found, save the identifier and memset all indicies above that.
float gpuMergeHistory(GPU_Data& input, ARRAY2D<int> mergeids) {
	size_t block_x = (input.width() / MERGE_SIZE) + (input.width() % MERGE_SIZE) > 1;
	size_t block_y = input.height();
	int* temparray;
	size_t pitch;
	hipMallocPitch(&temparray, &pitch, sizeof(int)*block_x, block_y);
	dim3 blocks(block_x, block_y);
#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	DPRINT("Blocks: (%lu, %lu), %d\n", block_x, block_y, MERGE_SIZE);
	kernReduce<<<blocks, MERGE_SIZE>>>(input.gpu().data, input.height(), input.gpu().pitch, 0, temparray, pitch);
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
	dim3 blocksmin(1, block_y);
	kernSetMin<<<blocksmin, MERGE_SIZE>>>(mergeids.data, mergeids.pitch, temparray, block_x, pitch, 0);
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}
