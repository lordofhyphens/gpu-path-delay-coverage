#include "hip/hip_runtime.h"
#include "mergekernel.h"
#include <hip/hip_runtime.h>

void HandleMergeError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleMergeError( err, __FILE__, __LINE__ ))

#define MIN(A,B,AR) ( \
		(AR[A] > 0)*(AR[A] < AR[B])*AR[A] + \
		(AR[B] > 0)*(AR[B] < AR[A])*AR[B] + \
		AR[A]*(AR[B]==0) + \
		AR[B]*(AR[A]==0) )

__global__ void kernReduce(char* input, size_t height, size_t pitch, int goffset,int* meta, int mpitch, int startGate) {
	int tid = threadIdx.x;
	int gid = blockIdx.y+startGate;
	__shared__ int sdata[MERGE_SIZE];
	char* row = input + pitch*(gid+goffset);
	unsigned int i = blockIdx.x*(MERGE_SIZE*2) + threadIdx.x;
	sdata[tid] = 0;
	// need to put the lower of i and i+MERGE_SIZE for which g_idata[i] == 1
	// Minimum ID given by this is 1.
	if (i < height) {
		if (i+MERGE_SIZE > height) { // correcting for blocks smaller than MERGE_SIZE
			sdata[tid] = (row[i] == 1)*(i+1);
			//printf("%s:%d - input[%d][%d] = %d\n", __FILE__,__LINE__,i, gid, row[i]);
		} else {
			sdata[tid] = (row[i] == 1)*(i+1) + (row[i+MERGE_SIZE] == 1)*(row[i] == 0)*(i+MERGE_SIZE+1);
		}
		__syncthreads();

		// this is loop unrolling
		// do reduction in shared mem, comparisons against MERGE_SIZE are done at compile time.
		if (MERGE_SIZE >= 1024) { if (tid < 512 && tid+512 < height) { sdata[tid] = MIN(tid, tid+512,sdata); } __syncthreads(); }
		if (MERGE_SIZE >= 512) { if (tid < 256 && tid+256 < height) { sdata[tid] = MIN(tid, tid+256,sdata); } __syncthreads(); }
		if (MERGE_SIZE >= 256) { if (tid < 128 && tid+128 < height) { sdata[tid] = MIN(tid, tid+128,sdata); } __syncthreads(); }
		if (MERGE_SIZE >= 128) { if (tid <  64 && tid+64 < height) { sdata[tid] = MIN(tid, tid+64,sdata); } __syncthreads(); }
		if (tid < 32) {
			// Within a warp,  don't need __syncthreads();
			if (MERGE_SIZE >=  64) { if (tid+32 < height) { sdata[tid] = MIN(tid, tid + 32,sdata); } }
			if (MERGE_SIZE >=  32) { if (tid+16 < height) { sdata[tid] = MIN(tid, tid + 16,sdata); } }
			if (MERGE_SIZE >=  16) { if (tid+8 < height) { sdata[tid] = MIN(tid, tid +  8,sdata); } }
			if (MERGE_SIZE >=   8) { if (tid+4 < height) { sdata[tid] = MIN(tid, tid +  4,sdata); } }
			if (MERGE_SIZE >=   4) { if (tid+2 < height) { sdata[tid] = MIN(tid, tid +  2,sdata); } }
			if (MERGE_SIZE >=   2) { if (tid+1 < height) { sdata[tid] = MIN(tid, tid +  1,sdata); } }
		}

		// at this point, we have the position of the lowest. Correct by 1 to compensate for above.

//		if (tid ==0 ) { printf("Final Tid: %d, line %d, data+1 %d \n", tid, blockIdx.y, sdata[tid] - 1); }
//		sdata[tid] = (sdata[0]-1)*(sdata[0]>0) + (sdata[0] == 0)*MERGE_SIZE*2;
		if (threadIdx.x == 0) { REF2D(int,meta,mpitch,blockIdx.x,gid) = sdata[0]-1; }
		__syncthreads();

	}
}

__global__ void kernSetMin(int* g_odata, size_t pitch,int* intermediate, int i_pitch,int length, int startGate) {
	unsigned int gid = blockIdx.y + startGate;
	// scan sequentially until a thread ID is discovered;
	int i = 0;
//	printf("%s:%d - int[%d][%d] = %d\n", __FILE__, __LINE__, i, gid, REF2D(int, intermediate, i_pitch, i, gid) );
	while (REF2D(int, intermediate, i_pitch, i, gid) < 0 && i < length) {
//		printf("%s:%d - int[%d][%d] = %d\n", __FILE__, __LINE__, i, gid, REF2D(int, intermediate, i_pitch, i, gid) );
		i++;
	}
	if (i == length) {
		g_odata[gid] = -1;
	} else {
		g_odata[gid] = REF2D(int, intermediate, i_pitch, i, gid);
//		printf("%s:%d - g_odata[%d] = int[%d][%d] = %d\n", __FILE__, __LINE__, gid, i, gid, REF2D(int, intermediate, i_pitch, i, gid) );
	}
}
// scan through input until the first 1 is found, save the identifier and memset all indicies above that.
float gpuMergeHistory(GPU_Data& input, ARRAY2D<int> mergeids) {
	size_t block_x = (input.width() / MERGE_SIZE) + ((input.width() % MERGE_SIZE) > 0);
	size_t remaining_blocks = input.height();
	int count = 0;
	int* temparray;
	size_t pitch;
	hipMallocPitch(&temparray, &pitch, sizeof(int)*block_x, remaining_blocks);
	hipMalloc(&mergeids.data, sizeof(int)*input.height());
//	int* debug = (int*)malloc(sizeof(int)*input.height());
	int* debugt = (int*)malloc(sizeof(int)*input.height()*block_x);
	memset(debugt, 0, input.height()*block_x);
#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	for (unsigned int chunk = 0; chunk < input.size(); chunk++) {
		size_t block_y = (remaining_blocks > 65535 ? 65535 : remaining_blocks);
		do {
			DPRINT("%s:%d - Blocks: %lu/%lu (%lu, %lu), %d\n", __FILE__, __LINE__, input.gpu(chunk).width, input.width(), block_x, block_y, MERGE_SIZE);
			dim3 blocks(block_x, block_y);
			kernReduce<<<blocks, MERGE_SIZE>>>(input.gpu(chunk).data, input.gpu(chunk).width, input.gpu(chunk).pitch, 0, temparray, pitch, count);
			hipDeviceSynchronize();
/*			hipMemcpy2D(debugt, sizeof(int)*block_x, temparray, pitch, sizeof(int)*block_x, input.height(), hipMemcpyDeviceToHost);
			for (unsigned int j = 0; j < block_x/2; j++) {
				for (unsigned int i = 0; i < input.height(); i++) {
					DPRINT("%4d ", REF2D(int, debugt, sizeof(int)*block_x,j,i));
				}
				DPRINT("\n");
			}
*/
			HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
			dim3 blocksmin(1, block_y);
			kernSetMin<<<blocksmin, 1>>>(mergeids.data, mergeids.pitch, temparray,  pitch, block_x/2, count);
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
			count+=65535;
			if (remaining_blocks > 65535) { remaining_blocks -= 65535; }
			block_y = (remaining_blocks > 65535 ? 65535 : remaining_blocks);
		} while (remaining_blocks > 65535);
/*		hipMemcpy(debug, mergeids.data, sizeof(int)*input.height(),  hipMemcpyDeviceToHost);
		for (unsigned int i = 0; i < input.height(); i++) {
			DPRINT("%2d ", debug[i]);
		}
		DPRINT("\n");
 */
	}
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}
