#include "hip/hip_runtime.h"
#include "mergekernel.h"
#include <hip/hip_runtime.h>
#include "util/gpuckt.h"
#include "util/gpudata.h"
#include "util/segment.cuh"
#include "util/utility.cuh"
#include "markkernel.h"

#include <mgpuhost.cuh>
#include <mgpudevice.cuh>
#include <device/ctascan.cuh>
#include <device/intrinsics.cuh>
#undef N
#undef MERGE_SIZE
#define MERGE_SIZE 512
void HandleMergeError( hipError_t err, const char *file, uint32_t line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
 }
#define HANDLE_ERROR( err ) (HandleMergeError( err, __FILE__, __LINE__ ))

template<int N>
void debugSegmentList(segment<N,int2>* seglist, const unsigned int& size, std::string outfile) {
	#ifndef NDEBUG
	segment<N,int2> *lvalues;
	std::ofstream ofile(outfile.c_str());
	lvalues = new segment<N,int2>[size];
	hipMemcpy(lvalues,seglist,size*sizeof(segment<N,int2>),hipMemcpyDeviceToHost);
	for (size_t r = 0;r < size; r++) {
		ofile << "Segment " << r << "(" ;
		segment<N,int2> z = lvalues[r];//REF2D(uint8_t, lvalues, results.pitch, r, i);
		#pragma unroll
		for (int j = 0; j < N; j++) {
			ofile << z.key.num[j];
			if (j != N-1) 
				ofile << ",";
		}
		ofile << "):\t";
		ofile << std::setw(OUTJUST) << z.pattern.x << "," << z.pattern.y << " ";
		ofile << std::endl;
		}
	delete lvalues;
	ofile.close();
#endif
}
namespace mgpu {
	struct ScanOpMinPos {
		enum { Communative = true};
		typedef int2 input_type;
		typedef uint2 value_type;
		typedef int2 result_type;

		MGPU_HOST_DEVICE value_type Extract (value_type t, int index) { return t;}
		MGPU_HOST_DEVICE value_type Plus(value_type t1, value_type t2) { return make_uint2(min(t1.x,t2.x),min(t1.y,t2.x));};
		MGPU_HOST_DEVICE value_type Combine(value_type t1, value_type t2) { return t2; }
		MGPU_HOST_DEVICE value_type Identity() { return make_uint2((unsigned)_ident.x,(unsigned)_ident.y); }
		MGPU_HOST_DEVICE ScanOpMinPos(input_type ident) : _ident(ident) { }
		MGPU_HOST_DEVICE ScanOpMinPos() {
			_ident = make_int2(numeric_limits<int>::max(), numeric_limits<int>::max());
		}
		input_type _ident;
	};
}

const unsigned int BLOCK_STEP = 65535; // # of SIDs to process at once.

inline __host__ __device__ int2 min(int2 a, int2 b) { 
	return make_int2(min((unsigned)a.x, (unsigned)b.x), min((unsigned)a.y, (unsigned)b.y));
}
inline __host__ __device__ bool operator==(const int2& a, const int2&b) {
	return (a.x == b.x) && (a.y == b.y);
}

template <unsigned int blockSize>
__device__ void warpReduceMin(volatile int2 sdata[], unsigned int tid) {
	if (blockSize >= 64) sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 32].x);
	if (blockSize >= 64) sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 32].y); 
	if (blockSize >= 32) sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 16].x);
	if (blockSize >= 32) sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 16].y);
	if (blockSize >= 16) sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 8].x);
	if (blockSize >= 16) sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 8].y);
	if (blockSize >= 8)  sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 4].x);
	if (blockSize >= 8)  sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 4].y);
	if (blockSize >= 4)  sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 2].x);
	if (blockSize >= 4)  sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 2].y);
	if (blockSize >= 2)  sdata[tid].x = min((unsigned)sdata[tid].x, (unsigned)sdata[tid + 1].x);
	if (blockSize >= 2)  sdata[tid].y = min((unsigned)sdata[tid].y, (unsigned)sdata[tid + 1].y);
}
__host__ __device__ inline uint32_t pred_gate(uint32_t a, bool b) { return ((unsigned)(0-(!b))) &a; }
// Read the segment from the entry, determine the earliest pattern that marks it, and then write that (atomically).
// (likely) RESTRICTION: blockDim.x MUST be a power of 2
template <int N, unsigned int blockSize>
__global__ void kernSegmentReduce(segment<N, int2>* seglist, const GPU_DATA_type<coalesce_t> mark, const GPU_DATA_type<coalesce_t> sim, uint32_t startSegment, uint32_t startPattern) {
	__shared__ int2 midWarp[blockSize];
	midWarp[threadIdx.x] = make_int2(-1,-1);
	__syncthreads();

	uint32_t pid = threadIdx.x + blockIdx.x*blockDim.x;
	uint32_t real_pid = pid * 4 + startPattern; // unroll constant for coalesce_t
	pid += startPattern;
	uint32_t sid = blockIdx.y+startSegment;
	int2 simple = make_int2(-1,-1);
	if (real_pid < mark.width) {
		// In each thread, get 4 results-worth. 
		// This should unroll, as the trip count is known at compile time.
		// Get a batch of mark and sim results
		unsigned int mark_set = 0xffffffff;
		#pragma unroll
		for (uint8_t i = 0; i < N; i++) {
			// AND each mark result together.
			mark_set &= REF2D(mark, pid, seglist[sid].key.num[i]);
		}

		// check to see which position got marked. This will be one of 4 possible positions:
		// 1, 9, 17, 25 (as returned by ffs).
		// Post brev:
		// 32 = offset 0
		// 24 = offset 1
		// 16 = offset 2 // 8 = offset 3
		// We OR the original set with itself, shifted and then mask off the garbage.
		mark_set =  (mark_set | (mark_set >> 7) | (mark_set >> 14) | (mark_set >> 20)) & 0x0000000F;
		// reversing the bits puts the lowest bit first, 
		mark_set = __brev(mark_set);
		//which lets us get its position with ffs.
		unsigned int offset = (32 - __ffs(mark_set));
		offset *= (offset < 5);

//		int sim_row = (uint32_t*)((char*)sim.data + sim.pitch*3)[2].r;

		// Figure out the relevant entry on the simualtion table.
		unsigned int sim_type = REF2D(sim, pid, seglist[sid].key.num[0]).rows[offset];
	//	printf("%d,%d: %8X\n",seglist[sid].key.num[0],  offset, sim_type);

		// If the simulation results is T0 (or 2), then the real PID needs to be put into shared mem, x location
		midWarp[threadIdx.x].x = (offset+real_pid+1)*((mark_set > 0)&&(sim_type == T0)) - 1;
		
		// If the simulation results is T1 (or 3), then the real PID needs to be put into shared mem, x location
		midWarp[threadIdx.x].y = (offset+real_pid+1)*((mark_set > 0)&&(sim_type == T1)) - 1;
		
		// actual PID + 1 we are comparing against or 0 if not found.
		// Place in shared memory, decrementing to correct real PID.


		__syncthreads();
		// Now do the reduction inside the same warp, looking for min-positive.
		if (blockSize >= 512) { if (threadIdx.x < 256) { midWarp[threadIdx.x] = min(midWarp[threadIdx.x], midWarp[threadIdx.x+256]); } __syncthreads();} 
		if (blockSize >= 256) { if (threadIdx.x < 128) { midWarp[threadIdx.x] =min(midWarp[threadIdx.x], midWarp[threadIdx.x+128]); } __syncthreads();} 
		if (blockSize >= 128) { if (threadIdx.x < 64) { midWarp[threadIdx.x] =min(midWarp[threadIdx.x], midWarp[threadIdx.x+64]); } __syncthreads();} 
		if (threadIdx.x < 32) warpReduceMin<blockSize>(midWarp,threadIdx.x);
			__syncthreads();
		if (threadIdx.x == 0) {
			// brief loop until we know that the item we wrote with AtomicMin is the correct positive minimum
			int2 candidate = midWarp[0];
			int2 evict = make_int2(-1,-1);

			do { evict.x = atomicMin(&(seglist[sid].pattern.x), (unsigned)(candidate.x));
			} while (min((unsigned)evict.x,(unsigned)candidate.x) == evict.x && evict.x != candidate.x);
			do { evict.y = atomicMin(&(seglist[sid].pattern.y), (unsigned)(candidate.y));
			} while (min((unsigned)evict.y,(unsigned)candidate.y) == evict.y && evict.y != candidate.y);

//			printf("Lowest for block %d: (%d, %d)\n", sid, midWarp[0].x, midWarp[0].y);
		}
	}
	
}
/* Reduction strategy - X/1024 pattern blocks, Y blocks of lines/gates. Each
 * block gets the minimum ID within the block and places it into a temporary
 * location [BLOCK_X,BLOCK_Y] 
*/

float gpuMergeSegments(GPU_Data& mark, GPU_Data& sim, GPU_Circuit& ckt, size_t chunk, uint32_t ext_startPattern, void** seglist, int& numseg) {

	// if g_hashlist == NULL, copy the hash list to the GPU
//	ckt.print();
	segment<2, int2>* dc_seglist = (segment<2, int2>*)*seglist;
	uint32_t startPattern = ext_startPattern;
	uint32_t segcount = 0;
	if (dc_seglist == NULL) { 
		segment<2, int2>* h_seglist = NULL;
		generateSegmentList(&h_seglist,ckt);
		displaySegmentList(h_seglist, ckt);
		while (h_seglist[segcount].key.num[0] < ckt.size()) { segcount++;}
		hipMalloc(&dc_seglist, sizeof(segment<2,int2>)*segcount);
		hipMemcpy(dc_seglist, h_seglist, sizeof(segment<2,int2>)*segcount, hipMemcpyHostToDevice);
		std::cerr << "Allocating hashmap space of " << segcount << ".\n";
		HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
		numseg = segcount;
	} else { segcount = numseg; }
	#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
	// assume that the 0th entry is the widest, which is true given the chunking method.
#endif // NTIMING
	uint32_t count = 0;
	size_t remaining_blocks = segcount;
	const size_t block_x = (mark.gpu(chunk).width / MERGE_SIZE) + ((mark.gpu(chunk).width % MERGE_SIZE) > 0);
	size_t block_y = (remaining_blocks > BLOCK_STEP ? BLOCK_STEP : remaining_blocks);
	std::cerr << "Working with " << block_y << " / " << remaining_blocks << " sids.\n"; 

	GPU_DATA_type<coalesce_t> marks = toPod<coalesce_t>(mark,chunk);
	GPU_DATA_type<coalesce_t> sims = toPod<coalesce_t>(sim,chunk);
	debugMarkOutput(&marks, ckt, chunk, ext_startPattern, "gpumark-test.log");
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting on memory allocation
	do {
		dim3 blocks(block_x, block_y);
		kernSegmentReduce<2,MERGE_SIZE><<<blocks, MERGE_SIZE>>>(dc_seglist, marks, sims,count, startPattern);
		HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
		dim3 blocksmin(1, block_y);
		count+=BLOCK_STEP;
		if (remaining_blocks < BLOCK_STEP) { remaining_blocks = 0;}
		if (remaining_blocks >= BLOCK_STEP) { remaining_blocks -= BLOCK_STEP; }
		block_y = (remaining_blocks > BLOCK_STEP ? BLOCK_STEP : remaining_blocks);
		hipDeviceSynchronize();
	} while (remaining_blocks > 0);
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting inside the kernels
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
#ifdef LOGEXEC
	debugSegmentList(dc_seglist, numseg, "gpumerge.log");
#endif //LOGEXEC
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING

}
// scan through input until the first 1 is found, save the identifier and set all indicies above that.
void debugMergeOutput(size_t size, const void* res, std::string outfile) {
#ifndef NDEBUG
	int2 *lvalues, *results = (int2*)res;
	std::ofstream ofile(outfile.c_str());
	lvalues = new int2[size];
	hipMemcpy(lvalues,results,size*sizeof(int2),hipMemcpyDeviceToHost);
	for (size_t r = 0;r < size; r++) {
		ofile << "Gate " << r << ":\t";
		int2 z = lvalues[r];//REF2D(uint8_t, lvalues, results.pitch, r, i);
		ofile << std::setw(OUTJUST) << z.x << "," << z.y << " ";
		ofile << std::endl;
		}
	delete lvalues;
	ofile.close();
#endif
}
