#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpudata.h"
#include "defines.h"

GPU_Data::GPU_Data() {
	this->_gpu = new ARRAY2D<uint8_t>();
	this->_block_size = 0;
}
GPU_Data::GPU_Data(size_t rows, size_t columns) {
	this->_gpu = new ARRAY2D<uint8_t>();
	this->initialize(rows, columns, rows);
}
GPU_Data::GPU_Data(size_t rows, size_t columns, uint32_t blockwidth) {
	this->_gpu = new ARRAY2D<uint8_t>();
	this->initialize(rows, columns, blockwidth);
}
GPU_Data::~GPU_Data() {
	if (this->_gpu->data != NULL) {
		hipFree(this->_gpu->data);
	}
}
ARRAY2D<uint8_t> GPU_Data::gpu(uint32_t ref) {
	if (ref == this->_current) {
		return *(this->_gpu);
	}
//	DPRINT("%s:%d - Switching to chunk %d\n", __FILE__, __LINE__, ref);
	uint32_t tmp = this->_current;
	uint32_t err; 
	try {
		err = this->copy(ref);
	} catch (std::out_of_range& oor) { 
		// handling the problem by returning NULL and ensuring that _current is not changed.
		this->_current = tmp;
		DPRINT("Out of range in swap.\n");
		return ARRAY2D<uint8_t>(NULL,0,0,0);
	}
	if (err != ERR_NONE) {
		DPRINT("Unknown error in swap.\n");
		return ARRAY2D<uint8_t>(NULL,0,0,0);
	}
	return *(this->_gpu);
}

// total size in columns, rows. 
uint32_t GPU_Data::initialize(size_t in_columns, size_t in_rows, uint32_t block_width) {
	uint32_t chunks = (in_columns / block_width) + ((in_columns % block_width) > 0);

	this->_gpu = new ARRAY2D<uint8_t>(NULL, in_rows, block_width, sizeof(uint8_t)*block_width);
	hipMallocPitch(&(this->_gpu->data), &(this->_gpu->pitch), sizeof(uint8_t)*this->_gpu->width, in_rows);
	uint32_t rem_columns = in_columns;
	for (uint32_t i = 0; i < chunks;i++) {
		uint8_t* data = new uint8_t[in_rows*sizeof(uint8_t)*min(block_width,rem_columns)];
		this->_data->push_back(ARRAY2D<uint8_t>(data, in_rows, min(block_width, rem_columns),sizeof(uint8_t)*min(block_width,rem_columns)));
		assert(this->_data->back().data != NULL);
		if (rem_columns > block_width) {
			rem_columns -= block_width;
		}
		
	}
	this->_current = 0;
	this->_block_size = block_width;
	this->_width = in_columns;
	this->_height = in_rows;
	return ERR_NONE;
}

// performs a swap-out of GPU memory. 
uint32_t GPU_Data::copy(uint32_t ref) {
	uint32_t error;
//	DPRINT("%s:%d - Copying chunk %d from GPU, %d to GPU.\n",__FILE__,__LINE__,_current, ref);
	ARRAY2D<uint8_t>* cpu = &(this->_data->at(this->_current));
	ARRAY2D<uint8_t>* gpu = this->_gpu;
//	DPRINT("%s:%d - Memcpy from GPU\n", __FILE__,__LINE__);
	hipMemcpy2D(cpu->data, cpu->pitch, gpu->data, gpu->pitch, cpu->width * sizeof(uint8_t), cpu->height, hipMemcpyDeviceToHost);
//	DPRINT("%s:%d - Memcpy from GPU\n", __FILE__,__LINE__);
	error = hipGetLastError();
//	DPRINT("%s:%d - Getting reference to CPU", __FILE__,__LINE__);
	cpu = &(this->_data->at(ref));
	hipMemcpy2D(gpu->data, gpu->pitch, cpu->data, cpu->pitch, cpu->width * sizeof(uint8_t), cpu->height, hipMemcpyHostToDevice);
	gpu->width = cpu->width;
	gpu->height = cpu->height;
	error = hipGetLastError();
	this->_current = ref;
//	DPRINT("%s:%d - Finished copy.\n", __FILE__,__LINE__);
	if (error != hipSuccess)
		return ERR_NONE;
	return error;
}

uint32_t GPU_Data::refresh() {
	uint32_t error;
	ARRAY2D<uint8_t>* cpu = &(this->_data->at(this->_current));
	ARRAY2D<uint8_t>* gpu = this->_gpu;
	hipMemcpy2D(gpu->data, gpu->pitch, cpu->data, cpu->pitch, cpu->width*sizeof(uint8_t), cpu->height, hipMemcpyHostToDevice);
	error = hipGetLastError();
	if (error != hipSuccess)
		return ERR_NONE;
	return error;
}
std::string GPU_Data::debug() {
	std::stringstream st; 
	st << "GPU DATA,width="<<this->width()<<",height="<< this->height()<< ",pitch="<<this->gpu().pitch<<",blocksize="<< this->_block_size << ",chunks="<<this->_data->size()<<",current="<<this->_current << std::endl;
	return st.str();
}

__global__ void kernShift(uint8_t* array, uint8_t* tmpar, uint32_t pitch, uint32_t width, uint32_t height) {
	uint8_t tmp;
	uint32_t tid = (blockIdx.x *THREAD_SHIFT) + threadIdx.x;
	
	if (threadIdx.x < height) {
		tmp = REF2D(uint8_t,array,pitch, 0, tid);
		for (uint32_t i = 0; i < width-1; i++) {
			REF2D(uint8_t,array,pitch, i, tid) = REF2D(uint8_t,array,pitch, i+1, tid);
		}
		REF2D(uint8_t,array,pitch, width-1, tid) = tmp;
	}
}

void gpu_shift(GPU_Data& pack) {
	uint32_t per = (pack.gpu().height / THREAD_SHIFT) + ((pack.gpu().height % THREAD_SHIFT) > 0);
	uint8_t* tmpspace;
	hipMalloc(&tmpspace, sizeof(uint8_t)*pack.gpu().height);
	kernShift<<<per,THREAD_SHIFT>>>(pack.gpu().data, tmpspace, pack.gpu().pitch,pack.gpu().width,pack.gpu().height);
	hipDeviceSynchronize();
	assert(hipGetLastError() == hipSuccess);
}

void debugDataOutput(ARRAY2D<uint8_t> results, std::string outfile = "simdata.log") {
#ifndef NDEBUG
	uint8_t *lvalues;
	std::ofstream ofile(outfile.c_str());

	lvalues = (uint8_t*)malloc(results.height*results.pitch);
	hipMemcpy2D(lvalues,results.pitch,results.data,results.pitch,results.width,results.height,hipMemcpyDeviceToHost);
	for (uint32_t r = 0;r < results.width; r++) {
		for (uint32_t i = 0; i < results.height; i++) {
			uint8_t z = REF2D(uint8_t, lvalues, results.pitch, r, i);
			ofile << (uint32_t)z;
		}
		ofile << std::endl;
	}
	free(lvalues);
	ofile.close();
#endif
}
