#include "utility.h"
#include <hip/hip_runtime.h>
ARRAY2D<int> gpuAllocateBlockResults(size_t height) {
	int* tgt = NULL;
	hipMalloc(&tgt, sizeof(int)*(height));
	hipMemset(tgt, -1, sizeof(int)*height);
	return ARRAY2D<int>(tgt, height, 1);
}
