#include "hip/hip_runtime.h"
#include "utility.h"
#include "defines.h"
#include <hip/hip_runtime.h>
ARRAY2D<int32_t> gpuAllocateBlockResults(size_t height) {
	int32_t* tgt = NULL;
	hipMalloc(&tgt, sizeof(int)*(height));
	hipMemset(tgt, -1, sizeof(int)*height);
	return ARRAY2D<int32_t>(tgt, 1, height, sizeof(int32_t)*height);
}
void selectGPU() {
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	hipDeviceProp_t properties;
	if (num_devices > 1) {
		unsigned int max_multiprocessors = 0, max_device = 0;
		for (device = 0; device < num_devices; device++) {
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.totalGlobalMem) {
				max_multiprocessors = properties.totalGlobalMem;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
		hipGetDeviceProperties(&properties, max_device);
		DPRINT("Selected %s as GPU.\n", properties.name);
	}
}

int gpuCalculateSimulPatterns(int lines, int patterns) {
	// get free memory
	size_t free_mem, total_mem;
	int allowed_patterns;
	hipMemGetInfo(&free_mem, &total_mem);
	// added a buffer 	
	allowed_patterns = (free_mem - (lines*sizeof(int))) / (lines*(sizeof(int)*2.5) + sizeof(char)*1.5);
	return min(patterns, allowed_patterns -(allowed_patterns % 32));
}
