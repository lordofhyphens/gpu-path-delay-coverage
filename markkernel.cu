#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "iscas.h"
#include "defines.h"
#include "markkernel.h"
void HandleMarkError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleMarkError( err, __FILE__, __LINE__ ))
#define THREAD_PER_BLOCK 32
#define MTHREAD_BLOCK 32
texture<int, 2> and2OutputPropLUT;
texture<int, 2> and2InputPropLUT;
texture<int, 2> or2OutputPropLUT;
texture<int, 2> or2InputPropLUT;
texture<int, 2> xor2OutputPropLUT;
texture<int, 2> xor2InputPropLUT;
texture<int, 2> fromPropLUT;
texture<int, 2> inptPropLUT;
texture<int, 2> mergeLUT;

texture<int, 2> AndInChainLUT;
texture<int, 2> AndOutChainLUT;
texture<int, 2> OrInChainLUT;
texture<int, 2> OrOutChainLUT;
texture<int, 2> XorInChainLUT;
texture<int, 2> XorOutChainLUT;
texture<char, 2> inputTexture;
// group all results together, this implementation will fail if # of lines > 1024
// will need to group lines into groups of 1024 or less
__global__ void kernMerge(char* input, char* results,int width, int height, int pitch, int rpitch) {
	char *mrow, *irow;
	__shared__ char current[MTHREAD_BLOCK];
	int tid = (blockDim.y * blockIdx.y) + threadIdx.x;
	if (tid < width) {
		results[tid] = input[tid];
		irow = input+(blockIdx.x*pitch);
		mrow = results+(blockIdx.x*rpitch);
		current[threadIdx.x] = irow[0];
		for (unsigned int i = 1; i < tid; i++){
			current[threadIdx.x] = irow[i] || current[threadIdx.x];
		}
		mrow[tid] = current[threadIdx.x];
	}
}

/*
   	Parallel reduction
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)
	Modified from Cuda SDK to fit our specific problem (not sum, but logical OR, which is close enough.)
*/
template <unsigned int blockSize>
__global__ void reduce6(char *g_idata, char *g_odata, size_t n, size_t pitch)
{
    extern __shared__ int sdata[];
	char* irow = g_idata+(blockIdx.x*pitch);
	char* mrow = g_odata+(blockIdx.x*pitch);

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.y*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.y;
    sdata[tid] = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridSize).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        sdata[tid] |= irow[i] || irow[i+blockSize];  
        i += gridSize;
    } 
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] |= sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] |= sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] |= sdata[tid +  64]; } __syncthreads(); }
    
	// unroll at compile time! 
    {
        if (blockSize >=  64) { sdata[tid] |= sdata[tid + 32];  }
        if (blockSize >=  32) { sdata[tid] |= sdata[tid + 16];  }
        if (blockSize >=  16) { sdata[tid] |= sdata[tid +  8];  }
        if (blockSize >=   8) { sdata[tid] |= sdata[tid +  4];  }
        if (blockSize >=   4) { sdata[tid] |= sdata[tid +  2];  }
        if (blockSize >=   2) { sdata[tid] |= sdata[tid +  1];  }
    }
    
    // write result for this block to global mem 
    if (tid == 0) mrow[blockIdx.x] = sdata[0];
}

void loadPropLUTs() {
	// Creating a set of static arrays that represent our LUTs
		// Addressing for the propagations:
	// 2 4x4 groups such that 
	int and2_output_prop[16]= {0,0,0,0,0,2,1,1,0,1,1,0,0,1,1,1};
	int and2_input_prop[16] = {0,0,0,0,0,0,1,1,0,0,1,0,0,0,1,1};
	int or2_output_prop[16] = {2,0,1,1,0,0,0,0,1,0,1,1,1,0,1,1};
	int or2_input_prop[16]  = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int xor2_input_prop[16] = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int xor2_output_prop[16]= {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int from_prop[16]       = {0,0,0,0,0,0,0,0,0,0,1,1,0,0,1,1};
	int inpt_prop[8]        = {0,0,0,0,0,0,1,1};

	int and_outp_chain[8]   = {0,0,1,0,0,1,1,1};
	int and_inp_chain[8]    = {0,0,0,0,0,1,1,1};
	int or_outp_chain[8]    = {0,0,0,1,1,0,1,1};
	int or_inp_chain[8]     = {0,0,0,0,1,0,1,1};
	int xor_outp_chain[8]   = {0,0,0,0,0,0,0,0};
	int xor_inp_chain[8]    = {0,0,0,0,0,0,0,0};

	hipExtent volumeSize = make_hipExtent(4,4,2);
	// device memory arrays, required. 
	hipArray *cuAndInptProp, *cuAndOutpProp, *cuOrInptProp, *cuOrOutpProp, *cuFromProp, *cuInptProp, *cuXorInptProp, *cuXorOutpProp;
	hipArray *cuAndOutChain, *cuAndInChain, *cuOrInChain, *cuOrOutChain, *cuXorInChain, *cuXorOutChain;
	// generic formatting information. All of our arrays are the same, so sharing it shouldn't be a problem.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	// Allocating memory on the device.
		
	HANDLE_ERROR(hipMallocArray(&cuFromProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuInptProp, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuAndOutpProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrOutpProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorOutpProp, &channelDesc, 4,4));

	HANDLE_ERROR(hipMallocArray(&cuXorOutChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuXorInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuOrInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuOrOutChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndOutChain, &channelDesc, 4,2));

	// Copying the LUTs Host->Device
	HANDLE_ERROR(hipMemcpyToArray(cuFromProp, 0,0, from_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndInptProp, 0,0, and2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndOutpProp, 0,0, and2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrInptProp, 0,0, or2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrOutpProp, 0,0, or2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorInptProp, 0,0, xor2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorOutpProp, 0,0, xor2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuInptProp, 0,0, inpt_prop, sizeof(int)*8,hipMemcpyHostToDevice));
	
	HANDLE_ERROR(hipMemcpyToArray(cuXorInChain, 0,0, xor_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorOutChain, 0,0, xor_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrInChain, 0,0, or_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrOutChain, 0,0, or_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndInChain, 0,0, and_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndOutChain, 0,0, and_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));

	// Marking them as textures. LUTs should be in texture memory and cached on
	// access.
	HANDLE_ERROR(hipBindTextureToArray(and2OutputPropLUT,cuAndOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(and2InputPropLUT,cuAndInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2OutputPropLUT,cuOrOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2InputPropLUT,cuOrInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2InputPropLUT,cuXorInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2OutputPropLUT,cuXorOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(fromPropLUT,cuFromProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(inptPropLUT,cuInptProp,channelDesc));
	
	HANDLE_ERROR(hipBindTextureToArray(XorOutChainLUT,cuXorOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(XorInChainLUT,cuXorInChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(OrOutChainLUT,cuOrOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(OrInChainLUT,cuOrInChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(AndOutChainLUT,cuAndOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(AndInChainLUT,cuAndInChain,channelDesc));
}

__global__ void kernMarkPathSegments(char *input, char* results, GPUNODE* node, int* fans, size_t width, size_t height, int start, int pitch) {
	int tid = (blockIdx.y * blockDim.y) + threadIdx.x, nfi, goffset,val,prev;
	int gid = (blockIdx.x) + start;
	__shared__ char rowCache[THREAD_PER_BLOCK][100];
	__shared__ char resultCache[THREAD_PER_BLOCK][100];
	char cache;
	int tmp = 1, pass = 0, fin1 = 0, fin2 = 0,fin = 1, type,g;
	char *rowResults;
	char *row;
	if (tid < height) {
		cache = 0;
		row = (char*)((char*)input + gid*pitch);
		rowResults = (char*)((char*)results + gid*pitch);
		tmp = 1;
		nfi = node[gid].nfi;
		type = node[gid].type;
		goffset = node[gid].offset;
		rowCache[threadIdx.x][0] = row[tid];
		resultCache[threadIdx.x][0] = rowResults[tid];
		for (int q = 0; q < nfi; q++) {
//			printf("T: %d BID: %d G: %d Placing %d in gate position %d\n", tid, blockIdx.x, gid, ((char*)input+(fans[goffset+q]*pitch))[tid], fans[goffset+q]);
			rowCache[threadIdx.x][q+1] = ((char*)input+(fans[goffset+q]*pitch))[tid];
			resultCache[threadIdx.x][q+1] = ((char*)results+(fans[goffset+q]*pitch))[tid];
		}
		__syncthreads();
		// switching based on value causes divergence, switch based on node type.
		val = (rowCache[threadIdx.x][0] > 1);
		if (node[gid].po) {
			resultCache[threadIdx.x][0] = val;
			prev = val;
		} else {
			prev = resultCache[threadIdx.x][0];
		}
		switch(type) {
			case FROM:
				// For FROM, only set the "input" line if it hasn't already
				// been set (otherwise it'll overwrite the decision of
				// another system somewhere else.
				val = (resultCache[threadIdx.x][0] > 0 && (rowCache[threadIdx.x][0] > 1));
				g = val || (resultCache[threadIdx.x][1] > 0);
				resultCache[threadIdx.x][1] |= g;
				resultCache[threadIdx.x][0] = val;
				break;
			case BUFF:
			case NOT:
				val = tex2D(inptPropLUT, rowCache[threadIdx.x][0],resultCache[threadIdx.x][0]) && prev;
				resultCache[threadIdx.x][1] = val;
				resultCache[threadIdx.x][0] = val;
				break;
				// For the standard gates, setting three values -- both the
				// input lines and the output line.  rowCache[threadIdx.x][i]-1 is the
				// transition on the output, offset to make the texture
				// calculations correct because there are 4 possible values
				// rowCache[threadIdx.x][i] can take: 0, 1, 2, 3.  0, 1 are the same, as are
				// 2,3, so we subtract 1 and clamp to an edge if we
				// overflow.
				// 0 becomes -1 (which clamps to 0)
				// 1 becomes 0
				// 2 becomes 1
				// 3 becomes 2 (which clamps to 1)
				// There's only 2 LUTs for each gate type. The input LUT
				// checks for path existance through the first input, so we
				// call it twice with the inputs reversed to check both
				// paths.

			case NAND:
			case AND:
				for (fin1 = 1; fin1 <= nfi; fin1++) {
					for (fin2 = 1; fin2 <= nfi; fin2++) {
						if (fin1 == fin2) continue;
						cache = tex2D(and2OutputPropLUT, rowCache[threadIdx.x][fin1], rowCache[threadIdx.x][fin2]);
						pass += (cache > 1);
						tmp = tmp && (cache > 0);
					}
				}
				resultCache[threadIdx.x][0] = val && tmp && (pass <= nfi) && prev;
//				printf("T: %d BID: %d G: %d PASS: %d, TMP: %d, PREV: %d, VAL: %d\n", tid, blockIdx.x, gid, pass, tmp, prev, resultCache[threadIdx.x][0]);
				break;
			case OR:
			case NOR:
				for (fin1 = 0; fin1 < nfi; fin1++) {
					fin = 1;
					for (fin2 = 0; fin2 < nfi; fin2++) {
						if (fin1 == fin2) continue;
						cache = tex2D(or2OutputPropLUT, rowCache[threadIdx.x][fin1], rowCache[threadIdx.x][fin2]);
						pass += (cache > 1);
						tmp = tmp && (cache > 0);
					}
				}
				resultCache[threadIdx.x][0] = val && tmp && (pass <= nfi) && prev;
				break;
			case XOR:
			case XNOR:
				for (fin1 = 0; fin1 < nfi; fin1++) {
					fin = 1;
					for (fin2 = 0; fin2 < nfi; fin2++) {
						if (fin1 == fin2) continue;
						cache = tex2D(xor2OutputPropLUT, rowCache[threadIdx.x][fin1], rowCache[threadIdx.x][fin2]);
						pass += (cache > 1);
						tmp = tmp && (cache > 0);
					}
				}
				resultCache[threadIdx.x][0] = val && tmp && (pass <= nfi) && prev;
				break;
			default:
				// if there is a transition that will propagate, set = to some positive #?
				break;
		}
		for (fin1 = 0; fin1 < nfi; fin1++) {
			if (nfi < 2) continue;
			fin = 1;
			for (fin2 = 0; fin2 < nfi; fin2++) {
				if (fin1 == fin2) continue;
				switch(type) {
					case AND:
					case NAND:
						cache = tex2D(and2InputPropLUT, rowCache[threadIdx.x][fin1+1], rowCache[threadIdx.x][fin2+1]); break;
					case OR:
					case NOR:
						cache = tex2D(or2InputPropLUT, rowCache[threadIdx.x][fin1+1], rowCache[threadIdx.x][fin2+1]); break;
					case XOR:
					case XNOR:
						cache = tex2D(xor2InputPropLUT, rowCache[threadIdx.x][fin1+1], rowCache[threadIdx.x][fin2+1]); break;
				}
				fin = cache && fin && prev;
			}
			resultCache[threadIdx.x][fin1+1] = fin;
		}
		// stick the contents of resultCache into the results array
		__syncthreads();

//		printf("T: %d BID: %d G: %d PASS: %d, TMP: %d, PREV: %d, VAL: %d\n", tid, blockIdx.x, gid, pass, tmp, prev, resultCache[threadIdx.x][0]);
		rowResults[tid] = resultCache[threadIdx.x][0];
		for (int j = 0; j < nfi; j++) {
			((char*)results+(fans[goffset+j]*pitch))[tid] = resultCache[threadIdx.x][j+1];
		}
	}
}

float gpuMarkPaths(ARRAY2D<char> input, ARRAY2D<char> results, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph,  int* fan, int maxlevels) {
	loadPropLUTs();
	int *gatesinLevel, startGate=0;
	gatesinLevel = new int[maxlevels];
	for (int i = 0; i < maxlevels; i++) {
		gatesinLevel[i] = 0;
		for (unsigned int j = 0; j < results.width; j++) {
			if (graph[j].level == i) {
				gatesinLevel[i]++;
			}
		}
		startGate += gatesinLevel[i];
	}
	int blockcount_y = (int)(results.height/THREAD_PER_BLOCK) + (results.height%THREAD_PER_BLOCK > 0);

#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	for (int i = maxlevels-1; i >= 0; i--) {
		dim3 numBlocks(gatesinLevel[i],blockcount_y);
		startGate -= gatesinLevel[i];
//		DPRINT("Starting gate: %d, level %d\n", startGate, i);
		kernMarkPathSegments<<<numBlocks,THREAD_PER_BLOCK>>>(input.data, results.data, dgraph.data, fan, results.width, results.height, startGate, results.pitch);
		hipDeviceSynchronize();
	}
	free(gatesinLevel);
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}
float gpuMergeHistory(ARRAY2D<char> input, ARRAY2D<char> *mergeresult, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, int* fan) {
//	int blockcount = (input.height / MTHREAD_BLOCK) + 1;
#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	
	dim3 blocks(input.width, input.height);
	//kernMerge<<<blocks,MTHREAD_BLOCK>>>(input.data, mergeresult->data, input.width, input.height, input.pitch, mergeresult->pitch);
	reduce6<MTHREAD_BLOCK><<< blocks, MTHREAD_BLOCK, MTHREAD_BLOCK*sizeof(int) >>>(input.data, mergeresult->data, input.height, input.pitch);

	hipDeviceSynchronize();
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
//	hipMemcpy2D(mergeresult->data, mergeresult->pitch, input.data, input.pitch, input.width, input.height, hipMemcpyDeviceToDevice);
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}

void debugMarkOutput(ARRAY2D<char> results) {
#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.
	char *lvalues, *row;
	DPRINT("Post-mark results\n");
	DPRINT("Vector:   \t");
	for (unsigned int i = 0; i < results.height; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (unsigned int r = 0;r < results.width; r++) {
		lvalues = (char*)malloc(results.height*sizeof(char));
		row = ((char*)results.data + r*results.pitch); // get the current row?
		hipMemcpy(lvalues,row,results.height*sizeof(char),hipMemcpyDeviceToHost);
		
		DPRINT("%s\t%d:\t","Line",r);
		for (unsigned int i = 0; i < results.height; i++) {
			DPRINT("%2d ", lvalues[i]);//== 0 ? 'N':'S'  );
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif 
}
void debugUnionOutput(ARRAY2D<int> results) {
#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.
	int *lvalues, *row;
	DPRINT("Post-union results\n");
	DPRINT("Line:   \t");
	for (unsigned int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (unsigned int r = 0;r < results.height; r++) {
		lvalues = (int*)malloc(results.bwidth());
		row = (int*)((char*)results.data + r*results.pitch); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		
		DPRINT("%s %d:\t", "Vector",r);
		for (unsigned int i = 0; i < results.width; i++) {
			DPRINT("%2c ", lvalues[i] == 0 ? 'N':'S'  );
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif 
}
