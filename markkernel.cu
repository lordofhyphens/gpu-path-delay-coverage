#include "hip/hip_runtime.h"
#include "markkernel.h"
void HandleMarkError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleMarkError( err, __FILE__, __LINE__ ))
texture<int, 2> and2OutputPropLUT;
texture<int, 2> and2InputPropLUT;
texture<int, 2> or2OutputPropLUT;
texture<int, 2> or2InputPropLUT;
texture<int, 2> xor2OutputPropLUT;
texture<int, 2> xor2InputPropLUT;
texture<int, 2> fromPropLUT;
texture<int, 2> inptPropLUT;
texture<int, 2> mergeLUT;

texture<int, 2> AndInChainLUT;
texture<int, 2> AndOutChainLUT;
texture<int, 2> OrInChainLUT;
texture<int, 2> OrOutChainLUT;
texture<int, 2> XorInChainLUT;
texture<int, 2> XorOutChainLUT;
texture<char, 2> inputTexture;


void loadPropLUTs() {
	// Creating a set of static arrays that represent our LUTs
		// Addressing for the propagations:
	// 2 4x4 groups such that 
	int and2_output_prop[16]= {0,0,0,0,0,2,1,1,0,1,1,0,0,1,1,1};
	int and2_input_prop[16] = {0,0,0,0,0,0,1,1,0,0,1,0,0,0,1,1};
	int or2_output_prop[16] = {2,0,1,1,0,0,0,0,1,0,1,1,1,0,1,1};
	int or2_input_prop[16]  = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int xor2_input_prop[16] = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int xor2_output_prop[16]= {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int from_prop[16]       = {0,0,0,0,0,0,0,0,0,0,1,1,0,0,1,1};
	int inpt_prop[8]        = {0,0,0,0,0,0,1,1};

	int and_outp_chain[8]   = {0,0,1,0,0,1,1,1};
	int and_inp_chain[8]    = {0,0,0,0,0,1,1,1};
	int or_outp_chain[8]    = {0,0,0,1,1,0,1,1};
	int or_inp_chain[8]     = {0,0,0,0,1,0,1,1};
	int xor_outp_chain[8]   = {0,0,0,0,0,0,0,0};
	int xor_inp_chain[8]    = {0,0,0,0,0,0,0,0};

	hipExtent volumeSize = make_hipExtent(4,4,2);
	// device memory arrays, required. 
	hipArray *cuAndInptProp, *cuAndOutpProp, *cuOrInptProp, *cuOrOutpProp, *cuFromProp, *cuInptProp, *cuXorInptProp, *cuXorOutpProp;
	hipArray *cuAndOutChain, *cuAndInChain, *cuOrInChain, *cuOrOutChain, *cuXorInChain, *cuXorOutChain;
	// generic formatting information. All of our arrays are the same, so sharing it shouldn't be a problem.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	// Allocating memory on the device.
		
	HANDLE_ERROR(hipMallocArray(&cuFromProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuInptProp, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuAndOutpProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrOutpProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorOutpProp, &channelDesc, 4,4));

	HANDLE_ERROR(hipMallocArray(&cuXorOutChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuXorInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuOrInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuOrOutChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndOutChain, &channelDesc, 4,2));

	// Copying the LUTs Host->Device
	HANDLE_ERROR(hipMemcpyToArray(cuFromProp, 0,0, from_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndInptProp, 0,0, and2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndOutpProp, 0,0, and2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrInptProp, 0,0, or2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrOutpProp, 0,0, or2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorInptProp, 0,0, xor2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorOutpProp, 0,0, xor2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuInptProp, 0,0, inpt_prop, sizeof(int)*8,hipMemcpyHostToDevice));
	
	HANDLE_ERROR(hipMemcpyToArray(cuXorInChain, 0,0, xor_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorOutChain, 0,0, xor_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrInChain, 0,0, or_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrOutChain, 0,0, or_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndInChain, 0,0, and_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndOutChain, 0,0, and_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));

	// Marking them as textures. LUTs should be in texture memory and cached on
	// access.
	HANDLE_ERROR(hipBindTextureToArray(and2OutputPropLUT,cuAndOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(and2InputPropLUT,cuAndInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2OutputPropLUT,cuOrOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2InputPropLUT,cuOrInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2InputPropLUT,cuXorInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2OutputPropLUT,cuXorOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(fromPropLUT,cuFromProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(inptPropLUT,cuInptProp,channelDesc));
	
	HANDLE_ERROR(hipBindTextureToArray(XorOutChainLUT,cuXorOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(XorInChainLUT,cuXorInChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(OrOutChainLUT,cuOrOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(OrInChainLUT,cuOrInChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(AndOutChainLUT,cuAndOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(AndInChainLUT,cuAndInChain,channelDesc));
}

__global__ void kernMarkPathSegments(char *input, char* results, GPUNODE* node, int* fans, size_t width, size_t height, int start, int pitch) {
	int tid = (blockIdx.y * blockDim.y) + threadIdx.x, nfi, goffset,val,prev;
	int gid = (blockIdx.x) + start;
	__shared__ char rowCache[MARK_BLOCK];
	__shared__ char resultCache[MARK_BLOCK];
	char cache, fin = 1;
	int tmp = 1, pass = 0, fin1 = 0, fin2 = 0,type, g;
	char *rowResults;
	char *row;
	if (tid < height) {
		cache = 0;
		row = (char*)((char*)input + gid*pitch);
		rowResults = (char*)((char*)results + gid*pitch);
		tmp = 1;
		nfi = node[gid].nfi;
		type = node[gid].type;
		goffset = node[gid].offset;
		rowCache[threadIdx.x] = row[tid];
		resultCache[threadIdx.x] = rowResults[tid];
/*		for (int q = 0; q < nfi; q++) {
			cache_f = fans[goffset+q];
			resultCache[threadIdx.x][q+1] = ((char*)results+(cache_f*pitch))[tid];
		}
 */
		__syncthreads();
		// switching based on value causes divergence, switch based on node type.
		val = (rowCache[threadIdx.x] > 1);
		if (node[gid].po > 0) {
			resultCache[threadIdx.x] = val;
			prev = val;
		} else {
			prev = resultCache[threadIdx.x];
		}
		switch(type) {
			case FROM:
				// For FROM, only set the "input" line if it hasn't already
				// been set (otherwise it'll overwrite the decision of
				// another system somewhere else.
				val = (resultCache[threadIdx.x] > 0 && (rowCache[threadIdx.x] > 1));
				g = val || (REF2D(char,input,pitch,FIN(fans,goffset,0),tid) > 0);
				REF2D(char,input,pitch,FIN(fans,goffset,0),tid) |= g;
				resultCache[threadIdx.x] = val;
				break;
			case BUFF:
			case NOT:
				val = tex2D(inptPropLUT, rowCache[threadIdx.x],resultCache[threadIdx.x]) && prev;
				((char*)row+(fans[goffset]*pitch))[tid] = val;
				resultCache[threadIdx.x] = val;
				break;
				// For the standard gates, setting three values -- both the
				// input lines and the output line.  rowCache[threadIdx.x][i]-1 is the
				// transition on the output, offset to make the texture
				// calculations correct because there are 4 possible values
				// rowCache[threadIdx.x][i] can take: 0, 1, 2, 3.  0, 1 are the same, as are
				// 2,3, so we subtract 1 and clamp to an edge if we
				// overflow.
				// 0 becomes -1 (which clamps to 0)
				// 1 becomes 0
				// 2 becomes 1
				// 3 becomes 2 (which clamps to 1)
				// There's only 2 LUTs for each gate type. The input LUT
				// checks for path existance through the first input, so we
				// call it twice with the inputs reversed to check both
				// paths.

			case NAND:
			case AND:
				for (fin1 = 1; fin1 <= nfi; fin1++) {
					fin = 1;
					for (fin2 = 1; fin2 <= nfi; fin2++) {
						if (fin1 == fin2) continue;
						cache = tex2D(and2OutputPropLUT, REF2D(char,input,pitch,FIN(fans,goffset,fin1),tid), REF2D(char,input,pitch,FIN(fans,goffset,fin2),tid));
						pass += (cache > 1);
						tmp = tmp && (cache > 0);
						if (nfi > 2) {
							cache = tex2D(and2InputPropLUT, REF2D(char,input,pitch,FIN(fans,goffset,fin1),tid), REF2D(char,input,pitch,FIN(fans,goffset,fin2),tid));
							fin = cache && fin && prev;
						}
					}
					((char*)results+(fans[goffset+fin1]*pitch))[tid] = fin;
				}

				resultCache[threadIdx.x] = val && tmp && (pass < nfi) && prev;
				break;
			case OR:
			case NOR:
				for (fin1 = 0; fin1 < nfi; fin1++) {
					fin = 1;
					for (fin2 = 0; fin2 < nfi; fin2++) {
						if (fin1 == fin2) continue;
						cache = tex2D(or2OutputPropLUT, REF2D(char,input,pitch,FIN(fans,goffset,fin1),tid), REF2D(char,input,pitch,FIN(fans,goffset,fin2),tid));
						pass += (cache > 1);
						tmp = tmp && (cache > 0);

						if (nfi > 2) {
							cache = tex2D(or2InputPropLUT, REF2D(char,input,pitch,FIN(fans,goffset,fin1),tid), REF2D(char,input,pitch,FIN(fans,goffset,fin2),tid));
							fin = cache && fin && prev;
						}

					}
					((char*)results+(fans[goffset+fin1]*pitch))[tid] = fin;
				}
				resultCache[threadIdx.x] = val && tmp && (pass <= nfi) && prev;
				break;
			case XOR:
			case XNOR:
				for (fin1 = 0; fin1 < nfi; fin1++) {
					fin = 1;
					for (fin2 = 0; fin2 < nfi; fin2++) {
						if (fin1 == fin2) continue;
						cache = tex2D(xor2OutputPropLUT, REF2D(char,input,pitch,FIN(fans,goffset,fin1),tid), REF2D(char,input,pitch,FIN(fans,goffset,fin2),tid));
						pass += (cache > 1);
						tmp = tmp && (cache > 0);
						if (nfi > 2) {
							cache = tex2D(xor2InputPropLUT, REF2D(char,input,pitch,FIN(fans,goffset,fin1),tid), REF2D(char,input,pitch,FIN(fans,goffset,fin2),tid));
							fin = cache && fin && prev;
						}
					}
					((char*)results+(fans[goffset+fin1]*pitch))[tid] = fin;
				}
				resultCache[threadIdx.x] = val && tmp && (pass <= nfi) && prev;
				break;
			default:
				// if there is a transition that will propagate, set = to some positive #?
				break;
		}
		// stick the contents of resultCache into the results array

		rowResults[tid] = resultCache[threadIdx.x];
	}
}

float gpuMarkPaths(ARRAY2D<char> input, ARRAY2D<char> results, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph,  int* fan, int maxlevels) {
	loadPropLUTs();
	int *gatesinLevel, startGate=0;
	gatesinLevel = new int[maxlevels];
	for (int i = 0; i < maxlevels; i++) {
		gatesinLevel[i] = 0;
		for (unsigned int j = 0; j < results.width; j++) {
			if (graph[j].level == i) {
				gatesinLevel[i]++;
			}
		}
		startGate += gatesinLevel[i];
	}
	int blockcount_y = (int)(input.height/MARK_BLOCK) + (input.height%MARK_BLOCK > 0);

#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	for (int i = maxlevels-1; i >= 0; i--) {
		dim3 numBlocks(gatesinLevel[i],blockcount_y);
		startGate -= gatesinLevel[i];
		kernMarkPathSegments<<<numBlocks,MARK_BLOCK>>>(input.data, results.data, dgraph.data, fan, results.width, results.height, startGate, results.pitch);
		hipDeviceSynchronize();
	}
	delete gatesinLevel;
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}


void debugMarkOutput(ARRAY2D<char> results) {
#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.
	char *lvalues, *row;
	DPRINT("Post-mark results\n");
	DPRINT("Vector:   \t");
	for (unsigned int i = 0; i < results.height; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (unsigned int r = 0;r < results.width; r++) {
		lvalues = (char*)malloc(results.height*sizeof(char));
		row = ((char*)results.data + r*results.pitch); // get the current row?
		hipMemcpy(lvalues,row,results.height*sizeof(char),hipMemcpyDeviceToHost);
		
		DPRINT("%s\t%d:\t","Line",r);
		for (unsigned int i = 0; i < results.height; i++) {
			DPRINT("%2d ", lvalues[i]);//== 0 ? 'N':'S'  );
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif 
}
void debugUnionOutput(ARRAY2D<int> results) {
#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.
	int *lvalues, *row;
	DPRINT("Post-union results\n");
	DPRINT("Line:   \t");
	for (unsigned int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (unsigned int r = 0;r < results.height; r++) {
		lvalues = (int*)malloc(results.bwidth());
		row = (int*)((char*)results.data + r*results.pitch); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		
		DPRINT("%s %d:\t", "Vector",r);
		for (unsigned int i = 0; i < results.width; i++) {
			DPRINT("%2c ", lvalues[i] == 0 ? 'N':'S'  );
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif 
}
