#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "iscas.h"
#include "defines.h"
#include "markkernel.h"

texture<int, 3> and2OutputPropLUT;
texture<int, 3> and2InputPropLUT;
texture<int, 3> or2OutputPropLUT;
texture<int, 3> or2InputPropLUT;
texture<int, 2> fromPropLUT;
texture<int, 2> inptPropLUT;
texture<int, 2> mergeLUT;

// group all results together, this implementation will fail if # of lines > 1024
// will need to group lines into groups of 1024 or less
__global__ void kernMerge(int* input, int* results, int width) {
	int *r,result, i;
	if (threadIdx.x < width) {
		result = 0;
		for (i = 0; i < blockIdx.x; i++) {
			r = (int*)((char*)input + sizeof(int)*i*width);
			result = tex2D(mergeLUT,result,r[threadIdx.x]);
		}
		r = (int*)((char*)results + sizeof(int)*width*blockIdx.x);
		r[threadIdx.x] = result;
	}
}
void loadMergeLUTs() {
	int merge[4] = {0,1,1,1};
	hipArray *cuMerge;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();	
	hipMallocArray(&cuMerge,&channelDesc, 2, 2);
	hipMemcpyToArray(cuMerge, 0,0, merge, sizeof(int)*4,hipMemcpyHostToDevice);
	hipBindTextureToArray(mergeLUT,cuMerge,channelDesc);
}
void loadPropLUTs() {
	// Creating a set of static arrays that represent our LUTs
		// Addressing for the propagations:
	// 2 4x4 groups such that 
	int and2_output_prop[32] ={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,1,1,1,1,1,1,0,1,1,1};
	int and2_input_prop[32] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,1,0,0,0,1,1};
	int or2_output_prop[32] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,1,0,1,1,1,0,1,1};
	int or2_input_prop[32] =  {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int from_prop[16]      =  {0,0,0,0,0,0,0,0,0,0,1,1,0,0,1,1};
	int inpt_prop[8] = {0,0,0,0,0,0,1,1};

	hipExtent volumeSize = make_hipExtent(4,4,2);
	// device memory arrays, required. 
	hipArray *cuAndInptProp, *cuAndOutpProp, *cuOrInptProp, *cuOrOutpProp, *cuFromProp, *cuInptProp;
	// generic formatting information. All of our arrays are the same, so sharing it shouldn't be a problem.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	// Allocating memory on the device.
		
	hipMallocArray(&cuFromProp, &channelDesc, 4,4);
	hipMallocArray(&cuInptProp, &channelDesc, 4,2);
	hipMalloc3DArray(&cuAndInptProp, &channelDesc, volumeSize);
	hipMalloc3DArray(&cuAndOutpProp, &channelDesc, volumeSize);
	hipMalloc3DArray(&cuOrInptProp, &channelDesc, volumeSize);
	hipMalloc3DArray(&cuOrOutpProp, &channelDesc, volumeSize);

	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr = make_hipPitchedPtr((void*)and2_output_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuAndOutpProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	and2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	copyParams.srcPtr = make_hipPitchedPtr((void*)and2_input_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuAndInptProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	and2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	copyParams.srcPtr = make_hipPitchedPtr((void*)or2_output_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuOrOutpProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	or2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	copyParams.srcPtr = make_hipPitchedPtr((void*)or2_input_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuOrInptProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	or2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	// Copying the LUTs Host->Device
	hipMemcpyToArray(cuFromProp, 0,0, from_prop, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuInptProp, 0,0, inpt_prop, sizeof(int)*8,hipMemcpyHostToDevice);

	// Marking them as textures. LUTs should be in texture memory and cached on
	// access.
	hipBindTextureToArray(and2OutputPropLUT,cuAndOutpProp,channelDesc);
	hipBindTextureToArray(and2InputPropLUT,cuAndInptProp,channelDesc);
	hipBindTextureToArray(or2OutputPropLUT,cuOrOutpProp,channelDesc);
	hipBindTextureToArray(or2InputPropLUT,cuOrInptProp,channelDesc);
	hipBindTextureToArray(fromPropLUT,cuFromProp,channelDesc);
	hipBindTextureToArray(inptPropLUT,cuInptProp,channelDesc);
}

__device__ int willPathPropagate(int tid, int* results, GPUNODE* node, int* fans, size_t width) {
	return -1;
}
__global__ void kernMarkPathSegments(int *results, GPUNODE* node, int* fans, size_t width, size_t height, int ncount) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, nfi, goffset,val;
	int *rowResults, *row;
	if (tid < height) {
		row = (int*)((char*)results + tid*(width)*sizeof(int));
		rowResults = (int*)malloc(sizeof(int)*width);
		for (int i = 0; i < width; i++) {
			rowResults[i] = UNINITIALIZED;
		}
		for (int i = ncount; i >= 0; i--) {
			val = UNINITIALIZED;
			goffset = node[i].offset;
			nfi = node[i].nfi;
			// switching based on value causes divergence, switch based on node type.
			switch(node[i].type) {
				
				case FROM:
					// For FROM, only set the "input" line if it hasn't already
					// been set (otherwise it'll overwrite the decision of
					// another system somewhere else.
					if (rowResults[fans[goffset]] == UNINITIALIZED) {
						val = tex2D(inptPropLUT, row[fans[goffset]],rowResults[fans[goffset+nfi]]);
						rowResults[fans[goffset]] = val;
						rowResults[fans[goffset+nfi]] = val;
					} else {
						val = tex2D(inptPropLUT, row[fans[goffset]],rowResults[fans[goffset+nfi]]);
						rowResults[fans[goffset+nfi]] = val;
					}
					break;
					// For the standard gates, setting three values -- both the input lines and the output line.
				case NAND:
				case AND:
					rowResults[fans[goffset]] = tex3D(and2InputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1);
					rowResults[fans[goffset+1]] = tex3D(and2InputPropLUT, row[fans[goffset+1]],row[fans[goffset]],row[fans[goffset+nfi]]-1);
					rowResults[fans[goffset+nfi]] = tex3D(and2OutputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1);
					break;
				case OR:
				case NOR:
					rowResults[fans[goffset]] = tex3D(or2InputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1);
					rowResults[fans[goffset+1]] = tex3D(or2InputPropLUT, row[fans[goffset+1]],row[fans[goffset]],row[fans[goffset+nfi]]-1);
					rowResults[fans[goffset+nfi]] = tex3D(or2OutputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1);
					break;
				case XOR:
				case XNOR:
				default:
					// if there is a transition that will propagate, set = to some positive #?
					break;
			}
		}
		__syncthreads();
		for (int i = 0; i < width; i++) {
			row[i] = rowResults[i];// * (tid+1);
		}
		free(rowResults);
	}
}

float gpuMarkPaths(ARRAY2D<int> results, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph,  int* fan) {
	loadPropLUTs();
#ifndef NTIMING
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
#endif // NTIMING
	kernMarkPathSegments<<<1,results.height>>>(results.data, dgraph.data, fan, results.width, results.height, dgraph.width);
	hipDeviceSynchronize();
#ifndef NTIMING
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}
float gpuMergeHistory(ARRAY2D<int> input, int** mergeresult, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, int* fan) {
	loadMergeLUTs();
	hipMalloc(mergeresult, sizeof(int)*input.height*input.width);
	// for bigger circuits or more patterns, need some logic here to divide work according to what will fit. 
#ifndef NTIMING
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
#endif // NTIMING
	kernMerge<<<input.height,input.width>>>(input.data, *mergeresult, input.width);
	hipDeviceSynchronize();
#ifndef NTIMING
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}
void runGpuSimulation(ARRAY2D<int> results, ARRAY2D<int> inputs, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, ARRAY2D<LINE> line, int* fan, int pass = 1) {
}

void debugMarkOutput(ARRAY2D<int> results) {
#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.
	int *lvalues, *row;
	DPRINT("Post-mark results\n");
	DPRINT("Line:   \t");
	for (int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (int r = 0;r < results.height; r++) {
		lvalues = (int*)malloc(results.bwidth());
		row = (int*)((char*)results.data + r*results.bwidth()); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		
		DPRINT("%s %d:\t","Vector",r);
		for (int i = 0; i < results.width; i++) {
			DPRINT("%2c ", lvalues[i] == 0 ? 'N':'S'  );
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif 
}
void debugUnionOutput(ARRAY2D<int> results) {
#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.
	int *lvalues, *row;
	DPRINT("Post-union results\n");
	DPRINT("Line:   \t");
	for (int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (int r = 0;r < results.height; r++) {
		lvalues = (int*)malloc(results.bwidth());
		row = (int*)((char*)results.data + r*results.bwidth()); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		
		DPRINT("%s %d:\t", "Vector",r);
		for (int i = 0; i < results.width; i++) {
			DPRINT("%2c ", lvalues[i] == 0 ? 'N':'S'  );
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif 
}
