#include "hip/hip_runtime.h"
#include "markkernel.h"
#include <hip/hip_runtime.h>
void HandleMarkError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleMarkError( err, __FILE__, __LINE__ ))
texture<int, 2> and2OutputPropLUT;
texture<int, 2> and2InputPropLUT;
texture<int, 2> or2OutputPropLUT;
texture<int, 2> or2InputPropLUT;
texture<int, 2> xor2OutputPropLUT;
texture<int, 2> xor2InputPropLUT;
texture<int, 2> fromPropLUT;
texture<int, 2> inptPropLUT;
texture<int, 2> mergeLUT;

texture<int, 2> AndInChainLUT;
texture<int, 2> AndOutChainLUT;
texture<int, 2> OrInChainLUT;
texture<int, 2> OrOutChainLUT;
texture<int, 2> XorInChainLUT;
texture<int, 2> XorOutChainLUT;
texture<char, 2> inputTexture;


void loadPropLUTs() {
	// Creating a set of static arrays that represent our LUTs
		// Addressing for the propagations:
	// 2 4x4 groups such that 
	int and2_output_prop[16]= {0,0,0,0,0,2,1,1,0,1,1,0,0,1,0,1};
	int and2_input_prop[16] = {0,0,0,0,0,0,1,1,0,0,1,0,0,0,0,1};
	int or2_output_prop[16] = {2,0,1,1,0,0,0,0,1,0,1,1,1,0,1,1};
	int or2_input_prop[16]  = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int xor2_input_prop[16] = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int xor2_output_prop[16]= {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	int from_prop[16]       = {0,0,0,0,0,0,0,0,0,0,1,1,0,0,1,1};
	int inpt_prop[8]        = {0,0,0,0,0,0,1,1};

	int and_outp_chain[8]   = {0,0,1,0,0,1,1,1};
	int and_inp_chain[8]    = {0,0,0,0,0,1,1,1};
	int or_outp_chain[8]    = {0,0,0,1,1,0,1,1};
	int or_inp_chain[8]     = {0,0,0,0,1,0,1,1};
	int xor_outp_chain[8]   = {0,0,0,0,0,0,0,0};
	int xor_inp_chain[8]    = {0,0,0,0,0,0,0,0};

	hipExtent volumeSize = make_hipExtent(4,4,2);
	// device memory arrays, required. 
	hipArray *cuAndInptProp, *cuAndOutpProp, *cuOrInptProp, *cuOrOutpProp, *cuFromProp, *cuInptProp, *cuXorInptProp, *cuXorOutpProp;
	hipArray *cuAndOutChain, *cuAndInChain, *cuOrInChain, *cuOrOutChain, *cuXorInChain, *cuXorOutChain;
	// generic formatting information. All of our arrays are the same, so sharing it shouldn't be a problem.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	// Allocating memory on the device.
		
	HANDLE_ERROR(hipMallocArray(&cuFromProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuInptProp, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuAndOutpProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuOrOutpProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorInptProp, &channelDesc, 4,4));
	HANDLE_ERROR(hipMallocArray(&cuXorOutpProp, &channelDesc, 4,4));

	HANDLE_ERROR(hipMallocArray(&cuXorOutChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuXorInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuOrInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuOrOutChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndInChain, &channelDesc, 4,2));
	HANDLE_ERROR(hipMallocArray(&cuAndOutChain, &channelDesc, 4,2));

	// Copying the LUTs Host->Device
	HANDLE_ERROR(hipMemcpyToArray(cuFromProp, 0,0, from_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndInptProp, 0,0, and2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndOutpProp, 0,0, and2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrInptProp, 0,0, or2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrOutpProp, 0,0, or2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorInptProp, 0,0, xor2_input_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorOutpProp, 0,0, xor2_output_prop, sizeof(int)*16,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuInptProp, 0,0, inpt_prop, sizeof(int)*8,hipMemcpyHostToDevice));
	
	HANDLE_ERROR(hipMemcpyToArray(cuXorInChain, 0,0, xor_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuXorOutChain, 0,0, xor_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrInChain, 0,0, or_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuOrOutChain, 0,0, or_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndInChain, 0,0, and_inp_chain, sizeof(int)*8,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToArray(cuAndOutChain, 0,0, and_outp_chain, sizeof(int)*8,hipMemcpyHostToDevice));

	// Marking them as textures. LUTs should be in texture memory and cached on
	// access.
	HANDLE_ERROR(hipBindTextureToArray(and2OutputPropLUT,cuAndOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(and2InputPropLUT,cuAndInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2OutputPropLUT,cuOrOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(or2InputPropLUT,cuOrInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2InputPropLUT,cuXorInptProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(xor2OutputPropLUT,cuXorOutpProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(fromPropLUT,cuFromProp,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(inptPropLUT,cuInptProp,channelDesc));
	
	HANDLE_ERROR(hipBindTextureToArray(XorOutChainLUT,cuXorOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(XorInChainLUT,cuXorInChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(OrOutChainLUT,cuOrOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(OrInChainLUT,cuOrInChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(AndOutChainLUT,cuAndOutChain,channelDesc));
	HANDLE_ERROR(hipBindTextureToArray(AndInChainLUT,cuAndInChain,channelDesc));
}
__device__ char markeval_out (char f1, char f2, int type) {
	char and2_output_prop[16]= {0,0,0,0,0,2,1,1,0,1,1,0,0,1,0,1};
	char or2_output_prop[16] = {2,0,1,1,0,0,0,0,1,0,1,1,1,0,1,1};
	char xor2_output_prop[16]= {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};

	switch(type) {
		case AND:
		case NAND:
			return REF2D(char,and2_output_prop,sizeof(char)*4,f1,f2);
		case OR:
		case NOR:
			return REF2D(char,or2_output_prop,sizeof(char)*4,f1,f2);
		case XOR:
		case XNOR:
			return REF2D(char,xor2_output_prop,sizeof(char)*4,f1,f2);
	}
	return 0xff;
}
__device__ char markeval_in (char f1, char f2, int type) {
	char and2_input_prop[16] = {0,0,0,0,0,0,1,1,0,0,1,0,0,0,0,1};
	char or2_input_prop[16]  = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	char xor2_input_prop[16] = {0,0,1,1,0,0,0,0,0,0,1,1,0,0,0,1};
	switch(type) {
		case AND:
		case NAND:
			return REF2D(char,and2_input_prop,sizeof(char)*4,f1,f2);
		case OR:
		case NOR:
			return REF2D(char,or2_input_prop,sizeof(char)*4,f1,f2);
		case XOR:
		case XNOR:
			return REF2D(char,xor2_input_prop,sizeof(char)*4,f1,f2);
	}
	return 0xff;
}

__global__ void kernMarkPathSegments(char *sim, size_t sim_pitch, char* mark, size_t pitch, size_t patterns, GPUNODE* node, int* fans, int start, int startPattern) {
	int tid = (blockIdx.y * blockDim.x) + threadIdx.x, nfi, goffset,val,prev;
	int gid = (blockIdx.x) + start;
	int pid = tid+startPattern;
	char rowCache, resultCache;
	char cache, fin = 1;
	int tmp = 1, pass = 0, fin1 = 0, fin2 = 0,type;
	if (pid < patterns) {
		cache = 0;
		rowCache = REF2D(char,sim,sim_pitch,tid,gid);
		resultCache = REF2D(char,mark,pitch,tid,gid);
		tmp = 1;
		nfi = node[gid].nfi;
		type = node[gid].type;
		goffset = node[gid].offset;
		__syncthreads();
		// switching based on value causes divergence, switch based on node type.
		// rowCache is from the simulation mark. 0-1, stable, 2-3, transition
		val = (rowCache > 1);

		if (node[gid].po > 0) {
			resultCache = val;
			prev = val;
		} else {
			prev = resultCache;
		}
		if (node[gid].nfo > 1) {
			prev = 0;
			resultCache = 0;
			for (int i = 0; i < node[gid].nfo; i++) {
				resultCache = (resultCache == 1) || (REF2D(char,mark,pitch,tid,FIN(fans,goffset,i+node[gid].nfi)) > 0);
			}
			prev = resultCache;
		}
		switch(type) {
			case FROM: break;
			case BUFF:
			case NOT:
				val = NOT_IN(rowCache) && prev;
				REF2D(char,mark,pitch,tid,FIN(fans,goffset,0)) = val;
				resultCache = val;
				break;
				// For the standard gates, setting three values -- both the
				// sim lines and the output line.  rowCache[threadIdx.x][i]-1 is the
				// transition on the output, offset to make the texture
				// calculations correct because there are 4 possible values
				// rowCache[threadIdx.x][i] can take: 0, 1, 2, 3.  0, 1 are the same, as are
				// 2,3, so we subtract 1 and clamp to an edge if we
				// overflow.
				// 0 becomes -1 (which clamps to 0)
				// 1 becomes 0
				// 2 becomes 1
				// 3 becomes 2 (which clamps to 1)
				// There's only 2 LUTs for each gate type. The sim LUT
				// checks for path existance through the first sim, so we
				// call it twice with the sims reversed to check both
				// paths.
			case OR:
			case NOR:
			case XOR:
			case XNOR:
			case NAND:
			case AND:
				for (fin1 = 0; fin1 < node[gid].nfi; fin1++) {
					fin = 1;
					for (fin2 = 0; fin2 < nfi; fin2++) {
						if (fin1 != fin2) {
							cache = markeval_out(REF2D(char,sim,sim_pitch,tid,FIN(fans,goffset,fin1)),REF2D(char,sim,sim_pitch,tid,FIN(fans,goffset,fin2)), type);
							pass += (cache > 1);
							tmp = tmp && (cache > 0);
							if (nfi > 1) {
								cache = markeval_in(REF2D(char,sim,sim_pitch,tid,FIN(fans,goffset,fin1)),REF2D(char,sim,sim_pitch,tid,FIN(fans,goffset,fin2)), type);
								fin = cache && fin && prev;
							}
						}
					}
					REF2D(char,mark,pitch,tid,FIN(fans,goffset,fin1)) = fin;
				}
				break;
			default: break;
		}
		// stick the contents of resultCache into the mark array
		REF2D(char,mark,pitch,tid,gid) = resultCache;


	}
}

float gpuMarkPaths(GPU_Data& results, GPU_Data& input, GPU_Circuit& ckt) {
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting before we hit this point.
	loadPropLUTs();
	int startGate;
	int blockcount_y;
#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	int startPattern = 0;
	for (unsigned int chunk = 0; chunk < input.size(); chunk++) {
		blockcount_y = (int)(results.gpu(chunk).width/MARK_BLOCK) + ((results.gpu(chunk).width% MARK_BLOCK) > 0);
		startGate=ckt.size()-1;
		DPRINT("Patterns to process in block %u: %lu\n", chunk, results.gpu(chunk).width);
		for (int i = ckt.levels(); i >= 0; i--) {
			int levelsize = ckt.levelsize(i);
			do { 
				int simblocks = min(MAX_BLOCKS, levelsize);
				dim3 numBlocks(simblocks,blockcount_y);
				startGate -= simblocks;
				kernMarkPathSegments<<<numBlocks,MARK_BLOCK>>>(input.gpu(chunk).data, input.gpu(chunk).pitch, results.gpu(chunk).data, results.gpu(chunk).pitch, results.gpu(chunk).width,ckt.gpu_graph(), ckt.offset(),  startGate+1, startPattern);
				if (levelsize > MAX_BLOCKS) {
					levelsize -= MAX_BLOCKS;
				} else {
					levelsize = 0;
				}
			} while (levelsize > 0);
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
		}
		startPattern += input.gpu(chunk).width;
	}
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
	return elapsed;
#else 
	return 0.0;
#endif // NTIMING
}


void debugMarkOutput(ARRAY2D<char> results, std::string outfile) {
#ifndef NDEBUG
	char *lvalues;
	std::ofstream ofile(outfile.c_str());
//	ofile << "Line:   \t";
//	for (unsigned int i = 0; i < results.height; i++) {
//		ofile << std::setw(OUTJUST) << i << " ";
//	}
//	ofile << std::endl;
	lvalues = (char*)malloc(results.height*results.pitch);
	hipMemcpy2D(lvalues,results.pitch,results.data,results.pitch,results.width,results.height,hipMemcpyDeviceToHost);
	for (unsigned int r = 0;r < results.width; r++) {
		ofile << "Vector " << r << ":\t";
		for (unsigned int i = 0; i < results.height; i++) {
			char z = REF2D(char, lvalues, results.pitch, r, i);
			switch(z) {
				case 0:
					ofile  << std::setw(OUTJUST) << "N" << " "; break;
				case 1:
					ofile  << std::setw(OUTJUST) << "Y" << " "; break;
				default:
					ofile << std::setw(OUTJUST) << (int)z << " "; break;
			}
		}
		ofile << std::endl;
	}
	free(lvalues);
	ofile.close();
#endif
}

