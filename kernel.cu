#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "iscas.h"
#include "kernel.h"
#include "defines.h"

texture<int, 2> and2LUT;
texture<int, 2> nand2LUT;
texture<int, 2> or2LUT;
texture<int, 2> nor2LUT;
texture<int, 2> xor2LUT;
texture<int, 2> xnor2LUT;
texture<int, 2> stableLUT;

__global__ void XOR_gate(int i, int* fans, GPUNODE* graph, int *res, int PATTERNS,size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(xor2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void XNOR_gate(int i, int* fans, GPUNODE* graph, int *res, int PATTERNS,size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(xnor2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void OR_gate(int i, int* fans, GPUNODE* graph, int *res, int PATTERNS,size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(or2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void NOR_gate(int i, int* fans, GPUNODE* graph, int *res, int PATTERNS,size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(nor2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void AND_gate(int i, int* fans, GPUNODE* graph, int *res, int PATTERNS,size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(and2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void NAND_gate(int i, int* fans, GPUNODE* graph, int *res, int PATTERNS, size_t width , int pass) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(nand2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		if (pass == 1) {
			row[fans[graph[i].offset+graph[i].nfi]] = val;
		} else {
			row[fans[graph[i].offset+graph[i].nfi]] = tex2D(stableLUT, row[fans[graph[i].offset+graph[i].nfi]], val);  
		}
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}

__global__ void FROM_gate(int i, int* fans,GPUNODE* graph, int *res, int PATTERNS, size_t width, int pass) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x;
	int *row;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*width*sizeof(int)); // get the current row?
		row[fans[graph[i].offset+graph[i].nfi]] = row[fans[graph[i].offset]];
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}

void loadLookupTables() {
	// Creating a set of static arrays that represent our LUTs
	int nand2[16] = {1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0};
	int and2[16]  = {0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1};
	int nor2[16]  = {1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0};
	int or2[16]   = {0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1};
	int xnor2[16] = {1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1};
	int xor2[16]  = {0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0};
	int stable[4] = {S0, T0, T1, S1};
	// device memory arrays, required. 
	hipArray* cuNandArray, *cuAndArray,*cuNorArray, *cuOrArray,*cuXnorArray,*cuXorArray;
	// generic formatting information. All of our arrays are the same, so sharing it shouldn't be a problem.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(sizeof(int)*8,0,0,0,hipChannelFormatKindUnsigned);
	
	// Allocating memory on the device.
	hipMallocArray(&cuNandArray, &channelDesc, 4,4);
	hipMallocArray(&cuAndArray, &channelDesc, 4,4);
	hipMallocArray(&cuNorArray, &channelDesc, 4,4);
	hipMallocArray(&cuOrArray, &channelDesc, 4,4);
	hipMallocArray(&cuXnorArray, &channelDesc, 4,4);
	hipMallocArray(&cuXorArray, &channelDesc, 4,4);
	hipMallocArray(&cuStableArray, &channelDesc, 2,2);

	// Copying the LUTs Host->Device
	hipMemcpyToArray(cuNandArray, 0,0, nand2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuAndArray, 0,0, and2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuNorArray, 0,0, nor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuOrArray, 0,0, or2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXnorArray, 0,0, xnor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXorArray, 0,0, xor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuStableArray, 0,0, stable, sizeof(int)*4,hipMemcpyHostToDevice);

	// Marking them as textures. LUTs should be in texture memory and cached on
	// access.
	hipBindTextureToArray(and2LUT,cuAndArray,channelDesc);
	hipBindTextureToArray(nand2LUT,cuNandArray,channelDesc);
	hipBindTextureToArray(or2LUT,cuOrArray,channelDesc);
	hipBindTextureToArray(nor2LUT,cuNorArray,channelDesc);
	hipBindTextureToArray(xor2LUT,cuXorArray,channelDesc);
	hipBindTextureToArray(xnor2LUT,cuXnorArray,channelDesc);
	hipBindTextureToArray(stableLUT,cuStableArray,channelDesc);
}
void runGpuSimulation(int* results,  int PATTERNS, size_t width, GPUNODE* ggraph, GPUNODE* graph, int maxid, LINE* line, int maxline, int* fan, int pass) {

	for (int i = 0; i <= maxid; i++) {
		DPRINT("ID: %d\tFanin: %d\tFanout: %d\tType: %d\t", i, graph[i].nfi, graph[i].nfo,graph[i].type);
		switch (graph[i].type) {
			case 0:
				continue;
			case XNOR:
				DPRINT("XNOR Gate");
				XNOR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, PATTERNS, width);
			case XOR:
				DPRINT("XOR Gate");
				XOR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, PATTERNS, width);
			case NOR:
				DPRINT("NOR Gate");
				NOR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, PATTERNS, width);
			case OR:
				DPRINT("OR Gate");
				OR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, PATTERNS, width);
			case AND:
				DPRINT("AND Gate");
				AND_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, PATTERNS, width);
			case NAND:
				DPRINT("NAND Gate");
				NAND_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, PATTERNS, width, 1);
				break;
			case FROM:
				DPRINT("FROM Gate");
				FROM_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, PATTERNS, width, 1);
				break;
			default:
				DPRINT("Other Gate");
				break;
		}
		DPRINT("\n");
		hipDeviceSynchronize();
	}

#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.

	DPRINT("Post-simulation device results:\n");
	int *lvalues = (int*)malloc(sizeof(int)*width), *row;
	for (int r = 0;r < PATTERNS; r++) {
		lvalues = (int*)malloc(sizeof(int)*width);
		row = (int*)((char*)results + r*width*sizeof(int)); // get the current row?
		hipMemcpy(lvalues,row,width*sizeof(int),hipMemcpyDeviceToHost);
		for (int i = 0; i < width; i++) {
			DPRINT("%d,%d:\t%d\n", r, i, lvalues[i]);
		}
		free(lvalues);
	}
#endif 
}
