#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "iscas.h"
#include "kernel.h"
#define N 32
#define PATTERNS 2

__global__ void NAND_gate(int i, int* fans, GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x;
	int *row;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*width*sizeof(int));
		row[fans[graph[i].offset+graph[i].nfi]] = row[fans[graph[i].offset]];
	}
}

__global__ void FROM_gate(int i, int* fans,GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x;
	int *row;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*width*sizeof(int)); // get the current row?
		row[fans[graph[i].offset+graph[i].nfi]] = row[fans[graph[i].offset]];
	}
}

void runGpuSimulation(int* results, size_t width, GPUNODE* ggraph, GPUNODE* graph, int maxid, LINE* line, int maxline, int* fan) {
	printf("Pre-simulation device memory check:\n");
	int *lvalues1 = (int*)malloc(sizeof(int)*width*2);
	hipMemcpy(lvalues1,results,width*2*sizeof(int),hipMemcpyDeviceToHost);
	for (int i = 0; i < 2*width; i++) {
		printf("%d:\t%d\n", i, lvalues1[i]);
	}

	for (int i = 0; i < maxid; i++) {
		printf("ID: %d, Type: %d\t", i, graph[i].type);
		switch (graph[i].type) {
			case NAND:
				printf("NAND Gate \n");
				NAND_gate<<<1,2>>>(i, fan, ggraph, results, width);
				break;
			case FROM:
				printf("FROM Gate \n");
				FROM_gate<<<1,2>>>(i, fan, ggraph, results, width);
				break;
			default:
				printf("Other Gate\n");
				break;
		}
		hipDeviceSynchronize();
	}
	printf("Post-simulation device memory check:\n");
	lvalues1 = (int*)malloc(sizeof(int)*width*2);
	hipMemcpy(lvalues1,results,width*2*sizeof(int),hipMemcpyDeviceToHost);
	for (int i = 0; i < 2*width; i++) {
		printf("%d:\t%d\n", i, lvalues1[i]);
	}
}
