#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "iscas.h"
#include "kernel.h"
#include "defines.h"

texture<int, 2> and2LUT;
texture<int, 2> nand2LUT;
texture<int, 2> or2LUT;
texture<int, 2> nor2LUT;
texture<int, 2> xor2LUT;
texture<int, 2> xnor2LUT;
texture<int, 2> stableLUT;
texture<int, 3> and2OutputPropLUT;
texture<int, 3> and2InputPropLUT;
texture<int, 3> or2OutputPropLUT;
texture<int, 3> or2InputPropLUT;
texture<int, 2> fromPropLUT;
texture<int, 2> inptPropLUT;

__global__ void INPT_gate(int i, int pi, ARRAY2D<int> results, ARRAY2D<int> input, GPUNODE* graph, int* fans,int pass) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, val;
	int *row;
	if (tid < results.height) {
		row = (int*)((char*)results.data + tid*results.width*sizeof(int)); // get the current row?
		val = *(input.data+(pi+input.width*tid));
		if (pass > 1) {
			row[fans[graph[i].offset+graph[i].nfi]] = tex2D(stableLUT, row[fans[graph[i].offset+graph[i].nfi]], val);  
		} else {
			row[fans[graph[i].offset+graph[i].nfi]] = val;
		}
	}
#ifdef GDEBUG // turn on GPU debugging printf statements.
	printf("Hello thread %d, i=%d, input count: %d/%d input value=%d\n", threadIdx.x, i,pi+1,input.width, input.data[pi]) ;
#endif
	__syncthreads();
}

void loadLookupTables() {
	// Creating a set of static arrays that represent our LUTs
	int nand2[16] = {1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0};
	int and2[16]  = {0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1};
	int nor2[16]  = {1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0};
	int or2[16]   = {0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1};
	int xnor2[16] = {1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1};
	int xor2[16]  = {0, 1, 0, 1, 1, 0, 1, 0, 0, 1, 0, 1, 1, 0, 1, 0};
	int stable[4] = {S0, T0, T1, S1};
	// Addressing for the propagations:
	// 2 4x4 groups such that 
	int and2_output_prop[32] ={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0, 1,0,0,0, 1, 1, 1, 1, 1, 1,0, 1, 1, 1};
	int and2_input_prop[32] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0, 1, 1,0,0, 1,0,0,0, 1, 1};
	int or2_output_prop[32] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0, 1, 1,0,0,0,0, 1,0, 1, 1, 1,0, 1, 1};
	int or2_input_prop[32] =  {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0, 1, 1,0,0,0,0,0,0, 1, 1,0,0,0, 1};
	int from_prop[16]      =  {0,0,0,0,0,0,0,0,0,0, 1, 1,0,0, 1, 1};
	int inpt_prop[8] = {0,0,0,0,0,0,1,1};

	hipExtent volumeSize = make_hipExtent(4,4,2);
	// device memory arrays, required. 
	hipArray *cuNandArray, *cuAndArray,*cuNorArray, *cuOrArray,*cuXnorArray,*cuXorArray, *cuStableArray;
	hipArray *cuAndInptProp, *cuAndOutpProp, *cuOrInptProp, *cuOrOutpProp, *cuFromProp, *cuInptProp;
	// generic formatting information. All of our arrays are the same, so sharing it shouldn't be a problem.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	DPRINT("%d,%d", volumeSize.height, volumeSize.width);
	
	// Allocating memory on the device.
	hipMallocArray(&cuNandArray, &channelDesc, 4,4);
	hipMallocArray(&cuAndArray, &channelDesc, 4,4);
	hipMallocArray(&cuNorArray, &channelDesc, 4,4);
	hipMallocArray(&cuOrArray, &channelDesc, 4,4);
	hipMallocArray(&cuXnorArray, &channelDesc, 4,4);
	hipMallocArray(&cuXorArray, &channelDesc, 4,4);
	hipMallocArray(&cuStableArray, &channelDesc, 2,2);
	
	hipMallocArray(&cuFromProp, &channelDesc, 4,4);
	hipMallocArray(&cuInptProp, &channelDesc, 4,2);
	hipMalloc3DArray(&cuAndInptProp, &channelDesc, volumeSize);
	hipMalloc3DArray(&cuAndOutpProp, &channelDesc, volumeSize);
	hipMalloc3DArray(&cuOrInptProp, &channelDesc, volumeSize);
	hipMalloc3DArray(&cuOrOutpProp, &channelDesc, volumeSize);

	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr = make_hipPitchedPtr((void*)and2_output_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuAndOutpProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	and2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	copyParams.srcPtr = make_hipPitchedPtr((void*)and2_input_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuAndInptProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	and2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	and2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	copyParams.srcPtr = make_hipPitchedPtr((void*)or2_output_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuOrOutpProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	or2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	copyParams.srcPtr = make_hipPitchedPtr((void*)or2_input_prop, volumeSize.width*sizeof(int), volumeSize.width, volumeSize.height);
	copyParams.dstArray = cuOrInptProp;
	copyParams.extent =  volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	or2OutputPropLUT.addressMode[2]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[0]=hipAddressModeClamp;
	or2OutputPropLUT.addressMode[1]=hipAddressModeClamp;

	// Copying the LUTs Host->Device
	hipMemcpyToArray(cuNandArray, 0,0, nand2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuAndArray, 0,0, and2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuNorArray, 0,0, nor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuOrArray, 0,0, or2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXnorArray, 0,0, xnor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXorArray, 0,0, xor2, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuFromProp, 0,0, from_prop, sizeof(int)*16,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuInptProp, 0,0, inpt_prop, sizeof(int)*8,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuStableArray, 0,0, stable, sizeof(int)*4,hipMemcpyHostToDevice);

	// Marking them as textures. LUTs should be in texture memory and cached on
	// access.
	hipBindTextureToArray(and2LUT,cuAndArray,channelDesc);
	hipBindTextureToArray(nand2LUT,cuNandArray,channelDesc);
	hipBindTextureToArray(or2LUT,cuOrArray,channelDesc);
	hipBindTextureToArray(nor2LUT,cuNorArray,channelDesc);
	hipBindTextureToArray(xor2LUT,cuXorArray,channelDesc);
	hipBindTextureToArray(xnor2LUT,cuXnorArray,channelDesc);
	hipBindTextureToArray(stableLUT,cuStableArray,channelDesc);
	hipBindTextureToArray(and2OutputPropLUT,cuAndOutpProp,channelDesc);
	hipBindTextureToArray(and2InputPropLUT,cuAndInptProp,channelDesc);
	hipBindTextureToArray(or2OutputPropLUT,cuOrOutpProp,channelDesc);
	hipBindTextureToArray(or2InputPropLUT,cuOrInptProp,channelDesc);
	hipBindTextureToArray(fromPropLUT,cuFromProp,channelDesc);
	hipBindTextureToArray(inptPropLUT,cuInptProp,channelDesc);
}

__global__ void LOGIC_gate(int i, GPUNODE* node, int* fans, int* res, size_t height, size_t width , int pass) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int goffset,nfi;
	int val;
	if (tid < height) {
		goffset = node[i].offset;
		nfi = node[i].nfi;
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[goffset]];
		while (j < nfi) {
			switch(node[i].type) {
				case XOR:
					val = tex2D(xor2LUT, val, row[fans[goffset+j]]);
				case XNOR:
					val = tex2D(xnor2LUT, val, row[fans[goffset+j]]);
				case OR:
					val = tex2D(or2LUT, val, row[fans[goffset+j]]);
				case NOR:
					val = tex2D(nor2LUT, val, row[fans[goffset+j]]);
				case AND:
					val = tex2D(and2LUT, val, row[fans[goffset+j]]);
				case NAND:
					val = tex2D(nand2LUT, val, row[fans[goffset+j]]);
			}
			j++;
		}
		if (pass > 1 && node[i].type != FROM) {
			row[fans[goffset+nfi]] = tex2D(stableLUT, row[fans[goffset+nfi]], val);  
		} else {
			row[fans[goffset+nfi]] = val;
		}
	}
	__syncthreads();
}
__device__ int willPathPropagate(int tid, int* results, GPUNODE* node, int* fans, size_t width) {
	return -1;
}
__global__ void gpuMarkPathSegments(int *results, GPUNODE* node, int* fans, size_t width, size_t height, int ncount) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, nfi, goffset,val;
	int *rowResults, *row;
	if (tid < height) {
		row = (int*)((char*)results + tid*(width)*sizeof(int));
		rowResults = (int*)malloc(sizeof(int)*width);
		for (int i = 0; i < width; i++) {
			rowResults[i] = UNINITIALIZED;
		}
		for (int i = ncount; i >= 0; i--) {
			val = UNINITIALIZED;
			goffset = node[i].offset;
			nfi = node[i].nfi;
			// switching based on value causes divergence, switch based on node type.
			switch(node[i].type) {
				case NAND:
					rowResults[fans[goffset]] = tex3D(and2InputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1);
					rowResults[fans[goffset+1]] = tex3D(and2InputPropLUT, row[fans[goffset+1]],row[fans[goffset]],row[fans[goffset+nfi]]-1) ;
					rowResults[fans[goffset+nfi]] = tex3D(and2OutputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1) ;
					break;
				case FROM:
					if (rowResults[fans[goffset]] == UNINITIALIZED) {
						val = tex2D(inptPropLUT, row[fans[goffset]],rowResults[fans[goffset+nfi]]);
						rowResults[fans[goffset]] = val;
						rowResults[fans[goffset+nfi]] = val;
					} else {
						val = tex2D(inptPropLUT, row[fans[goffset]],rowResults[fans[goffset+nfi]]);
						rowResults[fans[goffset+nfi]] = val;
					}
					break;
				case AND:
					rowResults[fans[goffset]] = tex3D(and2InputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1);
					rowResults[fans[goffset+1]] = tex3D(and2InputPropLUT, row[fans[goffset+1]],row[fans[goffset]],row[fans[goffset+nfi]]-1);
					rowResults[fans[goffset+nfi]] = tex3D(and2OutputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1);
					break;
				case OR:
					rowResults[fans[goffset]] = tex3D(or2InputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1) ;
					rowResults[fans[goffset+1]] = tex3D(or2InputPropLUT, row[fans[goffset+1]],row[fans[goffset]],row[fans[goffset+nfi]]-1) ;
					rowResults[fans[goffset+nfi]] = tex3D(or2OutputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1) ;
					break;
				case NOR:
					rowResults[fans[goffset]] = tex3D(or2InputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1) ;
					rowResults[fans[goffset+1]] = tex3D(or2InputPropLUT, row[fans[goffset+1]],row[fans[goffset]],row[fans[goffset+nfi]]-1) ;
					rowResults[fans[goffset+nfi]] = tex3D(or2OutputPropLUT, row[fans[goffset]],row[fans[goffset+1]],row[fans[goffset+nfi]]-1) ;
					break;
				case XOR:
				case XNOR:
				default:
					// if there is a transition that will propagate, set = to some positive #?
					break;
			}
		}
		__syncthreads();
		for (int i = 0; i < width; i++) {
			row[i] = rowResults[i] * (tid+1);
		}
		free(rowResults);
	}
}
void runGpuSimulation(ARRAY2D<int> results, ARRAY2D<int> inputs, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, ARRAY2D<LINE> line, int* fan, int pass = 1) {
	// Allocate a buffer memory for printf statements
	int piNumber = 0, curPI = 0;
	DPRINT("Pattern Count: %d\n", results.height );
#ifndef NDEBUG
	int eventCount = 0;
	hipEvent_t* eventQueue = (hipEvent_t*)malloc(sizeof(hipEvent_t)*10);
	hipEventCreate(eventQueue);
	hipEventCreate(eventQueue+1);
	eventCount++;
	hipEventRecord(eventQueue[0],0);
#endif // NDEBUG
	for (int i = 0; i <= dgraph.width; i++) {
		DPRINT("ID: %d\tFanin: %d\tFanout: %d\tType: %d\t", i, graph[i].nfi, graph[i].nfo,graph[i].type);
		curPI = piNumber;
		switch (graph[i].type) {
			case 0:
				continue;
			case INPT:
				DPRINT("INPT Gate");
				INPT_gate<<<1,results.height>>>(i, curPI, results, inputs, dgraph.data, fan, pass);
				piNumber++;
				break;
			default:
				LOGIC_gate<<<1,results.height>>>(i, dgraph.data, fan, results.data, results.height, results.width, pass);
				break;
		}
		DPRINT("\n");
		hipDeviceSynchronize();
	}
#ifndef NDEBUG
	hipEventRecord(eventQueue[1], 0);
	hipEventSynchronize(eventQueue[1]);

	DPRINT("Post-simulation device results, pass %d:\n\n", pass);
	DPRINT("Line:   \t");
	for (int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	int *lvalues, *row;
	for (int r = 0;r < results.height; r++) {
		lvalues = (int*)malloc(results.bwidth());
		row = (int*)((char*)results.data + r*results.bwidth()); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		
		DPRINT("%s %d:\t", pass > 1 ? "Vector" : "Pattern",r);
		for (int i = 0; i < results.width; i++) {
				switch(lvalues[i]) {
					case S0:
						DPRINT("S0 "); break;
					case S1:
						DPRINT("S1 "); break;
					case T0:
						DPRINT("T0 "); break;
					case T1:
						DPRINT("T1 "); break;
					default:
						DPRINT("%2d ", lvalues[i]); break;
				}

		}
		DPRINT("\n");
		free(lvalues);
	}

	hipEventCreate(eventQueue+2);
	hipEventCreate(eventQueue+3);
	eventCount++;
	hipEventRecord(eventQueue[2],0);
#endif
	if (pass > 1) {
		gpuMarkPathSegments<<<1,results.height>>>(results.data, dgraph.data, fan, results.width, results.height, dgraph.width);
		hipDeviceSynchronize();
	}
#ifndef NDEBUG
	hipEventRecord(eventQueue[3],0);
	hipEventSynchronize(eventQueue[3]);
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.

	DPRINT("Post-simulation device results, pass %d:\n\n", pass);
	DPRINT("Line:   \t");
	for (int i = 0; i < results.width; i++) {
		DPRINT("%2d ", i);
	}
	DPRINT("\n");
	for (int r = 0;r < results.height; r++) {
		lvalues = (int*)malloc(results.bwidth());
		row = (int*)((char*)results.data + r*results.bwidth()); // get the current row?
		hipMemcpy(lvalues,row,results.bwidth(),hipMemcpyDeviceToHost);
		
		DPRINT("%s %d:\t", pass > 1 ? "Vector" : "Pattern",r);
		for (int i = 0; i < results.width; i++) {
			if (pass > 1) {
				DPRINT("%2c ", lvalues[i] == 0 ? 'N':'S'  );
			} else {
				switch(lvalues[i]) {
					case S0:
						DPRINT("S0 "); break;
					case S1:
						DPRINT("S1 "); break;
					case T0:
						DPRINT("T0 "); break;
					case T1:
						DPRINT("T1 "); break;
					default:
						DPRINT("%2d ", lvalues[i]); break;
				}
			}

		}
		DPRINT("\n");
		free(lvalues);
	}
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,eventQueue[0],eventQueue[1]);
	DPRINT("Simulation time (pass %d): %fms\n", pass, elapsedTime);
	if (pass > 1) {
		hipEventElapsedTime(&elapsedTime,eventQueue[2],eventQueue[3]);
		DPRINT("Path Marking time (pass %d): %fms\n", pass, elapsedTime);
	}
	for (int f = 0; f < eventCount*2; f+=2) {
		hipEventDestroy(eventQueue[f]);
		hipEventDestroy(eventQueue[f+1]);
	}
	free(eventQueue);
#endif 
}

