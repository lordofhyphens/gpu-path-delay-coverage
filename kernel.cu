#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "iscas.h"
#include "kernel.h"
#include "defines.h"

texture<int, 2> and2LUT;
texture<int, 2> nand2LUT;
texture<int, 2> or2LUT;
texture<int, 2> nor2LUT;
texture<int, 2> xor2LUT;
texture<int, 2> xnor2LUT;

__global__ void XOR_gate(int i, int* fans, GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(xor2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void XNOR_gate(int i, int* fans, GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(xnor2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void OR_gate(int i, int* fans, GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(or2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void NOR_gate(int i, int* fans, GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(nor2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void AND_gate(int i, int* fans, GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(and2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}
__global__ void NAND_gate(int i, int* fans, GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, j = 1;
	int *row;
	int val;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*(width)*sizeof(int));
		val = row[fans[graph[i].offset]];
		while (j < graph[i].nfi) {
			val = tex2D(nand2LUT, val, row[fans[graph[i].offset+j]]);
			j++;
		}
		row[fans[graph[i].offset+graph[i].nfi]] = val;
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}

__global__ void FROM_gate(int i, int* fans,GPUNODE* graph, int *res, size_t width) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x;
	int *row;
	while (tid < PATTERNS) {
		row = (int*)((char*)res + tid*width*sizeof(int)); // get the current row?
		row[fans[graph[i].offset+graph[i].nfi]] = row[fans[graph[i].offset]];
		tid += blockDim.x * gridDim.x;
		__syncthreads();
	}
}

void loadLookupTables() {
	// Creating a set of static arrays that represent our LUTs
	int nand2[4] = {1, 1, 1, 0};
	int and2[4] = {0, 0, 0, 1};
	int nor2[4] = {1, 0, 0, 0};
	int or2[4] = {0,1,1,1};
	int xnor2[4] = {1,0,0,1};
	int xor2[4] = {0,1,1,0};
	// device memory arrays, required. 
	hipArray* cuNandArray, *cuAndArray,*cuNorArray, *cuOrArray,*cuXnorArray,*cuXorArray;
	// generic formatting information. All of our arrays are the same, so sharing it shouldn't be a problem.
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(sizeof(int)*8,0,0,0,hipChannelFormatKindUnsigned);
	
	// Allocating memory on the device.
	hipMallocArray(&cuNandArray, &channelDesc, 2,2);
	hipMallocArray(&cuAndArray, &channelDesc, 2,2);
	hipMallocArray(&cuNorArray, &channelDesc, 2,2);
	hipMallocArray(&cuOrArray, &channelDesc, 2,2);
	hipMallocArray(&cuXnorArray, &channelDesc, 2,2);
	hipMallocArray(&cuXorArray, &channelDesc, 2,2);

	// Copying the LUTs Host->Device
	hipMemcpyToArray(cuNandArray, 0,0, nand2, sizeof(int)*4,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuAndArray, 0,0, and2, sizeof(int)*4,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuNorArray, 0,0, nor2, sizeof(int)*4,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuOrArray, 0,0, or2, sizeof(int)*4,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXnorArray, 0,0, xnor2, sizeof(int)*4,hipMemcpyHostToDevice);
	hipMemcpyToArray(cuXorArray, 0,0, xor2, sizeof(int)*4,hipMemcpyHostToDevice);

	// Marking them as textures. LUTs should be in texture memory and cached on
	// access.
	hipBindTextureToArray(and2LUT,cuAndArray,channelDesc);
	hipBindTextureToArray(nand2LUT,cuNandArray,channelDesc);
	hipBindTextureToArray(or2LUT,cuOrArray,channelDesc);
	hipBindTextureToArray(nor2LUT,cuNorArray,channelDesc);
	hipBindTextureToArray(xor2LUT,cuXorArray,channelDesc);
	hipBindTextureToArray(xnor2LUT,cuXnorArray,channelDesc);
}
void runGpuSimulation(int* results, size_t width, GPUNODE* ggraph, GPUNODE* graph, int maxid, LINE* line, int maxline, int* fan) {

	for (int i = 0; i <= maxid; i++) {
		printf("ID: %d\tFanin: %d\tFanout: %d\tType: %d\t", i, graph[i].nfi, graph[i].nfo,graph[i].type);
		switch (graph[i].type) {
			case 0:
				continue;
			case XNOR:
				printf("XNOR Gate");
				XNOR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, width);
			case XOR:
				printf("XOR Gate");
				XOR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, width);
			case NOR:
				printf("NOR Gate");
				NOR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, width);
			case OR:
				printf("OR Gate");
				OR_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, width);
			case AND:
				printf("AND Gate");
				AND_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, width);
			case NAND:
				printf("NAND Gate");
				NAND_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, width);
				break;
			case FROM:
				printf("FROM Gate");
				FROM_gate<<<1,PATTERNS>>>(i, fan, ggraph, results, width);
				break;
			default:
				printf("Other Gate");
				break;
		}
		printf("\n");
		hipDeviceSynchronize();
	}

	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.

	printf("Post-simulation device results:\n");
	int *lvalues = (int*)malloc(sizeof(int)*width), *row;
	for (int r = 0;r < PATTERNS; r++) {
		lvalues = (int*)malloc(sizeof(int)*width);
		row = (int*)((char*)results + r*width*sizeof(int)); // get the current row?
		hipMemcpy(lvalues,row,width*sizeof(int),hipMemcpyDeviceToHost);
		for (int i = 0; i < width; i++) {
			printf("%d,%d:\t%d\n", r, i, lvalues[i]);
		}
		free(lvalues);
	}
}
