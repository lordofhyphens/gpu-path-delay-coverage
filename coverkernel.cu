#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "defines.h"
#include "coverkernel.h"

#define THREAD_PER_BLOCK 256
// badly sums everything and places it into row[0][0]
__global__ void kernSumAll(int toffset, char *results, char *history, GPUNODE* node, int* fans, size_t width, size_t height, size_t pitch, int ncount) {
	int tid = blockIdx.x * gridDim.x + threadIdx.x, nfi, goffset;
	char *row;
	__shared__ int sum;
	if (tid < 1) {
		sum = 0;
		for (int j = 0; j < height; j++) {
			row = (char*)((char*)results + j*(pitch));
			for (int c = ncount-1; c >= 0; c--) {
				goffset = node[c].offset;
				nfi = node[c].nfi;
				if (node[c].type == INPT)
					sum = sum + row[fans[goffset+nfi]];
				//printf("Sum Count: %d\n",sum);
			}
		}
		row = ((char*)results);
		row[0] = sum;
	}
}

// reference: design book 1, page 38.
__global__ void kernCountCoverage(GPUNODE* graph, char* result_array, size_t result_pitch, char* history_array, size_t history_pitch, size_t result_width, size_t pattern_count, int* fanout_index, size_t width, size_t start_gate) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x, nfi, goffset;
	int gid = blockIdx.x + start_gate;
	char *row, *historyRow;
	int *current, *historyCount;
	__shared__ int rowids[50]; // handle up to fanins of 50 /
	if (tid < pattern_count) {
		goffset = node[i].offset;
		row = ((char*)results + tid*pitch);
		if (tid == 0) {
			historyRow = (char*)malloc(sizeof(char)*width);
			memset(historyRow, 0, sizeof(char)*width);
		} else {
			historyRow = ((char*)history + (tid-1)*hpitch);
		}
		current = (int*)malloc(sizeof(int)*width);
		historyCount = (int*)malloc(sizeof(int)*width);
			nfi = node[i].nfi;
			if (tid == 0) {
				// preload all of the fanin line #s for this gate to shared memory.
				// Guaranteed 1 cycle access time.
				for (int j = 0; j < nfi;j++) {
					rowids[j] = fans[goffset+j];
				}
			}
			__syncthreads();
			if (node[i].po) {
				current[i] = (row[i] > historyRow[i]); // only set = 1 if there's a new line here
				historyCount[i] = historyRow[i];
			}
			switch(node[i].type) {
				case 0: continue;
				case FROM:
						// Add the current fanout count to the fanin if this line is marked (and the history isn't).
						current[rowids[0]] += current[i]*(row[rowids[0]] > historyRow[rowids[0]]);
						historyCount[rowids[0]] += historyCount[i]*(historyRow[rowids[0]]);
						break;
				case INPT:
						continue;
				default: 
						for (int fin = 0; fin < node[i].nfi; fin++) {
							// if the fanout total is 0 but this line is marked (and the history isn't), add a path to the count.
							// If the fanout total is > 1 and this line is marked (and the history isn't), assign the fanout total to the fanins.
							historyCount[rowids[fin]] += (historyRow[rowids[fin]] || historyCount[i] > 1) * historyCount[i];
							current[rowids[fin]] += ((row[rowids[fin]] > historyRow[rowids[fin]]) || current[i] > 1) * current[i] + historyCount[i]*(current[i] == 0 && row[rowids[fin]] > historyRow[rowids[fin]]);
						}

			}
		for (int i = 0; i < ncount; i++) {
			row[i] = current[i];
		}
		free(current);
		free(historyCount);
		if (tid == 0) {
			free(historyRow);
		}
	}

}
void debugCoverOutput(ARRAY2D<char> results) {
#ifndef NDEBUG
	// Routine to copy contents of our results array into host memory and print
	// it row-by-row.
	char *lvalues, *row;
	DPRINT("Path Count results\n");
	DPRINT("Line:   \t");
	for (unsigned int i = 0; i < results.width; i++) {
		DPRINT("%3d ", i);
	}
	DPRINT("\n");
	for (unsigned int r = 0;r < results.height; r++) {
		lvalues = (char*)malloc(results.pitch);
		row = ((char*)results.data + r*results.bwidth()); // get the current row?
		hipMemcpy(lvalues,row,results.pitch,hipMemcpyDeviceToHost);
		
		DPRINT("%s %3d:\t", "Vector",r);
		for (unsigned int i = 0; i < results.width; i++) {
			DPRINT("%3d ", lvalues[i] == 0 ? 255:lvalues[i]);
		}
		DPRINT("\n");
		free(lvalues);
	}
#endif 
}
float gpuCountPaths(ARRAY2D<char> results, ARRAY2D<char> history, GPUNODE* graph, ARRAY2D<GPUNODE> dgraph, int* fan) {
	int *gatesinLevel, startGate=0;
	gatesinLevel = new int[maxlevels];
	for (int i = 0; i < maxlevels; i++) {
		gatesinLevel[i] = 0;
		for (unsigned int j = 0; j < results.width; j++) {
			if (graph[j].level == i) {
				gatesinLevel[i]++;
			}
		}
		startGate += gatesinLevel[i];
	}
	int blockcount_y = (int)(input.height/THREAD_PER_BLOCK) + (input.height%THREAD_PER_BLOCK > 0);

#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif // NTIMING
	for (int i = maxlevels-1; i >= 0; i--) {
		dim3 numBlocks(gatesinLevel[i],blockcount_y);
		startGate -= gatesinLevel[i];
		kernCountCoverage<<<numBlocks,THREAD_PER_BLOCK>>>(input.data, results.data, dgraph.data, fan, results.width, results.height, startGate, results.pitch);
		hipDeviceSynchronize();
	}
	delete gatesinLevel;
//	kernSumAll<<<1,1>>>(0, results.data, history.data,dgraph.data, fan, results.width, results.height, results.pitch,dgraph.width);
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start,stop));
	return elapsed;
#else
	return 0.0;
#endif
}
char returnPathCount(ARRAY2D<char> results) {
	char tmp;
	hipMemcpy(&tmp, results.data, sizeof(char), hipMemcpyDeviceToHost);
	return tmp;
}
