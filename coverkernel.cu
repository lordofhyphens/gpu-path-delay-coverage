#include "hip/hip_runtime.h"
#include "coverkernel.h"
#include <hip/hip_runtime.h> 
void HandleCoverError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}


#define HANDLE_ERROR( err ) (HandleCoverError( err, __FILE__, __LINE__ ))
__global__ void kernCover(const GPUNODE* ckt, char* mark,size_t mark_pitch, int* history,  int* cover,size_t cover_pitch, int* hist_cover, size_t hcover_pitch,int start_offset, int start_pattern, int pattern_count, int* offsets) {
    // cover is the coverage ints we're working with for this pass.
    // mark is the fresh marks
    // hist is the history of the mark status of all lines.
	int tid = (blockIdx.y * COVER_BLOCK) + threadIdx.x;
	int pid = tid + start_pattern; 
	int g = blockIdx.x+start_offset;
	int resultCache = 0;
	int histCache = 0;
	char cache;
	GPUNODE gate = ckt[g];
	if (pid < pattern_count) {
		cache = REF2D(char,mark,mark_pitch,tid, g); // cache the current node's marked status.
		// shorthand references to current coverage and history count.
		int *c, *h;
		c = ADDR2D(int, cover     , cover_pitch , tid, g);
		h = ADDR2D(int, hist_cover, hcover_pitch, tid, g);

		if (gate.po == 1) {
			*c = 0;
            *h = (cache > 0); // set history = 1 if this line is marked.
        }

		if (gate.nfo > 1) {
			for (int i = 0; i < gate.nfo; i++) {
				int fot = FIN(offsets,gate.offset+gate.nfi,i); // reference to current fan-out
				resultCache += REF2D(int,cover,cover_pitch,tid,fot); // add this fanout's path count to this node.
				histCache += REF2D(int,hist_cover,hcover_pitch,tid,fot); // add this fanout's history path count to this node.
			}
			*c = resultCache;
			*h = histCache;
		}
		if (gate.type != FROM) {
			*c = (*c + *h)*(cache > 0)*(history[g] >= pid);
			*h *= ((cache > 0)*(history[g] >= pid) == 0);

            for (int i = 0; i < gate.nfi; i++) {
				int fin = FIN(offsets,gate.offset,i);
				REF2D(int,cover,cover_pitch,tid,fin) = *c; //REF2D(int,cover,cover_pitch,tid,g);
				REF2D(int,hist_cover,hcover_pitch,tid,fin) = *h; //REF2D(int,hist_cover,hcover_pitch,tid,g);
			}
        } 
	}
}


float gpuCountPaths(const GPU_Circuit& ckt, GPU_Data& mark, ARRAY2D<int> merges, long unsigned int* coverage) {
	int* results, *g_results, *gh_results;
	long* finalcoverage;
	*coverage = 0;
	int startGate=ckt.size()-1;
	size_t pitch, h_pitch;
	int startPattern = 0;
	hipMalloc(&finalcoverage, sizeof(long));
	hipMallocPitch(&g_results,&pitch, sizeof(int)*mark.block_width(),mark.height()); // using pinned memory because we're laaaazy.
	hipMallocPitch(&gh_results,&h_pitch,sizeof(int)*mark.block_width(),mark.height());
	results = (int*)malloc(mark.block_width()*sizeof(int)*mark.height());
//	h_results = (int*)malloc(mark.block_width()*sizeof(int)*mark.height());

	ARRAY2D<int> h = ARRAY2D<int>(results, mark.height(), mark.width(), sizeof(int)*mark.width()); // on CPU 
	ARRAY2D<int> hc = ARRAY2D<int>(NULL, mark.height(), mark.width(), sizeof(int)*mark.width()); // on CPU


#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif
	for (unsigned int chunk = 0; chunk < mark.size(); chunk++) {
		int blockcount_y = (int)(mark.gpu(chunk).width/COVER_BLOCK) + (mark.gpu(chunk).width%COVER_BLOCK > 0);
		DPRINT("Patterns to process in block %u: %lu\n", chunk, mark.gpu(chunk).width);
		for (int i = ckt.levels(); i >= 0; i--) {
			int levelsize = ckt.levelsize(i);
			do { 
				int simblocks = min(MAX_BLOCKS, levelsize);
				dim3 numBlocks(simblocks,blockcount_y);
				startGate -= simblocks;
				kernCover<<<numBlocks,COVER_BLOCK>>>(ckt.gpu_graph(), mark.gpu(chunk).data, mark.gpu(chunk).pitch,
						merges.data, g_results,pitch, gh_results, h_pitch, startGate+1, 
						chunk*mark.block_width(), startPattern, ckt.offset());
				if (levelsize > MAX_BLOCKS) {
					levelsize -= simblocks;
				} else {
					levelsize = 0;
				}
			} while (levelsize > 0);
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
		}
		hipMemcpy2D(results,h.pitch,g_results,pitch,sizeof(int)*mark.block_width(),mark.height(),hipMemcpyDeviceToHost);
		for (unsigned int j = 0; j < h.width;j++) {
			for (int i = 0; i < ckt.size(); i++) {
				if (ckt.at(i).typ == INPT) {
					*coverage = *coverage + REF2D(int,results, h.pitch, j, i);
				}
			}
		}
		startPattern += mark.gpu(chunk).width;
	}
	free(results);
	hipFree(g_results);
	hipFree(gh_results);
	hipFree(finalcoverage);
	#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif
}

void debugCoverOutput(ARRAY2D<int> results, std::string outfile) {
#ifndef NDEBUG
	std::ofstream ofile(outfile.c_str());
		ofile << "Line:   \t";
	for (unsigned int i = 0; i < results.height; i++) {
		ofile << std::setw(OUTJUST) << i << " ";
	}
	ofile << std::endl;
	for (unsigned int r = 0;r < results.width; r++) {
		ofile << "Vector " << r << ":\t";
		for (unsigned int i = 0; i < results.height; i++) {
			int z = REF2D(int, results.data, results.pitch, r, i);
			ofile << std::setw(OUTJUST) << (int)z << " "; break;
		}
		ofile << std::endl;
	}
	ofile.close();
#endif
}
