#include "hip/hip_runtime.h"
#include "coverkernel.h"
#include <hip/hip_runtime.h>
#include "util/g_utility.h"

#undef OUTJUST
#define OUTJUST 4
#define BLOCK_SIZE 512
void HandleCoverError( hipError_t err, const char *file, uint32_t line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleCoverError( err, __FILE__, __LINE__ ))

// bit masks for packing, trying to reduce # of fetches.
#define GET_H(SRC) (((SRC & 0xFFFF0000) >> 16))
#define GET_C(SRC) ((SRC & 0x0000FFFF))
#define PACK(H, C) ((H << 16) | (C & 0x0000FFFF))
template <uint16_t blockSize>
__device__ void warpReduce(volatile uint32_t* sdata, uint16_t tid) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__global__ void kernSumSingle(GPUNODE* ckt, size_t size, uint32_t* input, size_t height, size_t pitch, uint64_t* meta) {
	__shared__ uint32_t sdata[BLOCK_SIZE]; // only add positive #s
	uint16_t tid = threadIdx.x;
	for (size_t i = 0; i < size; i++) { // iterate over everything in the circuit
		if (ckt[i].type != INPT) 
			continue; // short-circuit for non-PIs
		for (size_t j = 0; j < height; j+=BLOCK_SIZE) { // unrolled to handle reductions of up to BLOCK_SIZE in parallel
			sdata[tid] = 0; // reset shared data segment to 0.
			if (tid+j < height) {
				sdata[tid] = GET_C(REF2D(uint32_t, input, pitch, tid+j, i)); __syncthreads();
//				printf("thread %hu - sdata[%hu] = %u = %u\n", tid, tid,  REF2D(uint32_t, input, pitch, tid+j, i));
			}
			if (BLOCK_SIZE >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
			if (BLOCK_SIZE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256];  } __syncthreads(); }
			if (BLOCK_SIZE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128];  }  __syncthreads();}
			if (BLOCK_SIZE >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64];  } __syncthreads();}
			if (tid < 32) { warpReduce<BLOCK_SIZE>(sdata, tid); } 
			__syncthreads();
			if (threadIdx.x == 0) {
//				printf("Adding %u to total %lu for line %lu pattern %lu/%lu\n", sdata[0], *meta, i, j,height);
				*meta += sdata[0];
			}
			__syncthreads();
		}
	}
}


__global__ void kernCover(const GPUNODE* ckt, uint8_t* mark, const size_t mark_pitch, int2* history,  uint32_t* cover,const size_t cover_pitch, const uint32_t start_offset,const uint32_t pattern_count,const uint32_t start_pattern, uint32_t* offsets) { //, uint32_t* subckt, size_t subckt_size) {
    // cover is the coverage ints we're working with for this pass.
	// mark is the fresh marks
	// hist is the history of the mark status of all lines.
	const uint32_t tid = (blockIdx.y * COVER_BLOCK) + threadIdx.x;
	const int32_t pid = tid + start_pattern; 
	const uint32_t g = blockIdx.x+start_offset;
	const GPUNODE& gate = ckt[g];

	if (tid < pattern_count) {
		const uint8_t cache = REF2D(uint8_t,mark,mark_pitch,tid, g); // cache the current node's marked status.
		// shorthand references to current coverage and history count.
		uint32_t c = 0, h = 0;

		if (gate.po == 1) {
			c = 0;
			h = (cache > 0); // set history = 1 if this line is marked.
		}

		uint32_t resultCache = 0, histCache = 0;
		for (uint32_t i = 0; i < gate.nfo; i++) {
			const uint32_t fot = FIN(offsets,gate.offset,gate.nfi+i);
			const uint32_t tmp = REF2D(uint32_t, cover, cover_pitch, tid, fot);
			resultCache += (GET_C(tmp)); // add this fanout's path count to this node.
			histCache   += (GET_H(tmp)); // add this fanout's history path count to this node.
		}
		assert(histCache <= 0xFFFF);
		assert(resultCache <= 0xFFFF);
		c += resultCache;
		h += histCache;
		{
			// c equals c+h if either history[g] >= pid and line is marked
			const uint32_t tmp = h*(cache > 0)*((history[g].x == pid) || (history[g].y == pid));
			c = c*(cache > 0) + tmp;
			// h equals 0 if neither history[g] >= pid, else h if this line is marked;
			h = h*(cache > 0)*((history[g].x != pid)*(history[g].y != pid));
		}
		if (gate.type == INPT) {
			h = 0; // history results at this point on input gates are useless.
		}
		assert (c <= 0xFFFF); // make sure everything fits into 16 bits
		assert (h <= 0xFFFF); // make sure everything fits into 16 bits
		// Cycle through the fanins of this node and assign them the current value
		const uint32_t tmp = PACK(h,c);
		REF2D(uint32_t, cover     , cover_pitch , tid, g) = tmp;
	}
}


float gpuCountPaths(const GPU_Circuit& ckt, GPU_Data& mark, const void* merge,
		uint64_t* coverage, size_t chunk, size_t startPattern) {

	int2* merges = (int2*)merge;
	HANDLE_ERROR(hipGetLastError()); // check to make sure there aren't any errors going into function.

	std::ofstream cfile("gpucover.log", std::ios::out);
	uint32_t *g_results;
#ifdef LOGEXEC
	uint32_t *d_results; // debug results 
#endif //LOGEXEC
	uint64_t *finalcoverage;
	uint32_t startGate;
	size_t pitch;
	//	const size_t summedPatterns = (mark.width() / (MERGE_SIZE*2)) + ((mark.width() % (MERGE_SIZE*2)) > 0);

	hipMalloc(&finalcoverage, sizeof(uint64_t));
	hipMemset(finalcoverage, 0, sizeof(uint64_t)); // set value to 0 explicitly
	HANDLE_ERROR(hipGetLastError()); // checking that last memory operation completed successfully.

#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif
	uint32_t pcount = 0;
	gpuCheckMemory();
	DPRINT("Allocating for chunk %lu \n", chunk);
	DPRINT("Allocating %lu bytes for results... ",sizeof(uint32_t)*mark.gpu(chunk).width*mark.height());
	hipMallocPitch(&g_results,&pitch, sizeof(uint32_t)*mark.gpu(chunk).width,mark.height());
	DPRINT("Allocated %lu bytes for results.\n",pitch*mark.height());
	HANDLE_ERROR(hipGetLastError()); // checking last function
#ifdef LOGEXEC
	d_results = (uint32_t*)malloc(sizeof(uint32_t)*mark.block_width()*mark.height());
#endif // LOGEXEC
	hipMemset(g_results, 0, mark.height()*pitch);
	HANDLE_ERROR(hipGetLastError()); // checking last function

	pcount += mark.gpu(chunk).width;
	startGate = ckt.size();
	const uint32_t blockcount_y = (uint32_t)(mark.gpu(chunk).width/COVER_BLOCK) + (mark.gpu(chunk).width%COVER_BLOCK > 0);
	for (uint32_t i2 = 0; i2 <= ckt.levels(); i2++) {
		const int32_t i = (ckt.levels() - (i2));
		uint32_t levelsize = ckt.levelsize(i);
		do { 
			uint32_t simblocks = min(MAX_BLOCKS, levelsize);
			dim3 numBlocks(simblocks,blockcount_y);
			startGate -= simblocks;
			assert((uint32_t)startGate + simblocks <= ckt.size());
			kernCover<<<numBlocks,COVER_BLOCK>>>(ckt.gpu_graph(), mark.gpu(chunk).data, mark.gpu(chunk).pitch,
					merges, g_results,pitch, startGate, 
					mark.gpu(chunk).width, startPattern, ckt.offset());
			if (levelsize > MAX_BLOCKS) {
				levelsize -= simblocks;
			} else {
				levelsize = 0;
			}
		} while (levelsize > 0);
		if (i == 0) {
			// Sum for all gates and patterns
			kernSumSingle<<<1,BLOCK_SIZE>>>(ckt.gpu_graph(), ckt.levelsize(0), g_results, mark.gpu(chunk).width, pitch, finalcoverage); // multithreaded, single block GPU add.
			hipDeviceSynchronize();
			hipMemcpy(coverage, finalcoverage, sizeof(uint64_t), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
		}
	}
	startPattern += mark.gpu(chunk).width;
	assert(startGate == 0);
	// dump to file for debugging.

#ifdef LOGEXEC
	hipMemcpy2D(d_results, sizeof(uint32_t)*mark.gpu(chunk).width, g_results, pitch, sizeof(uint32_t)*mark.gpu(chunk).width, mark.height(), hipMemcpyDeviceToHost);
	debugCover(ckt, d_results, mark.gpu(chunk).width, mark.height(), cfile);
	free(d_results);
#endif // LOGEXEC
	hipMemcpy(coverage, finalcoverage, sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipFree(finalcoverage);	hipFree(g_results); // clean up.
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting


#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif

#ifdef LOGEXEC
	cfile.close();
#endif
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif
}
void debugCover(const Circuit& ckt, uint32_t *cover, size_t patterns, size_t lines, std::ofstream& ofile) {
#ifndef NDEBUG
	std::cerr << "Patterns: " << patterns << "; Lines: " << lines << std::endl;
	ofile << "Gate:   \t";
	int i = 0;
	while (ckt.at(i).typ == INPT) {
		ofile << std::setw(OUTJUST) << i++ << " ";
	}
	ofile << "\n";
	for (uint32_t r = 0; r < patterns; r++) {
		ofile << "Vector " << r << ":\t";
		for (uint32_t i = 0; i < lines; i++) {
			if (ckt.at(i).typ == INPT) {
				const uint32_t z = GET_C(REF2D(uint32_t, cover, sizeof(uint32_t)*patterns, r, i));
				ofile << std::setw(OUTJUST) << z << " ";
			}
		}
		ofile << "\n";
	}
#endif
}
void debugCoverOutput(ARRAY2D<uint32_t> results, std::string outfile) {
#ifndef NDEBUG
	std::ofstream ofile(outfile.c_str());
		ofile << "Line:   \t";
	for ( uint32_t i = 0; i < results.height; i++) {
		ofile << std::setw(OUTJUST) << i << " ";
	}
	ofile << std::endl;
	for ( uint32_t r = 0;r < results.width; r++) {
		ofile << "Vector " << r << ":\t";
		for ( uint32_t i = 0; i < results.height; i++) {
			uint32_t z = REF2D(uint32_t, results.data, results.pitch, r, i);
			ofile << std::setw(OUTJUST) << (uint32_t)GET_C(z) << " "; break;
		}
		ofile << std::endl;
	}
	ofile.close();
#endif
}
