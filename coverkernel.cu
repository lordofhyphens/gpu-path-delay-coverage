#include "hip/hip_runtime.h"
#include "coverkernel.h"
#include <hip/hip_runtime.h> 
void HandleCoverError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define SUM(A, B, DATA) (DATA[A]+DATA[B])
__global__ void kernSum(GPUNODE* ckt, int* input, size_t height, size_t pitch,int* meta, int mpitch, int startGate) {
	int tid = threadIdx.x;
	int gid = blockIdx.y+startGate;
	__shared__ int sdata[MERGE_SIZE];
	unsigned int i = blockIdx.x*(MERGE_SIZE*2) + threadIdx.x;
	sdata[tid] = 0;
	// need to add i and i+MERGE_SIZE for those gates whose type is INPT
	if (ckt[gid].type == INPT && i < height) {
		if (i+MERGE_SIZE > height) { // correcting for blocks smaller than MERGE_SIZE
			sdata[tid] = (REF2D(int,input,pitch,i,gid));
//			printf("%s:%d - input[%d][%d] = %d\n", __FILE__,__LINE__, i, gid, REF2D(int,input,pitch,i,gid));
		} else {
			sdata[tid] = (REF2D(int,input,pitch,i,gid)) + (REF2D(int,input,pitch,i+MERGE_SIZE,gid));
		}
		__syncthreads();

		// this is loop unrolling
		// do reduction in shared mem, comparisons against MERGE_SIZE are done at compile time.
		if (MERGE_SIZE >= 1024) { if (tid < 512 && tid+512 < height) { sdata[tid] = SUM(tid, tid+512,sdata); } __syncthreads(); }
		if (MERGE_SIZE >= 512) { if (tid < 256 && tid+256 < height) { sdata[tid] = SUM(tid, tid+256,sdata); } __syncthreads(); }
		if (MERGE_SIZE >= 256) { if (tid < 128 && tid+128 < height) { sdata[tid] = SUM(tid, tid+128,sdata); } __syncthreads(); }
		if (MERGE_SIZE >= 128) { if (tid <  64 && tid+64 < height) { sdata[tid] = SUM(tid, tid+64,sdata); } __syncthreads(); }
		if (tid < 32) {
			// Within a warp,  don't need __syncthreads();
			if (MERGE_SIZE >=  64) { if (tid+32 < height) { sdata[tid] = SUM(tid, tid + 32,sdata); } }
			if (MERGE_SIZE >=  32) { if (tid+16 < height) { sdata[tid] = SUM(tid, tid + 16,sdata); } }
			if (MERGE_SIZE >=  16) { if (tid+8 < height) { sdata[tid] = SUM(tid, tid +  8,sdata); } }
			if (MERGE_SIZE >=   8) { if (tid+4 < height) { sdata[tid] = SUM(tid, tid +  4,sdata); } }
			if (MERGE_SIZE >=   4) { if (tid+2 < height) { sdata[tid] = SUM(tid, tid +  2,sdata); } }
			if (MERGE_SIZE >=   2) { if (tid+1 < height) { sdata[tid] = SUM(tid, tid +  1,sdata); } }
		}

		// at this point, we have the sum for this gate position of the lowest. Correct by 1 to compensate for above.

		__syncthreads();
		if (threadIdx.x == 0) { REF2D(int, meta, mpitch, blockIdx.x, gid) = sdata[0]; }
		//if (threadIdx.x == 0) { printf("%s:%d - %d\n", __FILE__,__LINE__, sdata[0]); }
		__syncthreads();
	} else {
		if (threadIdx.x == 0) {REF2D(int, meta, mpitch, blockIdx.x, gid) = 0;}
	}
}

__device__ inline int subCktFan(int* subckt, int subckt_size, int tgt) {
	// scan through the subckt list looking for tgt
	for (int i = 0; i < subckt_size; i++) { if (subckt[i] == tgt) { return i;} }
	return -1;
}
#define HANDLE_ERROR( err ) (HandleCoverError( err, __FILE__, __LINE__ ))

__global__ void kernCover(const GPUNODE* ckt, char* mark,size_t mark_pitch, int* history,  int* cover, size_t cover_pitch, int* hist_cover, size_t hcover_pitch,int start_offset, int pattern_count, int start_pattern, int* offsets) { //, int* subckt, size_t subckt_size) {
    // cover is the coverage ints we're working with for this pass.
    // mark is the fresh marks
    // hist is the history of the mark status of all lines.
	int tid = (blockIdx.y * COVER_BLOCK) + threadIdx.x;
	int pid = tid + start_pattern; 
//	int g = subckt[blockIdx.x+start_offset];
	int g = blockIdx.x+start_offset;
	int resultCache = 0;
	int histCache = 0;
	char cache;
	GPUNODE gate = ckt[g];
//	printf("%s:%d - pid = %d, pattern_count=%d\n",__FILE__,__LINE__,pid, pattern_count);
	if (pid < pattern_count) {
		cache = REF2D(char,mark,mark_pitch,tid, g); // cache the current node's marked status.
		// shorthand references to current coverage and history count.
		int c, h;
		c = REF2D(int, cover     , cover_pitch , tid, g);
		h = REF2D(int, hist_cover, hcover_pitch, tid, g);

		if (gate.po == 1) {
			c = 0;
            h = (cache > 0); // set history = 1 if this line is marked.
        }

		if (gate.nfo > 1) {
			for (int i = 0; i < gate.nfo; i++) {
				int fot = FIN(offsets,gate.offset+gate.nfi,i); // reference to current fan-out
				resultCache += REF2D(int,cover,cover_pitch,tid,fot); // add this fanout's path count to this node.
				histCache += REF2D(int,hist_cover,hcover_pitch,tid,fot); // add this fanout's history path count to this node.
			}
			c = resultCache;
			h = histCache;
		}
		if (gate.type != FROM) {
			// needs to equal c+h if history[g] >= pid and line is marked
			c = (c + h)*(cache > 0)*(history[g] >= pid);
			// needs to equal 0 if history[g] >= pid;
			h = h*(cache > 0)*(history[g] < pid);

            for (int i = 0; i < gate.nfi; i++) {
//				int fin = subCktFan(subckt, subckt_size, FIN(offsets,gate.offset,i));
				int fin = FIN(offsets,gate.offset,i);
				if (fin >= 0) {
					REF2D(int,cover,cover_pitch,tid,fin) = c; //REF2D(int,cover,cover_pitch,tid,g);
					REF2D(int,hist_cover,hcover_pitch,tid,fin) = h; //REF2D(int,hist_cover,hcover_pitch,tid,g);
				}
			}
        }
		REF2D(int, cover     , cover_pitch , tid, g) = c;
		REF2D(int, hist_cover, hcover_pitch, tid, g) = h;

//		printf("%s:%d - history[%d] = %d\n", __FILE__, __LINE__, g, history[g]);
//		printf("%s:%d - cover[%d][%d] = %d, history[%d][%d] = %d \n",__FILE__, __LINE__, tid,g, c, tid,g, h);
	}
}


float gpuCountPaths(const GPU_Circuit& ckt, GPU_Data& mark, ARRAY2D<int> merges, long unsigned int* coverage) {
	HANDLE_ERROR(hipGetLastError()); // check to make sure there aren't any errors going into function.
	int* results, *g_results, *gh_results;
	int* temp_coverage, *g_coverage;
	long* finalcoverage;
	*coverage = 0;
	int startGate;
	size_t pitch, h_pitch;
	int startPattern = 0;
	size_t summedPatterns = (mark.width() / (MERGE_SIZE*2)) + ((mark.width() % (MERGE_SIZE*2)) > 0);
//	int* debug; 
//	debug = (int*)malloc(sizeof(int)*mark.block_width()*mark.height());
	hipMalloc(&finalcoverage, sizeof(long));
	HANDLE_ERROR(hipGetLastError()); // checking last function
	hipMallocPitch(&g_results,&pitch, sizeof(int)*mark.block_width(),mark.height());
	HANDLE_ERROR(hipGetLastError()); // checking last function
	hipMallocPitch(&gh_results,&h_pitch,sizeof(int)*mark.block_width(),mark.height());
	HANDLE_ERROR(hipGetLastError()); // checking last function

	hipMemset(g_results, 0, mark.height()*pitch);
	HANDLE_ERROR(hipGetLastError()); // checking last function
	hipMemset(gh_results, 0, mark.height()*h_pitch);
	HANDLE_ERROR(hipGetLastError()); // checking last function
	results = (int*)malloc(mark.block_width()*sizeof(int)*mark.height());
	hipHostAlloc(&temp_coverage, sizeof(int)*mark.height()*summedPatterns, hipHostMallocWriteCombined | hipHostMallocMapped);
	HANDLE_ERROR(hipGetLastError()); // checking last function
	hipHostGetDevicePointer(&g_coverage, temp_coverage, 0);
	HANDLE_ERROR(hipGetLastError()); // checking last function
//	h_results = (int*)malloc(mark.block_width()*sizeof(int)*mark.height());

	ARRAY2D<int> h = ARRAY2D<int>(results, mark.height(), mark.width(), sizeof(int)*mark.width()); // on CPU 
	ARRAY2D<int> hc = ARRAY2D<int>(NULL, mark.height(), mark.width(), sizeof(int)*mark.width()); // on CPU


#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif
	int pcount = 0;
	////DPRINT("%s:%d - level count: %d\n", __FILE__,__LINE__, ckt.levels());
	for (unsigned int chunk = 0; chunk < mark.size(); chunk++) {
		pcount += mark.gpu(chunk).width;
		startGate = ckt.size();
		int blockcount_y = (int)(mark.gpu(chunk).width/COVER_BLOCK) + (mark.gpu(chunk).width%COVER_BLOCK > 0);
		//DPRINT("Patterns to process in block %u: %lu\n", chunk, mark.gpu(chunk).width);
		for (int i = ckt.levels(); i >= 0; i--) {
			int levelsize = ckt.levelsize(i);
			do { 
				int simblocks = min(MAX_BLOCKS, levelsize);
				dim3 numBlocks(simblocks,blockcount_y);
				startGate -= simblocks;
	//			//DPRINT("%s:%d - running cover %d - %d\n", __FILE__,__LINE__, i, levelsize);
				assert(startGate < ckt.size());
				assert(startGate >= 0);
				kernCover<<<numBlocks,COVER_BLOCK>>>(ckt.gpu_graph(), mark.gpu(chunk).data, mark.gpu(chunk).pitch,
						merges.data, g_results,pitch, gh_results, h_pitch, startGate, 
						pcount, startPattern, ckt.offset());
				if (levelsize > MAX_BLOCKS) {
					levelsize -= simblocks;
				} else {
					levelsize = 0;
				}
			} while (levelsize > 0);
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
		}
		/*
		hipMemcpy2D(debug,sizeof(int)*mark.block_width(),g_results,pitch,sizeof(int)*mark.block_width(),mark.height(),hipMemcpyDeviceToHost);
		for (unsigned int r = 0;r < mark.block_width(); r++) {
			//DPRINT("Vector %d:\t",r);
			for (unsigned int i = 0; i < mark.height(); i++) {
				int z = REF2D(int, debug, sizeof(int)*mark.block_width(), r, i);
				//DPRINT("%2d ", z);
			}
			//DPRINT("\n");
		}
*/
		size_t remaining_blocks = mark.height();
		int count = 0;
		do {
			size_t block_x = summedPatterns;//(h.width / MERGE_SIZE) + ((h.width % MERGE_SIZE) > 0);
			size_t block_y = (remaining_blocks > 65535 ? 65535 : remaining_blocks);
			dim3 blocks(block_x, block_y);
//			//DPRINT("%s:%d - (%lu,%lu)\n", __FILE__,__LINE__, block_x, block_y);
			kernSum<<<blocks, MERGE_SIZE>>>(ckt.gpu_graph(),g_results, h.width, pitch, g_coverage, sizeof(int)*summedPatterns, count);
			hipDeviceSynchronize();
			count++;
			if (remaining_blocks > 65535) { remaining_blocks -= 65535; } 
			count += 65535;
			block_y = (remaining_blocks > 65535 ? 65535 : remaining_blocks);
		} while (remaining_blocks > 65535);
//		//DPRINT("%s:%d - summedPatterns: %lu\n", __FILE__,__LINE__, summedPatterns);
		for (unsigned int j = 0; j < summedPatterns; j++) {
			for (int i = 0; i < ckt.size(); i++) {
//				//DPRINT("%d ", REF2D(int, temp_coverage, sizeof(int)*summedPatterns, i, j));
				*coverage = *coverage + REF2D(int, temp_coverage, sizeof(int)*summedPatterns, i, j);
			}
//			//DPRINT("\n");
		}
		startPattern += mark.gpu(chunk).width;
	}
	free(results);
	hipFree(g_results);
	hipFree(gh_results);
	hipFree(finalcoverage);
	#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif
}

void debugCoverOutput(ARRAY2D<int> results, std::string outfile) {
#ifndef NDEBUG
	std::ofstream ofile(outfile.c_str());
		ofile << "Line:   \t";
	for (unsigned int i = 0; i < results.height; i++) {
		ofile << std::setw(OUTJUST) << i << " ";
	}
	ofile << std::endl;
	for (unsigned int r = 0;r < results.width; r++) {
		ofile << "Vector " << r << ":\t";
		for (unsigned int i = 0; i < results.height; i++) {
			int z = REF2D(int, results.data, results.pitch, r, i);
			ofile << std::setw(OUTJUST) << (int)z << " "; break;
		}
		ofile << std::endl;
	}
	ofile.close();
#endif
}
