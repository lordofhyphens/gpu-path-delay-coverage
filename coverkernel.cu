#include "hip/hip_runtime.h"
#include "coverkernel.h"
#include <hip/hip_runtime.h> 
void HandleCoverError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        DPRINT( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}


#define HANDLE_ERROR( err ) (HandleCoverError( err, __FILE__, __LINE__ ))
__global__ void kernCover(const GPUNODE* ckt, char* mark,size_t mark_pitch, int* history,  int* cover,size_t cover_pitch, int* hist_cover, size_t hcover_pitch,int start_offset, int start_pattern, int pattern_count, int* offsets) {
    // cover is the coverage ints we're working with for this pass.
    // mark is the fresh marks
    // hist is the history of the mark status of all lines.
	int tid = (blockIdx.y * blockDim.y) + threadIdx.x;
	int pid = tid + start_pattern; 
	int g = blockIdx.x+start_offset;
	int resultCache = 0;
	int histCache = 0;
	GPUNODE gate = ckt[g];
	if (pid < pattern_count) {
		int *h = ADDR2D(int,cover,cover_pitch, tid, g), *c = ADDR2D(int,hist_cover,hcover_pitch,tid,g);
//		printf("%s - Line: %d, gate: %d\n",__FILE__, __LINE__,g);
		
        if (gate.po == 1) {
			*c = 0;
            *h = (REF2D(char,mark,mark_pitch, tid, g) > 0);
        }
		if (ckt[g].nfo > 1) {
			for (int i = 0; i < ckt[g].nfo; i++) {
				resultCache += REF2D(char,cover,cover_pitch,tid,FIN(offsets,ckt[g].offset+ckt[g].nfi,i));
				histCache += REF2D(char,hist_cover,hcover_pitch,tid,FIN(offsets,ckt[g].offset+ckt[g].nfi,i));
			}
//			printf("%s:%d - results[%d][%d] = %d\n", __FILE__, __LINE__, tid, g, resultCache);
			REF2D(int, cover, cover_pitch, tid, g) = resultCache;
			REF2D(int, hist_cover, hcover_pitch, tid, g) = histCache;
		}
		if (ckt[g].type == INPT) {
			*c = *c + (*h * (NMARKEDG(REF2D(char,mark,mark_pitch,tid,g),history,g,pid)));
			*h = *h * !(NMARKEDG(REF2D(char,mark,mark_pitch,tid,g),history,g,pid));
        } else if (gate.type != FROM) {
			*c = *c + (*h * (NMARKEDG(REF2D(char,mark,mark_pitch,tid,g),history,g,pid)));
			*h = *h * (!NMARKEDG(REF2D(char,mark,mark_pitch,tid,g),history,g,pid));
            for (int i = 0; i < gate.nfi; i++) {
				int *fin = ADDR2D(int,cover,cover_pitch,tid,FIN(offsets,gate.offset,i));
				int *fin_h = ADDR2D(int,hist_cover,hcover_pitch,tid,FIN(offsets,gate.offset,i));
                *fin = *c;
                *fin_h = *h;
            }
        }
		if (tid == 0)
			printf("%s:%d - history[%d] = %d, pid = %d\n", __FILE__,__LINE__,g, history[g], pid);
//		printf("%s:%d - results[%d][%d] = %d\n", __FILE__, __LINE__, tid, g, REF2D(int,cover,cover_pitch,tid,g));
//		printf("%s:%d - h_results[%d][%d] = %d\n", __FILE__, __LINE__, tid, g, REF2D(int,hist_cover,hcover_pitch,tid,g));
	}
}


float gpuCountPaths(const GPU_Circuit& ckt, GPU_Data& mark, ARRAY2D<int> merges, int* coverage) {
	int* results, *g_results, *gh_results, *h_results;
	int startGate=ckt.size();
	hipHostAlloc(&results,sizeof(int)*mark.width()*mark.height(), hipHostMallocMapped);
	hipHostAlloc(&h_results,sizeof(int)*mark.width()*mark.height(), hipHostMallocMapped);
	hipHostGetDevicePointer(&g_results, results, 0);
	hipHostGetDevicePointer(&gh_results, h_results, 0);
	ARRAY2D<int> h = ARRAY2D<int>(results, mark.height(), mark.width(), sizeof(int)*mark.width());
	ARRAY2D<int> hc = ARRAY2D<int>(h_results, mark.height(), mark.width(), sizeof(int)*mark.width());


	int blockcount_y = (int)(mark.block_width()/COVER_BLOCK) + (mark.block_width()%COVER_BLOCK > 0);
#ifndef NTIMING
	float elapsed;
	timespec start, stop;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
#endif
	for (unsigned int chunk = 0; chunk < mark.size(); chunk++) {
		for (int i = ckt.levels(); i >= 0; i--) {
			dim3 numBlocks(ckt.levelsize(i),blockcount_y);
			startGate -= ckt.levelsize(i);
			kernCover<<<numBlocks,COVER_BLOCK>>>(ckt.gpu_graph(),mark.gpu(chunk),mark.pitch(),merges.data,g_results,h.pitch, gh_results,hc.pitch,startGate, chunk*mark.block_width(),mark.width(),ckt.offset());
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError()); // check to make sure we aren't segfaulting
		}
	}
	*coverage = 0;
	for (unsigned int j = 0; j < h.width;j++) {
		for (int i = 0; i < ckt.size(); i++) {
			if (ckt.at(i).typ == INPT) {
				*coverage = *coverage + REF2D(int,h_results, hc.pitch, j, i);
	//			std::clog << "cover[" << j << "][" << i << "]: " << REF2D(int,h_results, sizeof(int)*mark.width(), j, i) << std::endl;
			}
		}
	}
#ifndef NTIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	elapsed = floattime(diff(start, stop));
#endif
	hipHostFree(results);
	hipHostFree(h_results);
#ifndef NTIMING
	return elapsed;
#else 
	return 0.0;
#endif
}

void debugCoverOutput(ARRAY2D<int> results, std::string outfile) {
#ifndef NDEBUG
	std::ofstream ofile(outfile.c_str());
		ofile << "Line:   \t";
	for (unsigned int i = 0; i < results.height; i++) {
		ofile << std::setw(OUTJUST) << i << " ";
	}
	ofile << std::endl;
	for (unsigned int r = 0;r < results.width; r++) {
		ofile << "Vector " << r << ":\t";
		for (unsigned int i = 0; i < results.height; i++) {
			int z = REF2D(int, results.data, results.pitch, r, i);
			ofile << std::setw(OUTJUST) << (int)z << " "; break;
		}
		ofile << std::endl;
	}
	ofile.close();
#endif
}
